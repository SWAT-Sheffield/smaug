#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_hde2.cuh"





__global__ void hyperdifesource2_parallel(struct params *p, real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii,ii1,ii0;
  real fip,fim1;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real rdx;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));


   

  rdx=(((p->dx[0])*(dim==0))+(p->dx[1])*(dim==1));

   
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;

//if(i<((p->n[0])) && j<((p->n[1])))
  if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
  {


 
//dwn1[fencode_hde2(p,i,j,field)]=( wtemp[fencode_hde2(p,i,j,hdnur)] * grad1r_hde2(wtemp,p,i,j,tmp1,dim) - wtemp[fencode_hde2(p,i,j,hdnul)] *grad1l_hde2(wtemp,p,i,j,tmp1,dim)             )/rdx;
//dwn1[fencode_hde2(p,i,j,field)]=( wd[fencode_hde2(p,i,j,hdnur)] * grad1r_hde2(wtemp,p,i,j,tmp1,dim) - wd[fencode_hde2(p,i,j,hdnul)] *grad1l_hde2(wtemp,p,i,j,tmp1,dim)             );

wtemp[fencode_hde2(p,i,j,tmp2)]= grad1l_hde2(wtemp,p,i,j,tmp1,dim) ;
wtemp[fencode_hde2(p,i,j,tmp3)]= grad1r_hde2(wtemp,p,i,j,tmp1,dim) ;


  }
}
__syncthreads();



   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;

if(i<((p->n[0])) && j<((p->n[1])))
//  if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
  {


 
//dwn1[fencode_hde2(p,i,j,field)]=( wtemp[fencode_hde2(p,i,j,hdnur)] * grad1r_hde2(wtemp,p,i,j,tmp1,dim) - wtemp[fencode_hde2(p,i,j,hdnul)] *grad1l_hde2(wtemp,p,i,j,tmp1,dim)             )/rdx;
//dwn1[fencode_hde2(p,i,j,field)]=( wd[fencode_hde2(p,i,j,hdnur)] * grad1r_hde2(wtemp,p,i,j,tmp1,dim) - wd[fencode_hde2(p,i,j,hdnul)] *grad1l_hde2(wtemp,p,i,j,tmp1,dim)             );

//wtemp[fencode_hde2(p,i,j,tmp2)]= grad1r_hde2(wtemp,p,i,j,tmp1,dim) ;
//wtemp[fencode_hde2(p,i,j,tmp3)]= grad1l_hde2(wtemp,p,i,j,tmp1,dim) ;
dwn1[fencode_hde2(p,i,j,field)]=( wtemp[fencode_hde2(p,i,j,hdnur)] *wtemp[fencode_hde2(p,i,j,tmp3)] - wtemp[fencode_hde2(p,i,j,hdnul)] *wtemp[fencode_hde2(p,i,j,tmp2)])/rdx;


  }
}
__syncthreads();



   
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;


			 //if(i>1 && j >1 && i<(ni-2) && j<(nj-2))
                         if(i<((p->n[0])) && j<((p->n[1])))
                         {
                              //                                                                                  - sign here same as vac maybe a +
                              wmod[fencode_hde2(p,i,j,field)+(ordero*NVAR*(p->n[0])*(p->n[1]))]=wmod[fencode_hde2(p,i,j,field)+(ordero*NVAR*(p->n[0])*(p->n[1]))]+dt*dwn1[fencode_hde2(p,i,j,field)]; 
//wmod[fencode_hde2(p,i,j,f)+ordero*NVAR*(p->n[0])*(p->n[1])]=dwn1[fencode_hde2(p,i,j,f2)];
                              //dwn1[fencode_hde2(p,i,j,f)]=0;
                         }
              //  }	
}
  __syncthreads();



 
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_hde2(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}





int cuhyperdifesource2(struct params **p, real **w, real **wnew, struct params **d_p, real **d_w, real **d_wnew,  real **d_wmod, real **d_dwn1, real **d_wd, int order,int ordero, real **d_wtemp, int field, int dim)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
     hyperdifesource2_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}







