#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_dc2.cuh"

__device__ __host__
real transportflux_dc2 (real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field, int direction) {

  real ddc=0;
  real fi, fim1;
  real  fip2=0, fim2=0;
 // real ddc1;
  real ddcx=0,ddcy=0;

   real flux=0;

   //transport flux
    switch(direction)
  {
     case 0:
      flux= wd[fencode_dc2(p,ix,iy,vel1)]*w[fencode_dc2(p,ix,iy,field)];
     break;
     case 1:
      flux= wd[fencode_dc2(p,ix,iy,vel2)]*w[fencode_dc2(p,ix,iy,field)];
     break;
     case 2:
      flux= wd[fencode_dc2(p,ix,iy,vel3)]*w[fencode_dc2(p,ix,iy,field)];
     break;
   }
  return flux;


  //return ( ddc1-ddc2);
}




__device__ __host__
real fluxb1(real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field, int direction) {

  real ddc=0;
  real fi, fim1;
  real  fip2=0, fim2=0;
 // real ddc1;
  real ddcx=0,ddcy=0;

   real flux=0;


        #ifdef USE_SAC
      		flux= -(w[fencode_dc2(p,ix,iy,field)]+w[fencode_dc2(p,ix,iy,field+5)])*w[fencode_dc2(p,ix,iy,mom1+direction)]/(w[fencode_dc2(p,ix,iy,rho)]+w[fencode_dc2(p,ix,iy,rhob)]);

         #else
      		flux= -w[fencode_dc2(p,ix,iy,field)]*w[fencode_dc2(p,ix,iy,mom1+direction)]/w[fencode_dc2(p,ix,iy,rho)];
         #endif


  return flux;
}



__device__ __host__
real fluxe1(real *dw, real *wd, real *w, struct params *p,int ix, int iy, int direction) {

  real ddc=0;
  real fi, fim1;
  real  fip2=0, fim2=0;
 // real ddc1;
  real ddcx=0,ddcy=0;

   real flux=0;


        #ifdef USE_SAC
      		flux= -w[fencode_dc2(p,ix,iy,b1+direction)]*wd[fencode_dc2(p,ix,iy,bdotv)]+(w[fencode_dc2(p,ix,iy,mom1+direction)]*wd[fencode_dc2(p,ix,iy,pressuret)]/w[fencode_dc2(p,ix,iy,rho)]);
         #else
      		flux= -w[fencode_dc2(p,ix,iy,b1+direction)]*wd[fencode_dc2(p,ix,iy,bdotv)]+(w[fencode_dc2(p,ix,iy,mom1+direction)]*wd[fencode_dc2(p,ix,iy,pressuret)]/w[fencode_dc2(p,ix,iy,rho)]);
         #endif

  return flux;


  //return ( ddc1-ddc2);
}








__device__ __host__
int computefluxe(real *dw, real *wd, real *w, struct params *p,int ix, int iy) {

  int field, direction;
  int status=0;
  for(direction=0;direction<3;direction++)
         #ifdef USE_SAC
	     wd[fencode_dc2(p,ix,iy,f1+direction)]= transportflux_dc2(dw,wd,w,p,ix,iy,energy,direction)+transportflux_dc2(dw,wd,w,p,ix,iy,energyb,direction)+fluxe1(dw,wd,w,p,ix,iy,direction);
         #else
             wd[fencode_dc2(p,ix,iy,f1+direction)]= transportflux_dc2(dw,wd,w,p,ix,iy,energy,direction)+fluxe1(dw,wd,w,p,ix,iy,direction);
         #endif
        
  return ( status);
}

__device__ __host__
int computefluxb (real *dw, real *wd, real *w, struct params *p,int ix, int iy, int field) {

  int direction;
  int status=0;
  for(direction=0;direction<3;direction++)
         #ifdef USE_SAC
     wd[fencode_dc2(p,ix,iy,f1+direction)]= transportflux_dc2(dw,wd,w,p,ix,iy,field,direction)+transportflux_dc2(dw,wd,w,p,ix,iy,field+5,direction)+fluxb1(dw,wd,w,p,ix,iy,field,direction);
         #else
     wd[fencode_dc2(p,ix,iy,f1+direction)]= transportflux_dc2(dw,wd,w,p,ix,iy,field,direction)+fluxb1(dw,wd,w,p,ix,iy,field,direction);
         #endif
        
  return ( status);
}

__device__ __host__
int divflux_dc2(real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field) {

  int direction;
  int status=0;
  real divflux=0;
  dw[fencode_dc2(p,ix,iy,field)]= grad_dc2(wd,p,ix,iy,f1,0)+grad_dc2(wd,p,ix,iy,f2,1);      

  return ( status);
}





//rho, mom1, mom2, mom3, energy, b1, b2, b3
__device__ __host__
void computeflux_dc2 (real *dw, real *wd, real *w, struct params *p,int ix, int iy, int field) {

  //int status=0;
  switch(field)
  {
     case energy:
      computefluxe(dw,wd,w,p,ix,iy);
      // add the following terms for SAC
      // del((b bb+ bb b).v)+ptb del v - bb bb del v
     break;
     case b1:
      computefluxb(dw,wd,w,p,ix,iy,field);
     break;
     case b2:
       computefluxb(dw,wd,w,p,ix,iy,field);
     break;
     case b3:
      computefluxb(dw,wd,w,p,ix,iy,field);
     break;
  }
  //return ( status);
}



__global__ void derivcurrent2_parallel(struct params *p, real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd, int order)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
 // int index;
  int ni=p->n[0];
  int nj=p->n[1];
 // real dt=p->dt;
  //real dy=p->dx[1];
 // real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  

   j=iindex/(p->n[0]);
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*(p->n[0]));


               for(int f=energy; f<NVAR; f++)
               {

			if(i>1 && j >1 && i<(ni-2) && j<(nj-2))
                  		computeflux_dc2(dwn1+(NVAR*(p->n[0])*(p->n[1])*order),wd,wmod,p,i,j,f); 

                        //might need to set boundaries correctly 
                        __syncthreads();
			if(i>1 && j >1 && i<(ni-2) && j<(nj-2))
                                divflux_dc2(dwn1+(NVAR*(p->n[0])*(p->n[1])*order),wd,wmod,p,i,j,f); 
                }


  
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_dc2(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}




int cuderivcurrent2(struct params **p, real **w, real **wnew, struct params **d_p, real **d_w, real **d_wnew,  real **d_wmod, real **d_dwn1, real **d_wd, int order)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;
 //  hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
 // if(order==0)
    hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
     derivcurrent2_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd, order);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}


