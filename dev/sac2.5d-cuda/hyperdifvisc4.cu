#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_hdv4.cuh"




__global__ void hyperdifvisc4_parallel(struct params *p, real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd, int order, real *wtemp, real *wtemp1, real *wtemp2, int field, int dim,int hand)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int is,js;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  real maxt=0,max3=0, max1=0;
  
   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));

int bfac1,bfac2,bfac3;
//int bfac1=(field==rho || field>mom2)+(field>rho && field<energy);
//int bfac2= (field==rho || field>mom2);
//int bfac3=(field>rho && field<energy);
int shift=order*NVAR*(p->n[0])*(p->n[1]);


   //tmp1  tmp_nuI
   //tmp2  d3r
    //tmp3 d1r
//tmp4    md3r
//tmp5    md1r
//tmp6    d3l
//tmp7    d1l
//tmp8    md3l
//tmp9    md1l





 /*  for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
          if( i<(ni) && j<(nj))
            {
                  bc_periodic1_hdv4(wtemp,p,i,j,tmp4);
                  bc_periodic1_hdv4(wtemp,p,i,j,tmp5);
             }

}
                __syncthreads();

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
          if( i<(ni) && j<(nj))
            {
                  bc_periodic2_hdv4(wtemp,p,i,j,tmp4);
                  bc_periodic2_hdv4(wtemp,p,i,j,tmp5);
             }
}
                __syncthreads();*/



   p->maxviscoef=0;


    //finally update nur and nul
//tmp4    md3r
//tmp5    md1r
   
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;


   if(i>1 && i<((p->n[0])-2) && j>1 && j<((p->n[1])-2))
   {
     //wd[fencode_hdv4(p,i,j,hdnur+hand)]=wtemp2[fencode_hdv4(p,i+1,j+1,tmpnui)];
     if(wtemp[fencode_hdv4(p,i,j,tmp5)]>0)
{
//p->cmax=1.0;
	wd[fencode_hdv4(p,i,j,hdnur+hand)]=((dim==0)*(p->dx[0])+(dim==1)*(p->dx[1]))*(p->cmax)*(p->chyp[field])*wtemp[fencode_hdv4(p,i,j,tmp4)]/wtemp[fencode_hdv4(p,i,j,tmp5)];

          //wd[fencode_hdv4(p,i,j,hdnur+hand)]=wtemp[fencode_hdv4(p,i,j,tmp4)];
	//wd[fencode_hdv4(p,i,j,hdnul+hand)]=0.01;
}
     else
        wd[fencode_hdv4(p,i,j,hdnur+hand)]=0;

    

     //temporary trap for debugging
     //if(wd[fencode_hdv4(p,i,j,hdnul+hand)]>0.02 ||wd[fencode_hdv4(p,i,j,hdnul+hand)] < -0.02)
     //                                                     wd[fencode_hdv4(p,i,j,hdnul+hand)]=0.0;

   }
}
 __syncthreads();



 
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_hdv4(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}





int cuhyperdifvisc4(struct params **p, real **w, real **wnew, struct params **d_p, real **d_w, real **d_wnew,  real **d_wmod, real **d_dwn1, real **d_wd, int order, real **d_wtemp, real **d_wtemp1, real **d_wtemp2, int field, int dim,int hand)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
     hyperdifvisc4_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd, order, *d_wtemp,*d_wtemp1,*d_wtemp2, field, dim,hand);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_hdv4,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_hdv4,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_hdv4,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_hdv4, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}







