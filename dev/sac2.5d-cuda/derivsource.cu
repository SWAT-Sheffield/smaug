#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_ds.cuh"


__device__ __host__
real sourcerho (real *dw, real *wd, real *w, struct params *p,int ix, int iy) {

 // real src=0;
 // int field=rho;
  real src=0;
               src= -(p->chyp[rho])*(grad2_ds(w,p,ix,iy,rho,0)+grad2_ds(w,p,ix,iy,rho,1));
        #ifdef USE_SAC
           src= -(p->chyp[rho])*(grad2_ds(w,p,ix,iy,rhob,0)+grad2_ds(w,p,ix,iy,rhob,1));
         #endif
 
  return src;
}

__device__ __host__
real sourcemom (real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field, int direction) {

  real src=0;

  switch(direction)
  {
	case 0:
         //src=(w[fencode_ds(p,ix,iy,rho)]*(p->g[0]))-grad_ds(wd,p,ix,iy,pressuret,0)-(p->chyp)*(grad2_ds(w,p,ix,iy,mom1,0)+grad2_ds(w,p,ix,iy,mom1,1));
         src=(w[fencode_ds(p,ix,iy,rho)]*(p->g[0]))-(p->chyp[field])*(grad2_ds(w,p,ix,iy,mom1,0)+grad2_ds(w,p,ix,iy,mom1,1));
        // src=(w[fencode_ds(p,ix,iy,rho)]*(p->g[0]));
	break;
	case 1:
         //src=(w[fencode_ds(p,ix,iy,rho)]*(p->g[1]))-grad_ds(wd,p,ix,iy,pressuret,1)-(p->chyp)*(grad2_ds(w,p,ix,iy,mom2,1)+grad2_ds(w,p,ix,iy,mom2,0));
         src=(w[fencode_ds(p,ix,iy,rho)]*(p->g[1]))-(p->chyp[field])*(grad2_ds(w,p,ix,iy,mom2,1)+grad2_ds(w,p,ix,iy,mom2,0));
         //src=(w[fencode_ds(p,ix,iy,rho)]*(p->g[1]));
	break;
	/*case 2:
         //src=(w[fencode_ds(p,ix,iy,rho)]*(p->g[2]))-grad_ds(wd,p,ix,iy,pressuret,2);
         src=(w[fencode_ds(p,ix,iy,rho)]*(p->g[2]));

	break;*/
  }

  return(isnan(src)?0:src);


}

__device__ __host__
real sourceb (real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field, int direction) {

  real src=0;
  switch(direction)
  {
	case 0:
        ;// src=(p->eta)*grad_ds(wd,p,ix,iy,current3,1);
	break;
	case 1:
        ;// src= -(p->eta)*grad_ds(wd,p,ix,iy,current3,0);
	break;
	/*case 2:
         src= (p->eta)*(grad_ds(wd,p,ix,iy,current2,0)-grad_ds(wd,p,ix,iy,current1,1));
	break;*/
  }
   return(isnan(src)?0:src);
}

__device__ __host__
real sourceenergy (real *dw, real *wd, real *w, struct params *p,int ix, int iy) {

 // real src=0;
  real srcg,srcb,srcc;
  int field=energy;
  real ddcx,ddcy;
  real fi,fim1,fip2,fim2;
  fip2=0;
  fim2=0;
  srcc=0.0;
  srcb=0.0;

         #ifdef USE_SAC
  	   // srcg=w[fencode_ds(p,ix,iy,rho)]*(((p->g[0])*w[fencode_ds(p,ix,iy,mom1)]+(p->g[1])*w[fencode_ds(p,ix,iy,mom2)]+(p->g[2])*w[fencode_ds(p,ix,iy,mom3)]))/(w[fencode_ds(p,ix,iy,rho)]+w[fencode_ds(p,ix,iy,rhob)]);
           srcg=w[fencode_ds(p,ix,iy,rho)]*(((p->g[0])*w[fencode_ds(p,ix,iy,mom1)]+(p->g[1])*w[fencode_ds(p,ix,iy,mom2)]))/(w[fencode_ds(p,ix,iy,rho)]+w[fencode_ds(p,ix,iy,rhob)]);
         #else
     		// srcg=(p->g[0])*w[fencode_ds(p,ix,iy,mom1)]+(p->g[1])*w[fencode_ds(p,ix,iy,mom2)]+(p->g[2])*w[fencode_ds(p,ix,iy,mom3)];
                srcg=(p->g[0])*w[fencode_ds(p,ix,iy,mom1)]+(p->g[1])*w[fencode_ds(p,ix,iy,mom2)];
         #endif


     /*    #ifdef USE_SAC
	       fi=((w[fencode_ds(p,ix+1,iy,b2)]+w[fencode_ds(p,ix+1,iy,b2b)])*wd[fencode_ds(p,ix+1,iy,current3)]-(w[fencode_ds(p,ix+1,iy,b3)]+w[fencode_ds(p,ix+1,iy,b3b)])*wd[fencode_ds(p,ix+1,iy,current2)]);
	       fim1=((w[fencode_ds(p,ix-1,iy,b2)]+w[fencode_ds(p,ix-1,iy,b2b)])*wd[fencode_ds(p,ix-1,iy,current3)]-(w[fencode_ds(p,ix-1,iy,b3)]+w[fencode_ds(p,ix-1,iy,b3b)])*wd[fencode_ds(p,ix-1,iy,current2)]);
	    if(p->sodifon)
	     {
	       fip2=((w[fencode_ds(p,ix+2,iy,b2)]+w[fencode_ds(p,ix+2,iy,b2b)])*wd[fencode_ds(p,ix+2,iy,current3)]-(w[fencode_ds(p,ix+2,iy,b3)]+w[fencode_ds(p,ix+2,iy,b3b)])*wd[fencode_ds(p,ix+2,iy,current2)]);
	       fim2=((w[fencode_ds(p,ix-2,iy,b2)]+w[fencode_ds(p,ix-2,iy,b2b)])*wd[fencode_ds(p,ix-2,iy,current3)]-(w[fencode_ds(p,ix-2,iy,b3)]+w[fencode_ds(p,ix-2,iy,b3b)])*wd[fencode_ds(p,ix-2,iy,current2)]);
	      }
         #else
	       fi=(w[fencode_ds(p,ix+1,iy,b2)]*wd[fencode_ds(p,ix+1,iy,current3)]-w[fencode_ds(p,ix+1,iy,b3)]*wd[fencode_ds(p,ix+1,iy,current2)]);
	       fim1=(w[fencode_ds(p,ix-1,iy,b2)]*wd[fencode_ds(p,ix-1,iy,current3)]-w[fencode_ds(p,ix-1,iy,b3)]*wd[fencode_ds(p,ix-1,iy,current2)]);
	    if(p->sodifon)
	     {
	       fip2=(w[fencode_ds(p,ix+2,iy,b2)]*wd[fencode_ds(p,ix+2,iy,current3)]-w[fencode_ds(p,ix+2,iy,b3)]*wd[fencode_ds(p,ix+2,iy,current2)]);
	       fim2=(w[fencode_ds(p,ix-2,iy,b2)]*wd[fencode_ds(p,ix-2,iy,current3)]-w[fencode_ds(p,ix-2,iy,b3)]*wd[fencode_ds(p,ix-2,iy,current2)]);
	      }

         #endif

       ddcx=evalgrad_ds(fi,fim1,fip2,fim2,p,0);*/
      //ddcx=evalgrad_ds(fi,fim1,0,0,p,0);

     /*    #ifdef USE_SAC
	      fi=((w[fencode_ds(p,ix,iy+1,b3)]+w[fencode_ds(p,ix,iy+1,b3b)])*wd[fencode_ds(p,ix,iy+1,current1)]-(w[fencode_ds(p,ix,iy+1,b1)]+w[fencode_ds(p,ix,iy+1,b1b)])*wd[fencode_ds(p,ix,iy+1,current3)]);
	       fim1=((w[fencode_ds(p,ix,iy-1,b3)]+w[fencode_ds(p,ix,iy-1,b3b)])*wd[fencode_ds(p,ix,iy-1,current1)]-(w[fencode_ds(p,ix,iy-1,b1)]+w[fencode_ds(p,ix,iy-1,b1b)])*wd[fencode_ds(p,ix,iy-1,current3)]);
	    if(p->sodifon)
	     {
	       fip2=((w[fencode_ds(p,ix,iy+2,b3)]+w[fencode_ds(p,ix,iy+2,b3b)])*wd[fencode_ds(p,ix,iy+2,current1)]-(w[fencode_ds(p,ix,iy+2,b1)]+w[fencode_ds(p,ix,iy+2,b1b)])*wd[fencode_ds(p,ix,iy+2,current3)]);
	       fim2=((w[fencode_ds(p,ix,iy-2,b3)]+w[fencode_ds(p,ix,iy-2,b3b)])*wd[fencode_ds(p,ix,iy-2,current1)]-(w[fencode_ds(p,ix,iy-2,b1)]+w[fencode_ds(p,ix,iy-2,b1b)])*wd[fencode_ds(p,ix,iy-2,current3)]);
         #else
	      fi=(w[fencode_ds(p,ix,iy+1,b3)]*wd[fencode_ds(p,ix,iy+1,current1)]-w[fencode_ds(p,ix,iy+1,b1)]*wd[fencode_ds(p,ix,iy+1,current3)]);
	       fim1=(w[fencode_ds(p,ix,iy-1,b3)]*wd[fencode_ds(p,ix,iy-1,current1)]-w[fencode_ds(p,ix,iy-1,b1)]*wd[fencode_ds(p,ix,iy-1,current3)]);
	    if(p->sodifon)
	     {
	       fip2=(w[fencode_ds(p,ix,iy+2,b3)]*wd[fencode_ds(p,ix,iy+2,current1)]-w[fencode_ds(p,ix,iy+2,b1)]*wd[fencode_ds(p,ix,iy+2,current3)]);
	       fim2=(w[fencode_ds(p,ix,iy-2,b3)]*wd[fencode_ds(p,ix,iy-2,current1)]-w[fencode_ds(p,ix,iy-2,b1)]*wd[fencode_ds(p,ix,iy-2,current3)]);
         #endif
 
      }
      // ddcx=evalgrad_ds(fi,fim1,fip2,fim2,p,0);
      ddcy=evalgrad_ds(fi,fim1,fip2,fim2,p,1);

      srcb=(p->eta)*((isnan(ddcx)?0:ddcx)+(isnan(ddcy)?0:ddcy));*/

        #ifdef USE_SAC
         //calc gradv
	      fip2=0;
	      fim2=0;
	       fi=(w[fencode_ds(p,ix+1,iy,mom1)]/(w[fencode_ds(p,ix+1,iy,rho)]+w[fencode_ds(p,ix+1,iy,rhob)]));
	       fim1=(w[fencode_ds(p,ix-1,iy,mom1)]/(w[fencode_ds(p,ix-1,iy,rho)]+w[fencode_ds(p,ix-1,iy,rhob)]));
	    if(p->sodifon){
	       fip2=(w[fencode_ds(p,ix+2,iy,mom1)]/(w[fencode_ds(p,ix+2,iy,rho)]+w[fencode_ds(p,ix+2,iy,rhob)]));
	       fim2=(w[fencode_ds(p,ix-2,iy,mom1)]/(w[fencode_ds(p,ix-2,iy,rho)]+w[fencode_ds(p,ix-2,iy,rhob)]));
	      }
            ddcx=evalgrad_ds(fi,fim1,fip2,fim2,p,0);
	      fip2=0;
	      fim2=0;
	       fi=(w[fencode_ds(p,ix,iy+1,mom2)]/(w[fencode_ds(p,ix,iy+1,rho)]+w[fencode_ds(p,ix,iy+1,rhob)]));
	       fim1=(w[fencode_ds(p,ix,iy-1,mom2)]/(w[fencode_ds(p,ix,iy-1,rho)]+w[fencode_ds(p,ix,iy-1,rhob)]));
	    if(p->sodifon){
	       fip2=(w[fencode_ds(p,ix,iy+2,mom2)]/(w[fencode_ds(p,ix,iy+2,rho)]+w[fencode_ds(p,ix,iy+2,rhob)]));
	       fim2=(w[fencode_ds(p,ix,iy-2,mom2)]/(w[fencode_ds(p,ix,iy-2,rho)]+w[fencode_ds(p,ix,iy-2,rhob)]));
	      }
            ddcy=evalgrad_ds(fi,fim1,fip2,fim2,p,1);

                  srcc=-(ddcx+ddcy)*wd[fencode_ds(p,ix,iy,ptb)]+(ddcx*(w[fencode_ds(p,ix,iy,b1b)])+ddcy*(w[fencode_ds(p,ix,iy,b2b)]))*(w[fencode_ds(p,ix,iy,b1b)]+w[fencode_ds(p,ix,iy,b2b)]);

         #endif


 // src=srcg+srcb;
  return ( srcg+srcb+srcc);
}


__device__ __host__
int derivsourcerho (real *dw, real *wd, real *w, struct params *p,int ix, int iy,int order) {

  int status=0;
  int field=rho;
        dw[fencode_ds(p,ix,iy,field)]=+0.3*sourcerho(dw,wd,w,p,ix,iy);
     	//dw[fencode_ds(p,ix,iy,field)]=w[fencode_ds(p,ix,iy,field)]+10;
  return ( status);
}

__device__ __host__
int derivsourcemom (real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field, int direction, int order) {

  int status=0;
     	//dw[fencode_ds(p,ix,iy,field)]=w[fencode_ds(p,ix,iy,field)]+20+5*(2*direction+1);
        //factor 0.3333 providwed to get agreement with adiabatic hd shallowwater
        dw[fencode_ds(p,ix,iy,field)]=+1.0*sourcemom(dw,wd,w,p,ix,iy,field,direction);
        //dw[fencode_ds(p,ix,iy,field)]=-ddotcurrentmom(dw,wd,w,p,ix,iy,field,direction);

  return ( status);
}

__device__ __host__
int derivsourceb (real *dw, real *wd, real *w, struct params *p,int ix, int iy, int field, int direction, int order) {

  int status=0;
        dw[fencode_ds(p,ix,iy,field)]=sourceb(dw,wd,w,p,ix,iy,field,direction);

  return ( status);
}

__device__ __host__
int derivsourceenergy (real *dw, real *wd, real *w, struct params *p,int ix, int iy, int order) {

  int status=0;
  int field=energy;
        dw[fencode_ds(p,ix,iy,field)]=sourceenergy(dw,wd,w,p,ix,iy);

  return ( status);
}


//rho, mom1, mom2, mom3, energy, b1, b2, b3
__device__ __host__
void derivsource (real *dw, real *wd, real *w, struct params *p,int ix, int iy, int field, int order,int ordero) {

  //int status=0;
  switch(field)
  {
     case rho:
      derivsourcerho(dw,wd,w,p,ix,iy,order);
     break;
     case mom1:
      derivsourcemom(dw,wd,w,p,ix,iy,field,0,order);
     break;
     case mom2:
      derivsourcemom(dw,wd,w,p,ix,iy,field,1,order);
     break;
     /*case mom3:
      derivsourcemom(dw,wd,w,p,ix,iy,field,2,order);
     break;*/
     case energy:
       derivsourceenergy(dw,wd,w,p,ix,iy,order);
     break;
     case b1:
    derivsourceb(dw,wd,w,p,ix,iy,field,0,order);
     break;
     case b2:
      derivsourceb(dw,wd,w,p,ix,iy,field,1,order);
     break;
     /*case b3:
      derivsourceb(dw,wd,w,p,ix,iy,field,2,order);
     break;*/
  }
  //return ( status);
}


__global__ void derivsource_parallel(struct params *p, real *w, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real dt)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  

   j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);
  if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
	{		               
              for(int f=rho; f<NVAR; f++)
               {              
                     derivsource(dwn1,wd,wmod,p,i,j,f,order,ordero);
                 }
 	}
 __syncthreads();
   if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
	{		               
              for(int f=rho; f<NVAR; f++)
              {              
                              //                                                                                  - sign here same as vac maybe a +
                              wmod[fencode_ds(p,i,j,f)+ordero*NVAR*(p->n[0])*(p->n[1])]=wmod[fencode_ds(p,i,j,f)]+dt*dwn1[fencode_ds(p,i,j,f)]; 
                              //wmod[fencode_ds(p,i,j,f)+ordero*NVAR*(p->n[0])*(p->n[1])]=wmod[fencode_ds(p,i,j,f)]+0.1;
                              //dwn1[fencode_ds(p,i,j,f)]=0;
              }
	}
 __syncthreads();
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_ds(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}





int cuderivsource(struct params **p, real **w, struct params **d_p, real **d_w,  real **d_wmod, real **d_dwn1, real **d_wd, int order, int ordero, real dt)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
     derivsource_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wmod, *d_dwn1,  *d_wd, order,ordero,dt);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}







