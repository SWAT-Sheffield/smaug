#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_u.cuh"


__device__ __host__
int updatestate (struct params *p, struct state *s, real *w ,int i, int j, int field) {

  int status=0;
                      // atomicExch(&(p->cmax),(wd[fencode_pre(p,i,j,soundspeed)]));
                    switch(field)
                    {
                      case rho:
                    	s->rho=s->rho+(w[fencode_u(p,i,j,field)]);
		      break;
                      case mom1:
                    	s->m1=s->m1+(w[fencode_u(p,i,j,field)]);
		      break;
                      case mom2:
                    	s->m2=s->m2+(w[fencode_u(p,i,j,field)]);
		      break;
                      /*case mom3:
                    	s->m3=s->m3+(w[fencode_u(p,i,j,field)]);
		      break;*/
                      case energy:
                    	s->e=s->e+(w[fencode_u(p,i,j,field)]);
		      break;
                      case b1:
                    	s->b1=s->b1+(w[fencode_u(p,i,j,field)]);
		      break;
                      case b2:
                    	s->b2=s->b2+(w[fencode_u(p,i,j,field)]);
		      break;
                      /*case b3:
                    	s->b3=s->b3+(w[fencode_u(p,i,j,field)]);
		      break;*/
                    };
  return status;
}



__global__ void update_parallel(struct params *p, struct state *s, real *w, real *wmod)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
   int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  __shared__ int ntot;

  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  //real g=p->g;
  real *u,  *v,  *h;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;
  h=w+(p->n[0])*(p->n[1])*rho;
  u=w+(p->n[0])*(p->n[1])*mom1;
  v=w+(p->n[0])*(p->n[1])*mom2;


   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));

  //if(i>2 && j >2 && i<((p->n[0])-3) && j<((p->n[1])-3))






/*if (threadIdx.x == 0) 
{
 ntot=(p->n[0])*(p->n[1]);
 for(int f=rho; f<=NVAR; f++) 
 {
                    switch(f)
                    {
                      case rho:
                    	s->rho=0;
		      break;
                      case mom1:
                    	s->m1=0;
		      break;
                      case mom2:
                    	s->m2=0;
		      break;
                     // case mom3:
                    //	s->m3=0;
		     // break;
                      case energy:
                    	s->e=0;
		      break;
                      case b1:
                    	s->b1=0;
		      break;
                      case b2:
                    	s->b2=0;
		      break;
                      //case b3:
                    	//s->b3=0;
		      //break;
                    };

  }              
                 
}
__syncthreads();*/
 // if(i>1 && j>1 && i<((p->n[0])-2) && j<((p->n[1])-2))
 //if(i>0 && j>0 && i<((p->n[0])) && j<((p->n[1])))

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;

             for(int f=rho; f<=b2; f++)
             {   
if( i<((p->n[0])) && j<((p->n[1])))
	{
            
                  w[fencode_u(p,i,j,f)]=wmod[fencode_u(p,i,j,f)];
                  updatestate (p, s, w ,i, j, f);
              
            // u[i+j*ni]=un[i+j*ni];
           // v[i+j*ni]=vn[i+j*ni];
	   // h[i+j*ni]=hn[i+j*ni];
	}

__syncthreads();
}
}
 

/*if (threadIdx.x == 0) 
{
 for(int f=rho; f<NVAR; f++) 
 {
                    switch(f)
                    {
                      case rho:
                    	s->rho=(s->rho)/ntot;
		      break;
                      case mom1:
                    	s->m1=(s->m1)/ntot;
		      break;
                      case mom2:
                    	s->m2=(s->m2)/ntot;
		      break;
                     // case mom3:
                    //	s->m3=(s->m3)/ntot;
		     // break;
                      case energy:
                    	s->e=(s->e)/ntot;
		      break;
                      case b1:
                    	s->b1=(s->b1)/ntot;
		      break;
                      case b2:
                    	s->b2=(s->b2)/ntot;
		      break;
                    //  case b3:
                    //	s->b3=(s->b3)/ntot;
		    //  break;
                    };

  }              
                 
}
__syncthreads();*/






  
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_u(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}


int cuupdate(struct params **p, real **w, real **wnew, struct state **state,struct params **d_p, real **d_w, real **d_wmod, real **d_dwn1, real **d_wd, struct state **d_state)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
    // prop_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_u,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_u,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     //hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_u,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_state,*d_w,*d_wmod);
	    //printf("called update\n"); 
    hipDeviceSynchronize();
    hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);

    //hipMemcpy(*w, *d_wd, 6*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);

   hipMemcpy(*state, *d_state, sizeof(struct state), hipMemcpyDeviceToHost);

//hipMemcpy(*wnew, *d_wnew, 8*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_u, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}


int cufinish(struct params **p, real **w, real **wnew, struct params **d_p, real **d_w, real **d_wnew, real **d_wmod, real **d_dwn1, real **d_wd)
{
  

 //hipMemcpy(*w, *d_w, 8*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, 8*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_u, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  checkErrors_u("copy data from device");


  hipFree(*d_p);
//  hipFree(*d_state);

  hipFree(*d_w);
  hipFree(*d_wnew);
 // hipFree(*d_u);

  hipFree(*d_wmod);
  hipFree(*d_dwn1);
  hipFree(*d_wd);



}
