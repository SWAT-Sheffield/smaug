#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_hdv2.cuh"




__global__ void hyperdifvisc2_parallel(struct params *p, real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd, int order, real *wtemp, real *wtemp1, real *wtemp2, int field, int dim,int hand)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int is,js;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  real maxt=0,max3=0, max1=0;
  
   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));

int bfac1,bfac2,bfac3;
//int bfac1=(field==rho || field>mom2)+(field>rho && field<energy);
//int bfac2= (field==rho || field>mom2);
//int bfac3=(field>rho && field<energy);
int shift=order*NVAR*(p->n[0])*(p->n[1]);


   //tmp1  tmp_nuI
   //tmp2  d3r
    //tmp3 d1r
//tmp4    md3r
//tmp5    md1r
//tmp6    d3l
//tmp7    d1l
//tmp8    md3l
//tmp9    md1l




   //tmp1  tmp_nuI
 
//compute d3r and d1r
   //tmp2  d3r
    //tmp3 d1r

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
 
   if(i>1 && j>1 && i<((p->n[0])) && j<((p->n[1])))
   //if(i<((p->n[0])-1) && j<((p->n[1])-1))            
   { 
     if(hand==0)
     {
           wtemp1[fencode_hdv2(p,i,j,d3)]=fabs(3.0*(wtemp2[fencode_hdv2(p,i+(dim==0),j+(dim==1),tmpnui)] - wtemp2[fencode_hdv2(p,i,j,tmpnui)] ) - (wtemp2[fencode_hdv2(p,i+2*(dim==0),j+2*(dim==1),tmpnui)] - wtemp2[fencode_hdv2(p,i-(dim==0),j-(dim==1),tmpnui)]    ));
     }
     else
     {
          // wtemp1[fencode_hdv2(p,i,j,d3)]=fabs(3.0*(wtemp2[fencode_hdv2(p,i+(dim==0),j+(dim==1),tmpnui)] - wtemp2[fencode_hdv2(p,i,j,tmpnui)]) - (wtemp2[fencode_hdv2(p,i+2*(dim==0),j+2*(dim==1),tmpnui)] - wtemp2[fencode_hdv2(p,i-(dim==0),j-(dim==1),tmpnui)]    ));
           wtemp1[fencode_hdv2(p,i,j,d3)]=fabs(3.0*(wtemp2[fencode_hdv2(p,i,j,tmpnui)] - wtemp2[fencode_hdv2(p,i-(dim==0),j-(dim==1),tmpnui)]) - (wtemp2[fencode_hdv2(p,i+(dim==0),j+(dim==1),tmpnui)] - wtemp2[fencode_hdv2(p,i-2*(dim==0),j-2*(dim==1),tmpnui)]    ));
     }
   }
}
   __syncthreads();
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;

   if(i>0 && j>0 && i<=((p->n[0])) && j<=((p->n[1])))            
   { 
     if(hand==0)
     {

           wtemp1[fencode_hdv2(p,i,j,d1)]=fabs((wtemp2[fencode_hdv2(p,i+(dim==0),j+(dim==1),tmpnui)] - wtemp2[fencode_hdv2(p,i,j,tmpnui)] ));
     }
     else
     {
           wtemp1[fencode_hdv2(p,i,j,d1)]=fabs((wtemp2[fencode_hdv2(p,i,j,tmpnui)] - wtemp2[fencode_hdv2(p,i-(dim==0),j-(dim==1),tmpnui)] ));
     }
   }
}
   __syncthreads();



/*   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
          if( i<(ni) && j<(nj))
            {
                  bc_periodic1_hdv2(wtemp,p,i,j,tmp2);
                  bc_periodic1_hdv2(wtemp,p,i,j,tmp3);
             }

}
                __syncthreads();

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
          if( i<(ni) && j<(nj))
            {
                  bc_periodic2_hdv2(wtemp,p,i,j,tmp2);
                  bc_periodic2_hdv2(wtemp,p,i,j,tmp3);
             }
}
                __syncthreads();*/








 
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_hdv2(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}





int cuhyperdifvisc2(struct params **p, real **w, real **wnew, struct params **d_p, real **d_w, real **d_wnew,  real **d_wmod, real **d_dwn1, real **d_wd, int order, real **d_wtemp, real **d_wtemp1, real **d_wtemp2, int field, int dim,int hand)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
     hyperdifvisc2_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd, order, *d_wtemp,*d_wtemp1,*d_wtemp2, field, dim,hand);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_hdv2,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_hdv2,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_hdv2,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_hdv2, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}







