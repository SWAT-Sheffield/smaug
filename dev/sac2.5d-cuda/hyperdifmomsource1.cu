#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_hdm1.cuh"



__global__ void hyperdifmomsource1_parallel(struct params *p, real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim, int ii, int ii0, real dt)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1;
  real fip,fim1,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  real rdx;
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));

  rdx=(((p->dx[0])*(dim==0))+(p->dx[1])*(dim==1));

   
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;

  //init rhol and rhor
  if(i<((p->n[0])) && j<((p->n[1])))
  {
    for(int f=tmp1; f<=tmp8; f++)	
        wtemp[fencode_hdm1(p,i,j,f)]=0.0;

//dwn1[(NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i,j,energy)]=0.0;
//dwn1[(NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i,j,mom1+ii0)]=0.0;
dwn1[fencode_hdm1(p,i,j,energy)]=0.0;
dwn1[fencode_hdm1(p,i,j,mom1+ii0)]=0.0;
   }


}
 __syncthreads();

//tmp2  rhor
//tmp3  rhol
//tmp1  mom+field/rho

//tmp4  rhoc

   
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
//if(i<((p->n[0])) && j<((p->n[1])))
if( i<((p->n[0])) && j<((p->n[1])))
  {

#ifdef USE_SAC


     wtemp[fencode_hdm1(p,i,j,tmp4)]=wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i,j,mom1+field)]/(wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i,j,rho)]+wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i,j,rhob)]);

#else
 

     wtemp[fencode_hdm1(p,i,j,tmp4)]=wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i,j,mom1+field)]/wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i,j,rho)];


#endif
  /*  wtemp[fencode_hdm1(p,i,j,tmp2)]=wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i,j,mom1+field)]+wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i+(dim==0),j+(dim==1),mom1+field)];
    wtemp[fencode_hdm1(p,i,j,tmp3)]=wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i,j,mom1+field)]+wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i-(dim==0),j+(dim==1),mom1+field)];*/
   }

}
__syncthreads();




   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
//if(i<((p->n[0])) && j<((p->n[1])))
if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
  {

#ifdef USE_SAC
       wtemp[fencode_hdm1(p,i,j,tmp2)]=(wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i,j,rho)]+wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i,j,rhob)]+wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i-(dim==0),j-(dim==1),rho)]+wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i-(dim==0),j-(dim==1),rhob)])/2;
       wtemp[fencode_hdm1(p,i,j,tmp3)]=(wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i,j,rho)]+wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i,j,rhob)]+wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i+(dim==0),j+(dim==1),rho)]+wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i+(dim==0),j+(dim==1),rhob)])/2;



#else
       wtemp[fencode_hdm1(p,i,j,tmp2)]=(wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i,j,rho)]+wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i-(dim==0),j-(dim==1),rho)])/2;
       wtemp[fencode_hdm1(p,i,j,tmp3)]=(wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i,j,rho)]+wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i+(dim==0),j+(dim==1),rho)])/2;




#endif
  /*  wtemp[fencode_hdm1(p,i,j,tmp2)]=wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i,j,mom1+field)]+wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i+(dim==0),j+(dim==1),mom1+field)];
    wtemp[fencode_hdm1(p,i,j,tmp3)]=wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i,j,mom1+field)]+wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i-(dim==0),j+(dim==1),mom1+field)];*/
   }

}
__syncthreads();


   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
//if(i<((p->n[0])) && j<((p->n[1])))
if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
  {
     wtemp[fencode_hdm1(p,i,j,tmp5)]=(wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i,j,mom1+ii0)]+wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i-(dim==0),j-(dim==1),mom1+ii0)])/2;
     wtemp[fencode_hdm1(p,i,j,tmp6)]=(wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i,j,mom1+ii0)]+wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i+(dim==0),j+(dim==1),mom1+ii0)])/2;
   }

}
__syncthreads();



  
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_hdm1(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}





int cuhyperdifmomsource1(struct params **p, real **w, real **wnew, struct params **d_p, real **d_w, real **d_wnew,  real **d_wmod, real **d_dwn1, real **d_wd, int order, int ordero, real **d_wtemp, int field, int dim, int ii, int ii0, real dt)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
     hyperdifmomsource1_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,ii,ii0,dt);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}







