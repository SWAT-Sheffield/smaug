__device__ __host__
int dimproduct_MODID (struct params *dp) {

  int tot=1;
  for(int i=0;i<NDIM;i++)
    tot*=dp->n[i];
  return tot; 
}


__device__ __host__
int encode_MODID (struct params *dp,int ix, int iy) {

  //int kSizeX=(dp)->n[0];
  //int kSizeY=(dp)->n[1];
  
  return ( iy * ((dp)->n[0]) + ix);
}

__device__ __host__
int encode3_MODID (struct params *dp,int ix, int iy, int iz) {

  return (iz*((dp)->n[0])*((dp)->n[1])  + iy * ((dp)->n[0]) + ix);
}

__device__ __host__
int fencode_MODID (struct params *dp,int ix, int iy, int field) {

  //int kSizeX=(dp)->n[0];
  //int kSizeY=(dp)->n[1];
  
  return ( (iy * ((dp)->n[0]) + ix)+(field*((dp)->n[0])*((dp)->n[1])));
}

 
__device__ __host__
int fencode3_MODID (struct params *dp,int ix, int iy, int iz, int field) {

  //int kSizeX=(dp)->ni;
  //int kSizeY=(dp)->nj;
  
  return(  iz*((dp)->n[0])*((dp)->n[1])+ (iy * ((dp)->n[0]) + ix)+(field*((dp)->n[0])*((dp)->n[1])*((dp)->n[2]))  );
}


__device__ __host__
real evalgrad_MODID(real fi, real fim1, real fip2, real fim2,struct params *p,int dir)
{
 //real valgrad_MODID;

 if(dir == 0)
 {
     //valgrad=(2.0/(3.0*(p->dx[0])))*(fi-fim1)-(1.0/(12.0*(p->dx[0])))*(fip2-fim2);
   //return((1.0/(2.0*(p->dx[0])))*(fi-fim1));
   return(p->sodifon?((1.0/(2.0*(p->dx[0])))*(fi-fim1)):((1.0/(12.0*(p->dx[0])))*((8*fi-8*fim1+fim2-fip2))));
 }
 else if(dir == 1)
 {
    // valgrad=(2.0/(3.0*(p->dx[1])))*(fi-fim1)-(1.0/(12.0*(p->dx[1])))*(fip2-fim2);
     // return((2.0/(1.0*(p->dx[1])))*(fi-fim1));
   return(p->sodifon?((1.0/(2.0*(p->dx[1])))*(fi-fim1)):((1.0/(12.0*(p->dx[1])))*((8*fi-8*fim1+fim2-fip2))));
 }

 return -1;
}


__device__ __host__
real grad_MODID(real *wmod,struct params *p,int i,int j,int field,int dir)
{
 //real valgrad_MODID;
 real grad=0;

 
 

 switch(dir)
 {
   case 0:
 
// return(  ( (p->sodifon)?((8*wmod[fencode_MODID(p,i+1,j,field)]-8*wmod[fencode_MODID(p,i-1,j,field)]+wmod[fencode_MODID(p,i-2,j,field)]-wmod[fencode_MODID(p,i+2,j,field)])/6.0):wmod[fencode_MODID(p,i+1,j,field)]-wmod[fencode_MODID(p,i-1,j,field)])/(2.0*(p->dx[0]))    );
 if(i>1 && i<((p->n[0])-2) )
 grad=(  ( ((8*wmod[fencode_MODID(p,i+1,j,field)]-8*wmod[fencode_MODID(p,i-1,j,field)]+wmod[fencode_MODID(p,i-2,j,field)]-wmod[fencode_MODID(p,i+2,j,field)])/6.0))/(2.0*(p->dx[0]))    );
   break;

   case 1:

// return(  ( (p->sodifon)?((8*wmod[fencode_MODID(p,i,j+1,field)]-8*wmod[fencode_MODID(p,i,j-1,field)]+wmod[fencode_MODID(p,i,j-2,field)]-wmod[fencode_MODID(p,i,j+2,field)])/6.0):wmod[fencode_MODID(p,i,j+1,field)]-wmod[fencode_MODID(p,i,j-1,field)])/(2.0*(p->dx[1]))    ); 
 if( j >1 &&  j<((p->n[1])-2))
	grad=(  ( ((8*wmod[fencode_MODID(p,i,j+1,field)]-8*wmod[fencode_MODID(p,i,j-1,field)]+wmod[fencode_MODID(p,i,j-2,field)]-wmod[fencode_MODID(p,i,j+2,field)])/6.0))/(2.0*(p->dx[1]))    ); 

   break;
}



 return grad;
}

__device__ __host__
real gradd0_MODID(real *wmod,struct params *p,int i,int j,int field,int dir)
{

 return(  ( ((8*wmod[fencode_MODID(p,i+1,j,field)]-8*wmod[fencode_MODID(p,i-1,j,field)]+wmod[fencode_MODID(p,i-2,j,field)]-wmod[fencode_MODID(p,i+2,j,field)])/6.0))/(2.0*(p->dx[0]))    );

}

__device__ __host__
real gradd1_MODID(real *wmod,struct params *p,int i,int j,int field,int dir)
{
 
return(  ( ((8*wmod[fencode_MODID(p,i,j+1,field)]-8*wmod[fencode_MODID(p,i,j-1,field)]+wmod[fencode_MODID(p,i,j-2,field)]-wmod[fencode_MODID(p,i,j+2,field)])/6.0))/(2.0*(p->dx[1]))    ); 

}




__device__ __host__
real evalgrad1_MODID(real fi, real fim1, struct params *p,int dir)
{
 //real valgrad_MODID;

 if(dir == 0)
 {

   return(((1.0/(2*(p->dx[0])))*(fi-fim1)));
 }
 else if(dir == 1)
 {

   return(((1.0/(2*(p->dx[1])))*(fi-fim1)));
 }

 return -1;
}
__device__ __host__
real grad1l_MODID(real *wmod,struct params *p,int i,int j,int field,int dir)
{
 if((dir == 0) && i>0 && i<(p->n[0]))
 {
    return(  ( wmod[fencode_MODID(p,i,j,field)]-wmod[fencode_MODID(p,i-1,j,field)]) /((p->dx[0]))    );
 }
 else if((dir == 1)    && j>0 && j<(p->n[1]))
 {
    return(  ( wmod[fencode_MODID(p,i,j,field)]-wmod[fencode_MODID(p,i,j-1,field)])/((p->dx[1]))    );
  }
 return 0;

}

__device__ __host__
real grad1r_MODID(real *wmod,struct params *p,int i,int j,int field,int dir)
{
  if((dir == 0) && i>=0 && i<((p->n[0])-1))
 {
    return(  ( wmod[fencode_MODID(p,i+1,j,field)]-wmod[fencode_MODID(p,i,j,field)]) /((p->dx[0]))    );
 }
 else if((dir == 1)    && j>=0 && j<((p->n[1])-1))
 {
    return(  ( wmod[fencode_MODID(p,i,j+1,field)]-wmod[fencode_MODID(p,i,j,field)])/((p->dx[1]))    );
  }
 return 0;

}



__device__ __host__
real grad1_MODID(real *wmod,struct params *p,int i,int j,int field,int dir)
{
 //real valgrad_MODID;

  if((dir == 0) && i>0 && i<(p->n[0]))
 {
  
 return(  (wmod[fencode_MODID(p,i+1,j,field)]-wmod[fencode_MODID(p,i-1,j,field)])/(2.0*(p->dx[0]))    );
 }
 else if((dir == 1)    && j>0 && j<(p->n[1]))
 {

 return(  (wmod[fencode_MODID(p,i,j+1,field)]-wmod[fencode_MODID(p,i,j-1,field)])/(2.0*(p->dx[1]))    );
  }
 return 0;
}



__device__ __host__
real grad2_MODID(real *wmod,struct params *p,int i,int j,int field,int dir)
{
 //real valgrad_MODID;

  if(dir == 0)
 {
    // valgrad=(2.0/(3.0*(p->dx[0])))*(wmod[fencode(p,i,j,field)]-wmod[fencode(p,i-1,j,field)])-(1.0/(12.0*(p->dx[0])))*(wmod[fencode(p,i+2,j,field)]-wmod[fencode(p,i-2,j,field)]);
//return((1.0/(2.0*(p->dx[0])))*(wmod[fencode_MODID(p,i+1,j,field)]-wmod[fencode_MODID(p,i-1,j,field)]));
 return(  ( (p->sodifon)?((16*wmod[fencode_MODID(p,i+1,j,field)]+16*wmod[fencode_MODID(p,i-1,j,field)]-wmod[fencode_MODID(p,i-2,j,field)]-wmod[fencode_MODID(p,i+2,j,field)]-30*wmod[fencode_MODID(p,i,j,field)])/6.0):2.0*(wmod[fencode_MODID(p,i+1,j,field)]-2*wmod[fencode_MODID(p,i,j,field)]-wmod[fencode_MODID(p,i-1,j,field)]))/(2.0*(p->dx[0])*(p->dx[0]))    );
 }
 else if(dir == 1)
 {
    // valgrad=(2.0/(3.0*(p->dx[1])))*(wmod[fencode(p,i,j,field)]-wmod[fencode(p,i,j-1,field)])-(1.0/(12.0*(p->dx[1])))*(wmod[fencode(p,i,j+2,field)]-wmod[fencode(p,i,j-2,field)]);
// return((1.0/(2.0*(p->dx[1])))*(wmod[fencode_MODID(p,i,j+1,field)]-wmod[fencode_MODID(p,i,j-1,field)]));
 return(  ( (p->sodifon)?((16*wmod[fencode_MODID(p,i,j+1,field)]+16*wmod[fencode_MODID(p,i,j,field)]-wmod[fencode_MODID(p,i,j-2,field)]-wmod[fencode_MODID(p,i,j+2,field)]-30*wmod[fencode_MODID(p,i,j,field)])/6.0):2.0*(wmod[fencode_MODID(p,i,j+1,field)]-2.0*wmod[fencode_MODID(p,i,j+1,field)]-wmod[fencode_MODID(p,i,j-1,field)]))/(2.0*(p->dx[1])*(p->dx[1]))    );
  }
 return 0;
}


__device__ __host__
real grad3_MODID(real *wmod,struct params *p,int *ix,int field,int dir)
{
 //real valgrad;

 if(dir == 0)
 {

 return(  ( (p->sodifon)?((NVAR*wmod[fencode3_MODID(p,ix[0]+1,ix[1],ix[2],field)]-NVAR*wmod[fencode3_MODID(p,ix[0]-1,ix[1],ix[2],field)]+wmod[fencode3_MODID(p,ix[0]-2,ix[1],ix[2],field)]-wmod[fencode3_MODID(p,ix[0]+2,ix[1],ix[2],field)])/6.0):wmod[fencode3_MODID(p,ix[0]+1,ix[1],ix[2],field)]-wmod[fencode3_MODID(p,ix[0]-1,ix[1],ix[2],field)])/(2.0*(p->dx[0]))    );
 }
 else if(dir == 1)
 {

 return(  ( (p->sodifon)?((NVAR*wmod[fencode3_MODID(p,ix[0],ix[1]+1,ix[2],field)]-NVAR*wmod[fencode3_MODID(p,ix[0],ix[1]-1,ix[2],field)]+wmod[fencode3_MODID(p,ix[0],ix[1]-2,ix[2],field)]-wmod[fencode3_MODID(p,ix[0],ix[1]+2,ix[2],field)])/6.0):wmod[fencode3_MODID(p,ix[0],ix[1]+1,ix[2],field)]-wmod[fencode3_MODID(p,ix[0],ix[1]-1,ix[2],field)])/(2.0*(p->dx[1]))    );

 }
else if(dir == 2)
 {

 return(  ( (p->sodifon)?((NVAR*wmod[fencode3_MODID(p,ix[0],ix[1],ix[2]+1,field)]-NVAR*wmod[fencode3_MODID(p,ix[0],ix[1],ix[2]-1,field)]+wmod[fencode3_MODID(p,ix[0],ix[1],ix[2]-2,field)]-wmod[fencode3_MODID(p,ix[0],ix[1],ix[2]+2,field)])/6.0):wmod[fencode3_MODID(p,ix[0],ix[1],ix[2]+1,field)]-wmod[fencode3_MODID(p,ix[0],ix[1],ix[2]-1,field)])/(2.0*(p->dx[2]))    );

 }
 return -1;
}

__device__ __host__
void bc_cont_MODID(real *wt, struct params *p,int i, int j, int f) {

                if(i<2 && j<2)
                {
                  if(i==j)
                    //wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i+2,j,f)];
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,2,j,f)];
                  else                  
                    //wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,j+2,f)];
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,2,f)];                  
                }
                else if(i<2 && j>((p->n[1])-3))
                {
                  if(i==(j-(p->n[1])))                  
                    //wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i+2,j,f)]; 
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,2,j,f)];                     
                  else                  
                    //wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,(j-3),f)];
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,((p->n[1])-3),f)];                   
                }
                else if(i>((p->n[0])-3) && j<2)
                {
                  if((i-(p->n[0]))==j)                  
                    //wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,(i-3),j,f)];
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,((p->n[0])-3),j,f)];                  
                  else                  
                   // wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,j+2,f)];
                   wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,2,f)];                        
                }
                else if(i>((p->n[0])-3) && j>((p->n[1])-3))
                {
                  if(i==j)                  
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,(i-3),j,f)];                   
                  else                  
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,(j-3),f)];                  
                }                       
                else if(i==0 || i==1)                
                  //wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i+2,j,f)];   
                  wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,2,j,f)];              
                else if((i==((p->n[0])-1)) || (i==((p->n[0])-2)))                
                  //wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,(i-3),j,f)];    
                  wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,(p->n[0])-3,j,f)];                            
                else if(j==0 || j==1)                
                  //wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,j+2,f)]; 
                   wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,2,f)];                    
                else if((j==((p->n[1])-1)) || (j==((p->n[1])-2)))                
                  //wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,(j-3),f)];
                  wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,(p->n[1])-3,f)];
                




}

__device__ __host__
void bc_fixed_MODID(real *wt, struct params *p,int i, int j, int f, real val) {


                //(UPPER or LOWER)*NDIM*NVAR+dim*NVAR+varnum = picks out correct value for fixed BC
                //for array of values for fixed BC's

                if(i<2 && j<2)
                {
                  if(i==j)
                    wt[fencode_MODID(p,i,j,f)]=val;
                  else                  
                    wt[fencode_MODID(p,i,j,f)]=val;                  
                }
                else if(i<2 && j>((p->n[1])-3))
                {
                  if(i==(j-(p->n[1])))                  
                    wt[fencode_MODID(p,i,j,f)]=val;                  
                  else                  
                    wt[fencode_MODID(p,i,j,f)]=val;                  
                }
                else if(i>((p->n[0])-3) && j<2)
                {
                  if((i-(p->n[0]))==j)                  
                    wt[fencode_MODID(p,i,j,f)]=val;                  
                  else                  
                    wt[fencode_MODID(p,i,j,f)]=val;                  
                }
                else if(i>((p->n[0])-3) && j>((p->n[1])-3))
                {
                  if(i==j)                  
                    wt[fencode_MODID(p,i,j,f)]=val;                   
                  else                  
                    wt[fencode_MODID(p,i,j,f)]=val;                  
                }                       
                else if(i==0 || i==1)                
                  wt[fencode_MODID(p,i,j,f)]=val;                
                else if((i==((p->n[0])-1)) || (i==((p->n[0])-2)))                
                  wt[fencode_MODID(p,i,j,f)]=val;                
                else if(j==0 || j==1)                
                  wt[fencode_MODID(p,i,j,f)]=val;                
                else if((j==((p->n[1])-1)) || (j==((p->n[1])-2)))                
                  wt[fencode_MODID(p,i,j,f)]=val;
                




}

__device__ __host__
void bc_periodic_MODID(real *wt, struct params *p,int i, int j, int f) {

                if(i==0 || i==1)                
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,(p->n[0])-3+i,j,f)];
                    //wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,(p->n[0])-3,j,f)];                
                else if((i==((p->n[0])-1)) || (i==((p->n[0])-2)))                
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,4-(p->n[0])+i,j,f)];
                    //wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,2,j,f)];                
                else if(j==0 || j==1)                
                  wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,(p->n[1])-3+j,f)];
                  //wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,(p->n[1])-3,f)];                                
               else if((j==((p->n[1])-1)) || (j==((p->n[1])-2)))                
                  wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,4-(p->n[1])+j,f)];
                 //wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,2,f)];


               if(i<2 && j<2)
                {
                  if(i==j)
                    //wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,(p->n[0])-3+i,j,f)];
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,(p->n[1])-3+j,f)];
                  else                  
                    //wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,(p->n[1])-3+j,f)];
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,(p->n[0])-3+i,j,f)];                                    
                }
                else if(i<2 && j>((p->n[1])-3))
                {
                  if(i==(j-(p->n[1])))                  
                    //wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,(p->n[0])-3+i,4-(p->n[1])+j,f)];
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,(p->n[0])-3+i,j,f)];                                     
                  else                  
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,4-(p->n[1])+j,f)];                                     
                }
                else if(i>((p->n[0])-3) && j<2)
                {
                  if((i-(p->n[0]))==j)                  
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,4-(p->n[0])+i,j,f)];                                    
                  else                  
                   wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,(p->n[1])-3+j,f)];                                    
                }
                else if(i>((p->n[0])-3) && j>((p->n[1])-3))
                {
                  if(i==j)                  
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,4-(p->n[1])+j,f)];                                    
                  else                  
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,4-(p->n[0])+i,j,f)];                                    
                }                       
                 
                




}

__device__ __host__
void bc_periodic1_test_MODID(real *wt, struct params *p,int i, int j, int f) {

                if(i==0 || i==1 )                
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i+2,j,f)];
                //else if((i==((p->n[0])-1)) || (i==((p->n[0])-2)) || (i==((p->n[0])-3)))
                else if((i==((p->n[0])-1)) || (i==((p->n[0])-2)))                
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i-2,j,f)];
                else if(j==0 || j==1 )                
                  wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,j+2,f)];
                //else if((j==((p->n[1])-1)) || (j==((p->n[1])-2)) || (j==((p->n[1])-3)))
                else if((j==((p->n[1])-1)) || (j==((p->n[1])-2)) )                 
                  wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,j-2,f)];

 


}

__device__ __host__
void bc_periodic2_test_MODID(real *wt, struct params *p,int i, int j, int f) {


               if(i<2 && j<2)
                {
                  if(i==j)
                    //wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,(p->n[0])-3+i,j,f)];
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,j+2,f)];
                  else                  
                    //wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,(p->n[1])-3+j,f)];
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i+2,j,f)];                                    
                }
                else if(i<2 && j>((p->n[1])-3))
                {
                  if(i==(j-(p->n[1])))                  
                    //wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,(p->n[0])-3+i,4-(p->n[1])+j,f)];
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i+2,j,f)];                                     
                  else                  
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,j-2,f)];                                     
                }
                else if(i>((p->n[0])-3) && j<2)
                {
                  if((i-(p->n[0]))==j)                  
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i-2,j,f)];                                    
                  else                  
                   wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,j+2,f)];                                    
                }
                else if(i>((p->n[0])-3) && j>((p->n[1])-3))
                {
                  if(i==j)                  
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,j-2,f)];                                    
                  else                  
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i-2,j,f)];                                    
                }                       
                 
                




}

//bc's are not applied to ghost cells?
__device__ __host__
void bc_periodic1a_MODID(real *wt, struct params *p,int i, int j, int f) {

                if(i==2 || i==3 )                
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,(p->n[0])-4+i,j,f)];
                //else if((i==((p->n[0])-1)) || (i==((p->n[0])-2)) || (i==((p->n[0])-3)))
                else if((i==((p->n[0])-3)) || (i==((p->n[0])-4)))                
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,4-(p->n[0])+i,j,f)];
                else if(j==2 || j==3 )                
                  wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,(p->n[1])-4+j,f)];
                //else if((j==((p->n[1])-1)) || (j==((p->n[1])-2)) || (j==((p->n[1])-3)))
                else if((j==((p->n[1])-3)) || (j==((p->n[1])-4)) )                 
                  wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,4-(p->n[1])+j,f)];



}

//periodic bc's labelled ori below
//are the original ones I used
__device__ __host__
void bc_periodic1_MODID(real *wt, struct params *p,int i, int j, int f) {

                if(i==0 || i==1 )                
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,(p->n[0])-4+i,j,f)];
                //else if((i==((p->n[0])-1)) || (i==((p->n[0])-2)) || (i==((p->n[0])-3)))
                else if((i==((p->n[0])-1)) || (i==((p->n[0])-2)))                
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,4-(p->n[0])+i,j,f)];
                else if(j==0 || j==1 )                
                  wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,(p->n[1])-4+j,f)];
                //else if((j==((p->n[1])-1)) || (j==((p->n[1])-2)) || (j==((p->n[1])-3)))
                else if((j==((p->n[1])-1)) || (j==((p->n[1])-2)) )                 
                  wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,4-(p->n[1])+j,f)];

 


}


__device__ __host__
void bc_periodic2_MODID(real *wt, struct params *p,int i, int j, int f) {


               if(i<2 && j<2)
                {
                  if(i==j)
                    //wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,(p->n[0])-3+i,j,f)];
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,(p->n[1])-4+j,f)];
                  else                  
                    //wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,(p->n[1])-3+j,f)];
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,(p->n[0])-4+i,j,f)];                                    
                }
                else if(i<2 && j>((p->n[1])-3))
                {
                  if(i==(j-(p->n[1])))                  
                    //wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,(p->n[0])-3+i,4-(p->n[1])+j,f)];
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,(p->n[0])-4+i,j,f)];                                     
                  else                  
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,4-(p->n[1])+j,f)];                                     
                }
                else if(i>((p->n[0])-3) && j<2)
                {
                  if((i-(p->n[0]))==j)                  
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,4-(p->n[0])+i,j,f)];                                    
                  else                  
                   wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,(p->n[1])-4+j,f)];                                    
                }
                else if(i>((p->n[0])-3) && j>((p->n[1])-3))
                {
                  if(i==j)                  
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,4-(p->n[1])+j,f)];                                    
                  else                  
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,4-(p->n[0])+i,j,f)];                                    
                }                       
                 
                




}

//periodic bc's labelled ori below
//are the original ones I used
__device__ __host__
void bc_periodic1_original_MODID(real *wt, struct params *p,int i, int j, int f) {

                if(i==0 || i==1 )                
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,(p->n[0])-4+i,j,f)];
                //else if((i==((p->n[0])-1)) || (i==((p->n[0])-2)) || (i==((p->n[0])-3)))
                else if((i==((p->n[0])-1)) || (i==((p->n[0])-2)))                
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,4-(p->n[0])+i,j,f)];
                else if(j==0 || j==1 )                
                  wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,(p->n[1])-4+j,f)];
                //else if((j==((p->n[1])-1)) || (j==((p->n[1])-2)) || (j==((p->n[1])-3)))
                else if((j==((p->n[1])-1)) || (j==((p->n[1])-2)) )                 
                  wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,4-(p->n[1])+j,f)];

 


}

__device__ __host__
void bc_periodic2_original_MODID(real *wt, struct params *p,int i, int j, int f) {


               if(i<2 && j<2)
                {
                  if(i==j)
                    //wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,(p->n[0])-3+i,j,f)];
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,(p->n[1])-4+j,f)];
                  else                  
                    //wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,(p->n[1])-3+j,f)];
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,(p->n[0])-4+i,j,f)];                                    
                }
                else if(i<2 && j>((p->n[1])-3))
                {
                  if(i==(j-(p->n[1])))                  
                    //wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,(p->n[0])-3+i,4-(p->n[1])+j,f)];
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,(p->n[0])-4+i,j,f)];                                     
                  else                  
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,4-(p->n[1])+j,f)];                                     
                }
                else if(i>((p->n[0])-3) && j<2)
                {
                  if((i-(p->n[0]))==j)                  
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,4-(p->n[0])+i,j,f)];                                    
                  else                  
                   wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,(p->n[1])-4+j,f)];                                    
                }
                else if(i>((p->n[0])-3) && j>((p->n[1])-3))
                {
                  if(i==j)                  
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,4-(p->n[1])+j,f)];                                    
                  else                  
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,4-(p->n[0])+i,j,f)];                                    
                }                       
                 
                




}


