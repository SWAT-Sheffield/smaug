#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////



__device__ __host__
int encode_dc (struct params *dp,int ix, int iy) {

  //int kSizeX=(dp)->ni;
  //int kSizeY=(dp)->nj;
  
  return ( iy * ((dp)->ni) + ix);
}

__device__ __host__
int fencode_dc (struct params *dp,int ix, int iy, int field) {

  //int kSizeX=(dp)->ni;
  //int kSizeY=(dp)->nj;
  
  return ( (iy * ((dp)->ni) + ix)+(field*((dp)->ni)*((dp)->nj)));
}

__device__ __host__
real evalgrad_dc(real fi, real fim1, real fip2, real fim2,struct params *p,int dir)
{
 //real valgrad_dc;

 if(dir == 0)
 {
     //valgrad=(2.0/(3.0*(p->dx)))*(fi-fim1)-(1.0/(12.0*(p->dx)))*(fip2-fim2);
   //return((1.0/(2.0*(p->dx)))*(fi-fim1));
   return(p->sodifon?((1.0/(2.0*(p->dx)))*(fi-fim1)):((1.0/(12.0*(p->dx)))*((8*fi-8*fim1+fim2-fip2))));
 }
 else if(dir == 1)
 {
    // valgrad=(2.0/(3.0*(p->dy)))*(fi-fim1)-(1.0/(12.0*(p->dy)))*(fip2-fim2);
     // return((2.0/(1.0*(p->dy)))*(fi-fim1));
   return(p->sodifon?((1.0/(2.0*(p->dy)))*(fi-fim1)):((1.0/(12.0*(p->dy)))*((8*fi-8*fim1+fim2-fip2))));
 }

 return -1;
}


__device__ __host__
real grad_dc(real *wmod,struct params *p,int i,int j,int field,int dir)
{
 //real valgrad_dc;

 if(dir == 0)
 {
    // valgrad=(2.0/(3.0*(p->dx)))*(wmod[fencode(p,i,j,field)]-wmod[fencode(p,i-1,j,field)])-(1.0/(12.0*(p->dx)))*(wmod[fencode(p,i+2,j,field)]-wmod[fencode(p,i-2,j,field)]);
//return((1.0/(2.0*(p->dx)))*(wmod[fencode_dc(p,i+1,j,field)]-wmod[fencode_dc(p,i-1,j,field)]));
 return(  ( (p->sodifon)?((8*wmod[fencode_dc(p,i+1,j,field)]-8*wmod[fencode_dc(p,i-1,j,field)]+wmod[fencode_dc(p,i-1,j,field)]-wmod[fencode_dc(p,i+1,j,field)])/6.0):wmod[fencode_dc(p,i+1,j,field)]-wmod[fencode_dc(p,i-1,j,field)])/(2.0*(p->dx))    );
 }
 else if(dir == 1)
 {
    // valgrad=(2.0/(3.0*(p->dy)))*(wmod[fencode(p,i,j,field)]-wmod[fencode(p,i,j-1,field)])-(1.0/(12.0*(p->dy)))*(wmod[fencode(p,i,j+2,field)]-wmod[fencode(p,i,j-2,field)]);
// return((1.0/(2.0*(p->dy)))*(wmod[fencode_dc(p,i,j+1,field)]-wmod[fencode_dc(p,i,j-1,field)]));
 return(  ( (p->sodifon)?((8*wmod[fencode_dc(p,i,j+1,field)]-8*wmod[fencode_dc(p,i,j-1,field)]+wmod[fencode_dc(p,i,j-1,field)]-wmod[fencode_dc(p,i,j+1,field)])/6.0):wmod[fencode_dc(p,i,j+1,field)]-wmod[fencode_dc(p,i,j-1,field)])/(2.0*(p->dy))    );

 }

 return 0;
}

__device__ __host__
real ddotcurrentrho (real *dw, real *wd, real *w, struct params *p,int ix, int iy) {

  real ddc=0;
//  int field=rho;

      ddc= grad_dc(w,p,ix,iy,mom1,0)+grad_dc(w,p,ix,iy,mom2,1);
  return ( isnan(ddc)?0:ddc);
}

__device__ __host__
real ddotcurrentmom (real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field, int direction) {

  real ddc=0;
  real fi, fim1;
  //real  fip2=0, fim2=0;
  real ddc1,ddc2;
  real ddcx,ddcy;
   //     ddc= grad_dc(w,p,ix,iy,mom1,0)+grad_dc(w,p,ix,iy,mom2,1);
//evalgrad_dc(real fi, real fim1, real fip2, real fim2,struct params *p,int dir)
  //fi=w(fencode_dc(p,ix,iy,rho))
  //calculate momentum current

//w[fencode_dc(p,ix,iy,rho)])=1;
//w[fencode_dc(p,ix-1,iy,rho)])=1;
//w[fencode_dc(p,ix+2,iy,rho)])=1;
//w[fencode_dc(p,ix-2,iy,rho)])=1;
//w[fencode_dc(p,ix,iy,rho)])=1;
//w[fencode_dc(p,ix,iy-1,rho)])=1;
//w[fencode_dc(p,ix,iy+2,rho)])=1;
//w[fencode_dc(p,ix,iy-2,rho)])=1;

  switch(direction)
  {
    case 0:
       fi=(w[fencode_dc(p,ix+1,iy,mom1)]/w[fencode_dc(p,ix+1,iy,rho)])*w[fencode_dc(p,ix+1,iy,mom1)];
       fim1=(w[fencode_dc(p,ix-1,iy,mom1)]/w[fencode_dc(p,ix-1,iy,rho)])*w[fencode_dc(p,ix-1,iy,mom1)];
    //   fip2=(w[fencode_dc(p,ix+2,iy,mom1)]/w[fencode_dc(p,ix+2,iy,rho)])*w[fencode_dc(p,ix+2,iy,mom1)];
     //  fim2=(w[fencode_dc(p,ix-2,iy,mom1)]/w[fencode_dc(p,ix-2,iy,rho)])*w[fencode_dc(p,ix-2,iy,mom1)];
      // ddcx=evalgrad_dc(fi,fim1,fip2,fim2,p,0);
      ddcx=evalgrad_dc(fi,fim1,0,0,p,0);
       //ddcx=fi-fim1;
       fi=(w[fencode_dc(p,ix,iy+1,mom1)]/w[fencode_dc(p,ix,iy+1,rho)])*w[fencode_dc(p,ix,iy+1,mom2)];
       fim1=(w[fencode_dc(p,ix,iy-1,mom1)]/w[fencode_dc(p,ix,iy-1,rho)])*w[fencode_dc(p,ix,iy-1,mom2)];
      // fip2=(w[fencode_dc(p,ix,iy+2,mom1)]/w[fencode_dc(p,ix,iy+2,rho)])*w[fencode_dc(p,ix,iy+2,mom2)];
      // fim2=(w[fencode_dc(p,ix,iy-2,mom1)]/w[fencode_dc(p,ix,iy-2,rho)])*w[fencode_dc(p,ix,iy-2,mom2)];
       //ddcy=fi;
       ddcy=evalgrad_dc(fi,fim1,0,0,p,1);
       //ddcy=evalgrad_dc(0,0,fip2,fim2,p,1);
    break;
    case 1:
       fi=(w[fencode_dc(p,ix+1,iy,mom2)]/w[fencode_dc(p,ix+1,iy,rho)])*w[fencode_dc(p,ix+1,iy,mom1)];
       fim1=(w[fencode_dc(p,ix-1,iy,mom2)]/w[fencode_dc(p,ix-1,iy,rho)])*w[fencode_dc(p,ix-1,iy,mom1)];
      // fip2=(w[fencode_dc(p,ix+2,iy,mom2)]/w[fencode_dc(p,ix+2,iy,rho)])*w[fencode_dc(p,ix+2,iy,mom1)];
      // fim2=(w[fencode_dc(p,ix-2,iy,mom2)]/w[fencode_dc(p,ix-2,iy,rho)])*w[fencode_dc(p,ix-2,iy,mom1)];
       ddcx=evalgrad_dc(fi,fim1,0,0,p,0);
       fi=(w[fencode_dc(p,ix,iy+1,mom2)]/w[fencode_dc(p,ix,iy+1,rho)])*w[fencode_dc(p,ix,iy+1,mom2)];
       fim1=(w[fencode_dc(p,ix,iy-1,mom2)]/w[fencode_dc(p,ix,iy-1,rho)])*w[fencode_dc(p,ix,iy-1,mom2)];
      // fip2=(w[fencode_dc(p,ix,iy+2,mom2)]/w[fencode_dc(p,ix,iy+2,rho)])*w[fencode_dc(p,ix,iy+2,mom2)];
      // fim2=(w[fencode_dc(p,ix,iy-2,mom2)]/w[fencode_dc(p,ix,iy-2,rho)])*w[fencode_dc(p,ix,iy-2,mom2)];
       ddcy=evalgrad_dc(fi,fim1,0,0,p,1);
    break;
    case 2:
       fi=(w[fencode_dc(p,ix+1,iy,mom3)]/w[fencode_dc(p,ix+1,iy,rho)])*w[fencode_dc(p,ix+1,iy,mom1)];
       fim1=(w[fencode_dc(p,ix-1,iy,mom3)]/w[fencode_dc(p,ix-1,iy,rho)])*w[fencode_dc(p,ix-1,iy,mom1)];
      // fip2=(w[fencode_dc(p,ix+2,iy,mom3)]/w[fencode_dc(p,ix+2,iy,rho)])*w[fencode_dc(p,ix+2,iy,mom1)];
     //  fim2=(w[fencode_dc(p,ix-2,iy,mom3)]/w[fencode_dc(p,ix-2,iy,rho)])*w[fencode_dc(p,ix-2,iy,mom1)];
       ddcx=evalgrad_dc(fi,fim1,0,0,p,0);
       fi=(w[fencode_dc(p,ix,iy+1,mom3)]/w[fencode_dc(p,ix,iy+1,rho)])*w[fencode_dc(p,ix,iy+1,mom2)];
       fim1=(w[fencode_dc(p,ix,iy-1,mom3)]/w[fencode_dc(p,ix,iy-1,rho)])*w[fencode_dc(p,ix,iy-1,mom2)];
     //  fip2=(w[fencode_dc(p,ix,iy+2,mom3)]/w[fencode_dc(p,ix,iy+2,rho)])*w[fencode_dc(p,ix,iy+2,mom2)];
     //  fim2=(w[fencode_dc(p,ix,iy-2,mom3)]/w[fencode_dc(p,ix,iy-2,rho)])*w[fencode_dc(p,ix,iy-2,mom2)];
       ddcy=evalgrad_dc(fi,fim1,0,0,p,1);
    break;
  }
  
  ddc1=(isnan(ddcx)?0:ddcx)+(isnan(ddcy)?0:ddcy);
//fip2=0, fim2=0;
  //calculate bfield current
  switch(direction)
  {
    case 0:
       fi=w[fencode_dc(p,ix+1,iy,b1)]*w[fencode_dc(p,ix+1,iy,b1)];
       fim1=w[fencode_dc(p,ix-1,iy,b1)]*w[fencode_dc(p,ix-1,iy,b1)];
     // fip2=w[fencode_dc(p,ix+2,iy,b1)]*w[fencode_dc(p,ix+2,iy,b1)];
     //  fim2=w[fencode_dc(p,ix-2,iy,b1)]*w[fencode_dc(p,ix-2,iy,b1)];
       ddcx=evalgrad_dc(fi,fim1,0,0,p,0);
       fi=w[fencode_dc(p,ix,iy+1,b1)]*w[fencode_dc(p,ix,iy+1,b2)];
       fim1=w[fencode_dc(p,ix,iy-1,b1)]*w[fencode_dc(p,ix,iy-1,b2)];
     //  fip2=w[fencode_dc(p,ix,iy+2,b1)]*w[fencode_dc(p,ix,iy+2,b2)];
      // fim2=w[fencode_dc(p,ix,iy-2,b1)]*w[fencode_dc(p,ix,iy-2,b2)];
       ddcy=evalgrad_dc(fi,fim1,0,0,p,1);
    break;
    case 1:
       fi=w[fencode_dc(p,ix+1,iy,b2)]*w[fencode_dc(p,ix+1,iy,b1)];
       fim1=w[fencode_dc(p,ix-1,iy,b2)]*w[fencode_dc(p,ix-1,iy,b1)];
     //  fip2=w[fencode_dc(p,ix+2,iy,b2)]*w[fencode_dc(p,ix+2,iy,b1)];
      // fim2=w[fencode_dc(p,ix-2,iy,b2)]*w[fencode_dc(p,ix-2,iy,b1)];
       ddcx=evalgrad_dc(fi,fim1,0,0,p,0);
       fi=w[fencode_dc(p,ix,iy+1,b2)]*w[fencode_dc(p,ix,iy+1,b2)];
       fim1=w[fencode_dc(p,ix,iy-1,b2)]*w[fencode_dc(p,ix,iy-1,b2)];
      // fip2=w[fencode_dc(p,ix,iy+2,b2)]*w[fencode_dc(p,ix,iy+2,b2)];
      // fim2=w[fencode_dc(p,ix,iy-2,b2)]*w[fencode_dc(p,ix,iy-2,b2)];
       ddcy=evalgrad_dc(fi,fim1,0,0,p,1);
    break;
    case 2:
       fi=w[fencode_dc(p,ix+1,iy,b3)]*w[fencode_dc(p,ix+1,iy,b1)];
       fim1=w[fencode_dc(p,ix-1,iy,b3)]*w[fencode_dc(p,ix-1,iy,b1)];
      // fip2=w[fencode_dc(p,ix+2,iy,b3)]*w[fencode_dc(p,ix+2,iy,b1)];
      // fim2=w[fencode_dc(p,ix-2,iy,b3)]*w[fencode_dc(p,ix-2,iy,b1)];
       ddcx=evalgrad_dc(fi,fim1,0,0,p,0);
       fi=w[fencode_dc(p,ix,iy+1,b3)]*w[fencode_dc(p,ix,iy+1,b2)];
       fim1=w[fencode_dc(p,ix,iy-1,b3)]*w[fencode_dc(p,ix,iy-1,b2)];
      // fip2=w[fencode_dc(p,ix,iy+2,b3)]*w[fencode_dc(p,ix,iy+2,b2)];
     //  fim2=w[fencode_dc(p,ix,iy-2,b3)]*w[fencode_dc(p,ix,iy-2,b2)];
       ddcy=evalgrad_dc(fi,fim1,0,0,p,1);
    break;
  }
  //ddc2=ddcx+ddcy;
  ddc2=(isnan(ddcx)?0:ddcx)+(isnan(ddcy)?0:ddcy);

  //ddc=ddc1-ddc2;

  return ( ddc1-ddc2);
}

__device__ __host__
real ddotcurrentb (real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field, int direction) {

  //real ddc=0;

  real fi, fim1;// fip2=0, fim2=0;
  real ddc1,ddc2;
  real ddcx,ddcy;

  switch(direction)
  {
	case 0:
	       fi=w[fencode_dc(p,ix+1,iy,mom1)]*w[fencode_dc(p,ix+1,iy,b1)]/w[fencode_dc(p,ix+1,iy,rho)];
	       fim1=w[fencode_dc(p,ix-1,iy,mom1)]*w[fencode_dc(p,ix-1,iy,b1)]/w[fencode_dc(p,ix-1,iy,rho)];
	       //fip2=w[fencode_dc(p,ix+2,iy,mom1)]*w[fencode_dc(p,ix+2,iy,b1)]/w[fencode_dc(p,ix+2,iy,rho)];
	       //fim2=w[fencode_dc(p,ix-2,iy,mom1)]*w[fencode_dc(p,ix-2,iy,b1)]/w[fencode_dc(p,ix-2,iy,rho)];
	       ddcx=evalgrad_dc(fi,fim1,0,0,p,0);
	       fi=w[fencode_dc(p,ix,iy+1,mom1)]*w[fencode_dc(p,ix,iy+1,b2)]/w[fencode_dc(p,ix,iy+1,rho)];
	       fim1=w[fencode_dc(p,ix,iy-1,mom1)]*w[fencode_dc(p,ix,iy-1,b2)]/w[fencode_dc(p,ix,iy-1,rho)];
	       //fip2=w[fencode_dc(p,ix,iy+2,mom1)]*w[fencode_dc(p,ix,iy+2,b2)]/w[fencode_dc(p,ix,iy+2,rho)];
	       //fim2=w[fencode_dc(p,ix,iy-2,mom1)]*w[fencode_dc(p,ix,iy-2,b2)]/w[fencode_dc(p,ix,iy-2,rho)];
	       ddcy=evalgrad_dc(fi,fim1,0,0,p,1);
        break;
	case 1:
	       fi=w[fencode_dc(p,ix+1,iy,mom2)]*w[fencode_dc(p,ix+1,iy,b1)]/w[fencode_dc(p,ix+1,iy,rho)];
	       fim1=w[fencode_dc(p,ix-1,iy,mom2)]*w[fencode_dc(p,ix-1,iy,b1)]/w[fencode_dc(p,ix-1,iy,rho)];
	       //fip2=w[fencode_dc(p,ix+2,iy,mom2)]*w[fencode_dc(p,ix+2,iy,b1)]/w[fencode_dc(p,ix+2,iy,rho)];
	       //fim2=w[fencode_dc(p,ix-2,iy,mom2)]*w[fencode_dc(p,ix-2,iy,b1)]/w[fencode_dc(p,ix-2,iy,rho)];
	       ddcx=evalgrad_dc(fi,fim1,0,0,p,0);
	       fi=w[fencode_dc(p,ix,iy+1,mom2)]*w[fencode_dc(p,ix,iy+1,b2)]/w[fencode_dc(p,ix,iy+1,rho)];
	       fim1=w[fencode_dc(p,ix,iy-1,mom2)]*w[fencode_dc(p,ix,iy-1,b2)]/w[fencode_dc(p,ix,iy-1,rho)];
	       //fip2=w[fencode_dc(p,ix,iy+2,mom2)]*w[fencode_dc(p,ix,iy+2,b2)]/w[fencode_dc(p,ix,iy+2,rho)];
	       //fim2=w[fencode_dc(p,ix,iy-2,mom2)]*w[fencode_dc(p,ix,iy-2,b2)]/w[fencode_dc(p,ix,iy-2,rho)];
	       ddcy=evalgrad_dc(fi,fim1,0,0,p,1);
        break;
	case 2:
	       fi=w[fencode_dc(p,ix+1,iy,mom3)]*w[fencode_dc(p,ix+1,iy,b1)]/w[fencode_dc(p,ix+1,iy,rho)];
	       fim1=w[fencode_dc(p,ix-1,iy,mom3)]*w[fencode_dc(p,ix-1,iy,b1)]/w[fencode_dc(p,ix-1,iy,rho)];
	       //fip2=w[fencode_dc(p,ix+2,iy,mom3)]*w[fencode_dc(p,ix+2,iy,b1)]/w[fencode_dc(p,ix+2,iy,rho)];
	       //fim2=w[fencode_dc(p,ix-2,iy,mom3)]*w[fencode_dc(p,ix-2,iy,b1)]/w[fencode_dc(p,ix-2,iy,rho)];
	       ddcx=evalgrad_dc(fi,fim1,0,0,p,0);
	       fi=w[fencode_dc(p,ix,iy+1,mom3)]*w[fencode_dc(p,ix,iy+1,b2)]/w[fencode_dc(p,ix,iy+1,rho)];
	       fim1=w[fencode_dc(p,ix,iy-1,mom3)]*w[fencode_dc(p,ix,iy-1,b2)]/w[fencode_dc(p,ix,iy-1,rho)];
	       //fip2=w[fencode_dc(p,ix,iy+2,mom3)]*w[fencode_dc(p,ix,iy+2,b2)]/w[fencode_dc(p,ix,iy+2,rho)];
	       //fim2=w[fencode_dc(p,ix,iy-2,mom3)]*w[fencode_dc(p,ix,iy-2,b2)]/w[fencode_dc(p,ix,iy-2,rho)];
	       ddcy=evalgrad_dc(fi,fim1,0,0,p,1);

        break;
  }
  ddc1=(isnan(ddcx)?0:ddcx)+(isnan(ddcy)?0:ddcy);



  switch(direction)
  {
	case 0:
	       fi=w[fencode_dc(p,ix+1,iy,b1)]*w[fencode_dc(p,ix+1,iy,mom1)]/w[fencode_dc(p,ix+1,iy,rho)];
	       fim1=w[fencode_dc(p,ix-1,iy,b1)]*w[fencode_dc(p,ix-1,iy,mom1)]/w[fencode_dc(p,ix-1,iy,rho)];
	       //fip2=w[fencode_dc(p,ix+2,iy,b1)]*w[fencode_dc(p,ix+2,iy,mom1)]/w[fencode_dc(p,ix+2,iy,rho)];
	      // fim2=w[fencode_dc(p,ix-2,iy,b1)]*w[fencode_dc(p,ix-2,iy,mom1)]/w[fencode_dc(p,ix-2,iy,rho)];
	       ddcx=evalgrad_dc(fi,fim1,0,0,p,0);
	       fi=w[fencode_dc(p,ix,iy+1,b1)]*w[fencode_dc(p,ix,iy+1,mom2)]/w[fencode_dc(p,ix,iy+1,rho)];
	       fim1=w[fencode_dc(p,ix,iy-1,b1)]*w[fencode_dc(p,ix,iy-1,mom2)]/w[fencode_dc(p,ix,iy-1,rho)];
	       //fip2=w[fencode_dc(p,ix,iy+2,b1)]*w[fencode_dc(p,ix,iy+2,mom2)]/w[fencode_dc(p,ix,iy+2,rho)];
	       //fim2=w[fencode_dc(p,ix,iy-2,b1)]*w[fencode_dc(p,ix,iy-2,mom2)]/w[fencode_dc(p,ix,iy-2,rho)];
	       ddcy=evalgrad_dc(fi,fim1,0,0,p,1);
        break;
	case 1:
	       fi=w[fencode_dc(p,ix+1,iy,b2)]*w[fencode_dc(p,ix+1,iy,mom1)]/w[fencode_dc(p,ix+1,iy,rho)];
	       fim1=w[fencode_dc(p,ix-1,iy,b2)]*w[fencode_dc(p,ix-1,iy,mom1)]/w[fencode_dc(p,ix-1,iy,rho)];
	       //fip2=w[fencode_dc(p,ix+2,iy,b2)]*w[fencode_dc(p,ix+2,iy,mom1)]/w[fencode_dc(p,ix+2,iy,rho)];
	      // fim2=w[fencode_dc(p,ix-2,iy,b2)]*w[fencode_dc(p,ix-2,iy,mom1)]/w[fencode_dc(p,ix-2,iy,rho)];
	       ddcx=evalgrad_dc(fi,fim1,0,0,p,0);
	       fi=w[fencode_dc(p,ix,iy+1,b2)]*w[fencode_dc(p,ix,iy+1,mom2)]/w[fencode_dc(p,ix,iy+1,rho)];
	       fim1=w[fencode_dc(p,ix,iy-1,b2)]*w[fencode_dc(p,ix,iy-1,mom2)]/w[fencode_dc(p,ix,iy-1,rho)];
	      // fip2=w[fencode_dc(p,ix,iy+2,b2)]*w[fencode_dc(p,ix,iy+2,mom2)]/w[fencode_dc(p,ix,iy+2,rho)];
	      // fim2=w[fencode_dc(p,ix,iy-2,b2)]*w[fencode_dc(p,ix,iy-2,mom2)]/w[fencode_dc(p,ix,iy-2,rho)];
	       ddcy=evalgrad_dc(fi,fim1,0,0,p,1);
        break;
	case 2:
	       fi=w[fencode_dc(p,ix+1,iy,b3)]*w[fencode_dc(p,ix+1,iy,mom1)]/w[fencode_dc(p,ix+1,iy,rho)];
	       fim1=w[fencode_dc(p,ix-1,iy,b3)]*w[fencode_dc(p,ix-1,iy,mom1)]/w[fencode_dc(p,ix-1,iy,rho)];
	       //fip2=w[fencode_dc(p,ix+2,iy,b3)]*w[fencode_dc(p,ix+2,iy,mom1)]/w[fencode_dc(p,ix+2,iy,rho)];
	       //fim2=w[fencode_dc(p,ix-2,iy,b3)]*w[fencode_dc(p,ix-2,iy,mom1)]/w[fencode_dc(p,ix-2,iy,rho)];
	       ddcx=evalgrad_dc(fi,fim1,0,0,p,0);
	       fi=w[fencode_dc(p,ix,iy+1,b3)]*w[fencode_dc(p,ix,iy+1,mom2)]/w[fencode_dc(p,ix,iy+1,rho)];
	       fim1=w[fencode_dc(p,ix,iy-1,b3)]*w[fencode_dc(p,ix,iy-1,mom2)]/w[fencode_dc(p,ix,iy-1,rho)];
	       //fip2=w[fencode_dc(p,ix,iy+2,b3)]*w[fencode_dc(p,ix,iy+2,mom2)]/w[fencode_dc(p,ix,iy+2,rho)];
	       //fim2=w[fencode_dc(p,ix,iy-2,b3)]*w[fencode_dc(p,ix,iy-2,mom2)]/w[fencode_dc(p,ix,iy-2,rho)];
	       ddcy=evalgrad_dc(fi,fim1,0,0,p,1);
        break;
  }
  ddc2=(isnan(ddcx)?0:ddcx)+(isnan(ddcy)?0:ddcy);
  return(ddc1-ddc2);

}

__device__ __host__
real ddotcurrentenergy (real *dw, real *wd, real *w, struct params *p,int ix, int iy) {

 // real ddc=0;
  real dd1,dd2,dd3;
 
  real ddcx,ddcy;
  real fi, fim1,fip2=0, fim2=0;
  //real dpi, dpim1;//, dpip2=0, dpim2=0;


  //int field=energy;

  //fi=w[fencode_dc(p,ix+1,iy,energy)]*w[fencode_dc(p,ix+1,iy,mom1)]/w[fencode_dc(p,ix,iy,rho)];
  //fim1=w[fencode_dc(p,ix-1,iy,energy)]*w[fencode_dc(p,ix-1,iy,mom1)]/w[fencode_dc(p,ix-1,iy,rho)];
if(p->sodifon==1)
{
  fip2=w[fencode_dc(p,ix+2,iy,energy)]*w[fencode_dc(p,ix+2,iy,mom1)]/w[fencode_dc(p,ix+2,iy,rho)];
  fim2=w[fencode_dc(p,ix-2,iy,energy)]*w[fencode_dc(p,ix-2,iy,mom1)]/w[fencode_dc(p,ix-2,iy,rho)];
}
 // ddcx=evalgrad_dc(fi,fim1,0,0,p,0);
  ddcx=evalgrad_dc(w[fencode_dc(p,ix+(p->sodifon==0),iy,energy)]*w[fencode_dc(p,ix+(p->sodifon==0),iy,mom1)]/w[fencode_dc(p,ix+(p->sodifon==0),iy,rho)],w[fencode_dc(p,ix-(p->sodifon==0),iy,energy)]*w[fencode_dc(p,ix-(p->sodifon==0),iy,mom1)]/w[fencode_dc(p,ix-(p->sodifon==0),iy,rho)],fip2,fim2,p,0);

 // fi=w[fencode_dc(p,ix,iy+1,energy)]*w[fencode_dc(p,ix,iy+1,mom2)]/w[fencode_dc(p,ix,iy+1,rho)];
 // fim1=w[fencode_dc(p,ix,iy-1,energy)]*w[fencode_dc(p,ix,iy-1,mom2)]/w[fencode_dc(p,ix,iy-1,rho)];
if(p->sodifon==1)
{
  fip2=w[fencode_dc(p,ix,iy+2,energy)]*w[fencode_dc(p,ix,iy+2,mom2)]/w[fencode_dc(p,ix,iy+2,rho)];
  fim2=w[fencode_dc(p,ix,iy-2,energy)]*w[fencode_dc(p,ix,iy-2,mom2)]/w[fencode_dc(p,ix,iy-2,rho)];
}
  //ddcy=evalgrad_dc(fi,fim1,0,0,p,1);
  ddcy=evalgrad_dc(w[fencode_dc(p,ix,iy+(p->sodifon==0),energy)]*w[fencode_dc(p,ix,iy+(p->sodifon==0),mom2)]/w[fencode_dc(p,ix,iy+(p->sodifon==0),rho)],w[fencode_dc(p,ix,iy-(p->sodifon==0),energy)]*w[fencode_dc(p,ix,iy-(p->sodifon==0),mom2)]/w[fencode_dc(p,ix,iy-(p->sodifon==0),rho)],fip2,fim2,p,1);

  dd1=(isnan(ddcx)?0:ddcx)+(isnan(ddcy)?0:ddcy);


 // dpi=(w[fencode_dc(p,ix+1,iy,b1)]*w[fencode_dc(p,ix+1,iy,mom1)]+w[fencode_dc(p,ix+1,iy,b2)]*w[fencode_dc(p,ix+1,iy,mom2)]+w[fencode_dc(p,ix+1,iy,b3)]*w[fencode_dc(p,ix+1,iy,mom3)])/w[fencode_dc(p,ix+1,iy,rho)];
 // dpim1=(w[fencode_dc(p,ix-1,iy,b1)]*w[fencode_dc(p,ix-1,iy,mom1)]+w[fencode_dc(p,ix-1,iy,b2)]*w[fencode_dc(p,ix-1,iy,mom2)]+w[fencode_dc(p,ix-1,iy,b3)]*w[fencode_dc(p,ix-1,iy,mom3)])/w[fencode_dc(p,ix-1,iy,rho)];
  //dpip2=(w[fencode_dc(p,ix+2,iy,b1)]*w[fencode_dc(p,ix+2,iy,mom1)]+w[fencode_dc(p,ix+2,iy,b2)]*w[fencode_dc(p,ix+2,iy,mom2)]+w[fencode_dc(p,ix+2,iy,b3)]*w[fencode_dc(p,ix+2,iy,mom3)])/w[fencode_dc(p,ix+2,iy,rho)];
 // dpim2=(w[fencode_dc(p,ix-2,iy,b1)]*w[fencode_dc(p,ix-2,iy,mom1)]+w[fencode_dc(p,ix-2,iy,b2)]*w[fencode_dc(p,ix-2,iy,mom2)]+w[fencode_dc(p,ix-2,iy,b3)]*w[fencode_dc(p,ix-2,iy,mom3)])/w[fencode_dc(p,ix-2,iy,rho)];

 // fi=dpi*w[fencode_dc(p,ix+1,iy,b1)];
 // fim1=dpim1*w[fencode_dc(p,ix-1,iy,b1)];
  //fip2=dpip2*w[fencode_dc(p,ix+2,iy,b1)];
 // fim2=dpim2*w[fencode_dc(p,ix-2,iy,b1)];
 // ddcx=evalgrad_dc(fi,fim1,0,0,p,0);
 //  ddcx=evalgrad_dc(((w[fencode_dc(p,ix+1,iy,b1)]*w[fencode_dc(p,ix+1,iy,mom1)]+w[fencode_dc(p,ix+1,iy,b2)]*w[fencode_dc(p,ix+1,iy,mom2)]+w[fencode_dc(p,ix+1,iy,b3)]*w[fencode_dc(p,ix+1,iy,mom3)])/w[fencode_dc(p,ix+1,iy,rho)])*w[fencode_dc(p,ix+1,iy,b1)],((w[fencode_dc(p,ix-1,iy,b1)]*w[fencode_dc(p,ix-1,iy,mom1)]+w[fencode_dc(p,ix-1,iy,b2)]*w[fencode_dc(p,ix-1,iy,mom2)]+w[fencode_dc(p,ix-1,iy,b3)]*w[fencode_dc(p,ix-1,iy,mom3)])/w[fencode_dc(p,ix-1,iy,rho)])*w[fencode_dc(p,ix-1,iy,b1)],0,0,p,0);

if(p->sodifon==1)
{
  fip2=wd[fencode_dc(p,ix+2,iy,bdotv)]*w[fencode_dc(p,ix+2,iy,b1)];
  fim2=wd[fencode_dc(p,ix-2,iy,bdotv)]*w[fencode_dc(p,ix-2,iy,b1)];
}

  ddcx=evalgrad_dc(wd[fencode_dc(p,ix+(p->sodifon==0),iy,bdotv)]*w[fencode_dc(p,ix+(p->sodifon==0),iy,b1)],wd[fencode_dc(p,ix-(p->sodifon==0),iy,bdotv)]*w[fencode_dc(p,ix-(p->sodifon==0),iy,b1)],fip2,fim2,p,1);

 // dpi=(w[fencode_dc(p,ix,iy+1,b1)]*w[fencode_dc(p,ix,iy+1,mom1)]+w[fencode_dc(p,ix,iy+1,b2)]*w[fencode_dc(p,ix,iy+1,mom2)]+w[fencode_dc(p,ix,iy+1,b3)]*w[fencode_dc(p,ix,iy+1,mom3)])/w[fencode_dc(p,ix,iy+1,rho)];
 // dpim1=(w[fencode_dc(p,ix,iy-1,b1)]*w[fencode_dc(p,ix,iy-1,mom1)]+w[fencode_dc(p,ix,iy-1,b2)]*w[fencode_dc(p,ix,iy-1,mom2)]+w[fencode_dc(p,ix,iy-1,b3)]*w[fencode_dc(p,ix,iy-1,mom3)])/w[fencode_dc(p,ix,iy-1,rho)];  
  //dpip2=(w[fencode_dc(p,ix,iy+2,b1)]*w[fencode_dc(p,ix,iy+2,mom1)]+w[fencode_dc(p,ix,iy+2,b2)]*w[fencode_dc(p,ix,iy+2,mom2)]+w[fencode_dc(p,ix,iy+2,b3)]*w[fencode_dc(p,ix,iy+2,mom3)])/w[fencode_dc(p,ix,iy+2,rho)];
  //dpim2=(w[fencode_dc(p,ix,iy-2,b1)]*w[fencode_dc(p,ix,iy-2,mom1)]+w[fencode_dc(p,ix,iy-2,b2)]*w[fencode_dc(p,ix,iy-2,mom2)]+w[fencode_dc(p,ix,iy-2,b3)]*w[fencode_dc(p,ix,iy-2,mom3)])/w[fencode_dc(p,ix,iy-2,rho)];

 // fi=dpi*w[fencode_dc(p,ix,iy+1,b2)];
 // fim1=dpim1*w[fencode_dc(p,ix,iy-1,b2)];
if(p->sodifon==1)
{
  fip2=wd[fencode_dc(p,ix,iy+2,bdotv)]*w[fencode_dc(p,ix,iy+2,b2)];
  fim2=wd[fencode_dc(p,ix,iy-2,bdotv)]*w[fencode_dc(p,ix,iy-2,b2)];
}

//fi=w[fencode_dc(p,ix,iy+1,b2)];
//  fim1=w[fencode_dc(p,ix,iy-1,b2)];
  ddcy=evalgrad_dc(wd[fencode_dc(p,ix,iy+(p->sodifon==0),bdotv)]*w[fencode_dc(p,ix,iy+(p->sodifon==0),b2)],wd[fencode_dc(p,ix,iy-(p->sodifon==0),bdotv)]*w[fencode_dc(p,ix,iy-(p->sodifon==0),b2)],fip2,fim2,p,1);
//ddcx=0;
//ddcy=evalgrad_dc(((w[fencode_dc(p,ix,iy+1,b1)]*w[fencode_dc(p,ix,iy+1,mom1)]+w[fencode_dc(p,ix,iy+1,b2)]*w[fencode_dc(p,ix,iy+1,mom2)]+w[fencode_dc(p,ix,iy+1,b3)]*w[fencode_dc(p,ix,iy+1,mom3)])/w[fencode_dc(p,ix,iy+1,rho)])*w[fencode_dc(p,ix,iy+1,b2)],((w[fencode_dc(p,ix,iy-1,b1)]*w[fencode_dc(p,ix,iy-1,mom1)]+w[fencode_dc(p,ix,iy-1,b2)]*w[fencode_dc(p,ix,iy-1,mom2)]+w[fencode_dc(p,ix,iy-1,b3)]*w[fencode_dc(p,ix,iy-1,mom3)])/w[fencode_dc(p,ix,iy-1,rho)])*w[fencode_dc(p,ix,iy-1,b2)],0,0,p,1);

  dd2=(isnan(ddcx)?0:ddcx)+(isnan(ddcy)?0:ddcy);



  ddcx=wd[fencode_dc(p,ix,iy,pressuret)]*grad_dc(w,p,ix,iy,mom1,0)/w[fencode_dc(p,ix,iy,rho)];
  ddcy=wd[fencode_dc(p,ix,iy,pressuret)]*grad_dc(w,p,ix,iy,mom2,1)/w[fencode_dc(p,ix,iy,rho)];


  dd3=(isnan(ddcx)?0:ddcx)+(isnan(ddcy)?0:ddcy);

  
  return(dd1+dd2+dd3);
 //return dd1;
 // return ( ddc);
}

__device__ __host__
int derivcurrentrho (real *dw, real *wd, real *w, struct params *p,int ix, int iy) {

  int status=0;
  int field=rho;
        dw[fencode_dc(p,ix,iy,field)]= -ddotcurrentrho(dw,wd,w,p,ix,iy);
     	//dw[fencode_dc(p,ix,iy,field)]=w[fencode_dc(p,ix,iy,field)]+10;
  return ( status);
}

__device__ __host__
int derivcurrentmom (real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field, int direction) {

  int status=0;
     	//dw[fencode_dc(p,ix,iy,field)]=w[fencode_dc(p,ix,iy,field)]+20+5*(2*direction+1);
        dw[fencode_dc(p,ix,iy,field)]= -ddotcurrentmom(dw,wd,w,p,ix,iy,field,direction);
        //dw[fencode_dc(p,ix,iy,field)]=-ddotcurrentmom(dw,wd,w,p,ix,iy,field,direction);

  return ( status);
}

__device__ __host__
int derivcurrentb (real *dw, real *wd, real *w, struct params *p,int ix, int iy, int field, int direction) {

  int status=0;
        dw[fencode_dc(p,ix,iy,field)]=0;// -ddotcurrentb(dw,wd,w,p,ix,iy,field,direction);

  return ( status);
}

__device__ __host__
int derivcurrentenergy (real *dw, real *wd, real *w, struct params *p,int ix, int iy) {

  int status=0;
  int field=energy;
        dw[fencode_dc(p,ix,iy,field)]= -ddotcurrentenergy(dw,wd,w,p,ix,iy);

  return ( status);
}

//rho, mom1, mom2, mom3, energy, b1, b2, b3
__device__ __host__
void derivcurrent (real *dw, real *wd, real *w, struct params *p,int ix, int iy, int field) {

  //int status=0;
  switch(field)
  {
     case rho:
      derivcurrentrho(dw,wd,w,p,ix,iy);
     break;
     case mom1:
      derivcurrentmom(dw,wd,w,p,ix,iy,field,0);
     break;
     case mom2:
       derivcurrentmom(dw,wd,w,p,ix,iy,field,1);
     break;
     case mom3:
      derivcurrentmom(dw,wd,w,p,ix,iy,field,2);
     break;
     case energy:
       derivcurrentenergy(dw,wd,w,p,ix,iy);
     break;
     case b1:
       derivcurrentb(dw,wd,w,p,ix,iy,field,0);
     break;
     case b2:
      derivcurrentb(dw,wd,w,p,ix,iy,field,1);
     break;
     case b3:
      derivcurrentb(dw,wd,w,p,ix,iy,field,2);
     break;
  }
  //return ( status);
}



__global__ void derivcurrent_parallel(struct params *p, real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->ni;
  int nj=p->nj;
  real dt=p->dt;
  real dy=p->dy;
  real dx=p->dx;
  real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  

   j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);


  //if(i>(1+(p->sodifon==1)) && j >(1+(p->sodifon==1)) && i<((p->ni)-1-(p->sodifon==1)) && j<((p->nj)-1-(p->sodifon==1)))
if(i>1 && j >1 && i<(ni-2) && j<(nj-2))
	{		               
               /*for(int f=rho; f<=b3; f++)               
                  wmod[fencode_dc(p,i,j,f)]=w[fencode_dc(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               computebdotv(wmod,wd,p,i,j);*/
          
               for(int f=rho; f<=b3; f++)
               {              
                  derivcurrent(dwn1,wd,wmod,p,i,j,f);
                  //dwn1[fencode_dc(p,i,j,f)]=1.0;
                  //__syncthreads();
               }
               
               /*for(int f=rho; f<=b3; f++) 
                  wmod[fencode_dc(p,i,j,f)]=w[fencode_dc(p,i,j,f)]+0.5*dt*dwn1[fencode_dc(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn2,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  wmod[fencode_dc(p,i,j,f)]=w[fencode_dc(p,i,j,f)]+0.5*dt*dwn2[fencode_dc(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn3,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  wmod[fencode_dc(p,i,j,f)]=w[fencode_dc(p,i,j,f)]+dt*dwn3[fencode_dc(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn4,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  {
                  wnew[fencode_dc(p,i,j,f)]=w[fencode_dc(p,i,j,f)]+(dt/6.0)*(
                     dwn1[fencode_dc(p,i,j,f)]+2.0*dwn2[fencode_dc(p,i,j,f)]
                         +2.0*dwn3[fencode_dc(p,i,j,f)]+dwn4[fencode_dc(p,i,j,f)]);
               }*/
              //  __syncthreads();
              /* for(int f=rho; f<=b3; f++)
                   wnew[fencode_dc(p,i,j,f)]=w[fencode_dc(p,i,j,f)]+dt*dwn1[fencode_dc(p,i,j,f)];
               computej(wnew,wd,p,i,j);
               computepk(wnew,wd,p,i,j);
               computept(wnew,wd,p,i,j);*/ 


	}
 __syncthreads();
  
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_dc(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}




int cuderivcurrent(struct params **p, real **w, real **wnew, struct params **d_p, real **d_w, real **d_wnew,  real **d_wmod, real **d_dwn1, real **d_wd, int order)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
    dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
   int numBlocks = (((*p)->ni)*((*p)->nj)+numThreadsPerBlock-1) / numThreadsPerBlock;
 //  hipMemcpy(*w, *d_w, 8*((*p)->ni)* ((*p)->nj)*sizeof(real), hipMemcpyDeviceToHost);
 // if(order==0)
    hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
     derivcurrent_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, 8*((*p)->ni)* ((*p)->nj)*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, 8*((*p)->ni)* ((*p)->nj)*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->ni)* ((*p)->nj))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}


