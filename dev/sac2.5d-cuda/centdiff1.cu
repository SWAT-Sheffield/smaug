#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_cd1.cuh"
#include "dervfields_cd1.cuh"
__device__ __host__
void bc_periodic1_cd1t1(real *wt, struct params *p,int i, int j, int f) {

                if(i==0 || i==1 )                
                    wt[fencode_cd1(p,i,j,flux)]=wt[fencode_cd1(p,(p->n[0])-4+i,j,flux)];
                else if((i==((p->n[0])-1)) || (i==((p->n[0])-2)) )                
                   ;// wt[fencode_cd1(p,i,j,f)]=wt[fencode_cd1(p,4-(p->n[0])+i,j,f)];
                else if(j==0 || j==1 )                
                 ;// wt[fencode_cd1(p,i,j,f)]=wt[fencode_cd1(p,i,(p->n[1])-4+j,f)];
                else if((j==((p->n[1])-1)) || (j==((p->n[1])-2)) )                
                 ;// wt[fencode_cd1(p,i,j,f)]=wt[fencode_cd1(p,i,4-(p->n[1])+j,f)];

 


}


__device__ __host__
void bc_periodic2_cd1t1(real *wt, struct params *p,int i, int j, int f) {


               if(i<2 && j<2)
                {
                  if(i==j)
                    //wt[fencode_cd1(p,i,j,f)]=wt[fencode_cd1(p,(p->n[0])-3+i,j,f)];
                    wt[fencode_cd1(p,i,j,f)]=wt[fencode_cd1(p,i,(p->n[1])-4+j,f)];
                  else                  
                    //wt[fencode_cd1(p,i,j,f)]=wt[fencode_cd1(p,i,(p->n[1])-3+j,f)];
                    wt[fencode_cd1(p,i,j,f)]=wt[fencode_cd1(p,(p->n[0])-4+i,j,f)];                                    
                }
                else if(i<2 && j>((p->n[1])-3))
                {
                  if(i==(j-(p->n[1])))                  
                    //wt[fencode_cd1(p,i,j,f)]=wt[fencode_cd1(p,(p->n[0])-3+i,4-(p->n[1])+j,f)];
                    wt[fencode_cd1(p,i,j,f)]=wt[fencode_cd1(p,(p->n[0])-4+i,j,f)];                                     
                  else                  
                    wt[fencode_cd1(p,i,j,f)]=wt[fencode_cd1(p,i,4-(p->n[1])+j,f)];                                     
                }
                else if(i>((p->n[0])-3) && j<2)
                {
                  if((i-(p->n[0]))==j)                  
                    wt[fencode_cd1(p,i,j,f)]=wt[fencode_cd1(p,4-(p->n[0])+i,j,f)];                                    
                  else                  
                   wt[fencode_cd1(p,i,j,f)]=wt[fencode_cd1(p,i,(p->n[1])-4+j,f)];                                    
                }
                else if(i>((p->n[0])-3) && j>((p->n[1])-3))
                {
                  if(i==j)                  
                    wt[fencode_cd1(p,i,j,f)]=wt[fencode_cd1(p,i,4-(p->n[1])+j,f)];                                    
                  else                  
                    wt[fencode_cd1(p,i,j,f)]=wt[fencode_cd1(p,4-(p->n[0])+i,j,f)];                                    
                }                       
                 
                




}



__device__ __host__
real transportflux (real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field, int direction) {

  real ddc=0;
  real fi, fim1;
  real  fip2=0, fim2=0;
 // real ddc1;
  real ddcx=0,ddcy=0;

   real flux=0;

   //transport flux
    switch(direction)
  {
     case 0:
     //flux= wd[fencode_cd1(p,ix,iy,vel1)]*w[fencode_cd1(p,ix,iy,field)];
        #ifdef USE_SAC
     flux= w[fencode_cd1(p,ix,iy,mom1)]*w[fencode_cd1(p,ix,iy,field)]/(w[fencode_cd1(p,ix,iy,rho)]+w[fencode_cd1(p,ix,iy,rhob)]);
    // flux= w[fencode_cd1(p,ix,iy,mom1)]*w[fencode_cd1(p,ix,iy,field)]/w[fencode_cd1(p,ix,iy,rho)];

        #else
     flux= w[fencode_cd1(p,ix,iy,mom1)]*w[fencode_cd1(p,ix,iy,field)]/w[fencode_cd1(p,ix,iy,rho)];

        #endif
     break;
     case 1:
        #ifdef USE_SAC
     flux= w[fencode_cd1(p,ix,iy,mom2)]*w[fencode_cd1(p,ix,iy,field)]/(w[fencode_cd1(p,ix,iy,rho)]+w[fencode_cd1(p,ix,iy,rhob)]);
     //flux= w[fencode_cd1(p,ix,iy,mom2)]*w[fencode_cd1(p,ix,iy,field)]/w[fencode_cd1(p,ix,iy,rho)];

        #else
     //flux= wd[fencode_cd1(p,ix,iy,vel2)]*w[fencode_cd1(p,ix,iy,field)];
     flux= w[fencode_cd1(p,ix,iy,mom2)]*w[fencode_cd1(p,ix,iy,field)]/w[fencode_cd1(p,ix,iy,rho)];

        #endif
     break;
    /* case 2:
     flux= wd[fencode_cd1(p,ix,iy,vel3)]*w[fencode_cd1(p,ix,iy,field)];
     break;*/
   }
  return flux;


  //return ( ddc1-ddc2);
}






__device__ __host__
real fluxmom1 (real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field, int direction) {

  real ddc=0;
  real fi, fim1;
  real  fip2=0, fim2=0;
 // real ddc1;
  real ddcx=0,ddcy=0;

   real flux=0;

   //transport flux
    switch(direction)
  {
     case 0:
        #ifdef USE_SAC
     		flux= -w[fencode_cd1(p,ix,iy,field+3)]*w[fencode_cd1(p,ix,iy,b1)]-w[fencode_cd1(p,ix,iy,field+3)]*w[fencode_cd1(p,ix,iy,b1b)]-w[fencode_cd1(p,ix,iy,field+7)]*w[fencode_cd1(p,ix,iy,b1)];
        #endif
        #ifdef USE_VAC
                flux= -w[fencode_cd1(p,ix,iy,field+3)]*w[fencode_cd1(p,ix,iy,b1)];
         #endif

     break;
     case 1:
        #ifdef USE_SAC
                flux= -w[fencode_cd1(p,ix,iy,field+3)]*w[fencode_cd1(p,ix,iy,b2)]-w[fencode_cd1(p,ix,iy,field+3)]*w[fencode_cd1(p,ix,iy,b2b)]-w[fencode_cd1(p,ix,iy,field+7)]*w[fencode_cd1(p,ix,iy,b2)];
         #endif
        #ifdef USE_VAC
              flux= -w[fencode_cd1(p,ix,iy,field+3)]*w[fencode_cd1(p,ix,iy,b2)];
         #endif
     break;

   }




  return flux;


  //return ( ddc1-ddc2);
}










__device__ __host__
int computefluxrho (real *dw, real *wd, real *w, struct params *p,int ix, int iy,int direction) {

  int field;
  int status=0;

  // for(direction=0;direction<2;direction++)
         #ifdef USE_SAC
	      wd[fencode_cd1(p,ix,iy,flux)]= transportflux(dw,wd,w,p,ix,iy,rho,direction)+(w[fencode_cd1(p,ix,iy,rhob)]*w[fencode_cd1(p,ix,iy,mom1+direction)])/(w[fencode_cd1(p,ix,iy,rhob)]+w[fencode_cd1(p,ix,iy,rho)]);
         #else
             wd[fencode_cd1(p,ix,iy,flux)]= transportflux(dw,wd,w,p,ix,iy,rho,direction);
         #endif
  
  return ( status);
}

__device__ __host__
int computefluxmom (real *dw, real *wd, real *w, struct params *p,int ix, int iy, int field,int direction) {

 
  int status=0;
  //for(direction=0;direction<2;direction++)
  //{
    switch(field)
  {
     case mom1:
        #ifdef ADIABHYDRO
     		wd[fencode_cd1(p,ix,iy,flux)]= transportflux(dw,wd,w,p,ix,iy,field,direction);
        #endif
        #ifdef USE_VAC
    		wd[fencode_cd1(p,ix,iy,flux)]= transportflux(dw,wd,w,p,ix,iy,field,direction)+fluxmom1(dw,wd,w,p,ix,iy,field,direction);
 
        #endif
        #ifdef USE_SAC
    		wd[fencode_cd1(p,ix,iy,flux)]= transportflux(dw,wd,w,p,ix,iy,field,direction)+fluxmom1(dw,wd,w,p,ix,iy,field,direction);
               //if(direction==0)
               //   wd[fencode_cd1(p,ix,iy,f1)]+=wd[fencode_cd1(p,ix,iy,ptb)];
 
        #endif
               if(direction==0)
               {

                 computept_cd1(w,wd,p,ix,iy);
                 wd[fencode_cd1(p,ix,iy,flux)]+=wd[fencode_cd1(p,ix,iy,pressuret)];

        #ifdef USE_SAC

                 // wd[fencode_cd1(p,ix,iy,flux)]+=wd[fencode_cd1(p,ix,iy,ptb)];
       #endif
               }
 
     break;
     case mom2:
        #ifdef ADIABHYDRO
     		wd[fencode_cd1(p,ix,iy,flux)]= transportflux(dw,wd,w,p,ix,iy,field,direction);
        #endif
        #ifdef USE_VAC
    		wd[fencode_cd1(p,ix,iy,flux)]= transportflux(dw,wd,w,p,ix,iy,field,direction)+fluxmom1(dw,wd,w,p,ix,iy,field,direction);
 
        #endif
        #ifdef USE_SAC
    		wd[fencode_cd1(p,ix,iy,flux)]= transportflux(dw,wd,w,p,ix,iy,field,direction)+fluxmom1(dw,wd,w,p,ix,iy,field,direction);
               //if(direction==1)
               //   wd[fencode_cd1(p,ix,iy,f1)]+=wd[fencode_cd1(p,ix,iy,ptb)];
 
        #endif
               if(direction==1)
               {
                computept_cd1(w,wd,p,ix,iy);
                 wd[fencode_cd1(p,ix,iy,flux)]+=wd[fencode_cd1(p,ix,iy,pressuret)];

        #ifdef USE_SAC

                 // wd[fencode_cd1(p,ix,iy,f1)]+=wd[fencode_cd1(p,ix,iy,ptb)];
        #endif

               }
 
     break;
 
  // }
}
        
  return ( status);
}







//rho, mom1, mom2, mom3, energy, b1, b2, b3
__device__ __host__
void computeflux (real *dw, real *wd, real *w, struct params *p,int ix, int iy, int field,int dir) {

  //int status=0;
  switch(field)
  {
     case rho:
      computefluxrho(dw,wd,w,p,ix,iy,dir);
     break;
     case mom1:
      computefluxmom(dw,wd,w,p,ix,iy,field,dir);
      //wd[fencode_cd1(p,ix,iy,f1)]+=wd[fencode_cd1(p,ix,iy,pressuret)];
     break;
     case mom2:
       computefluxmom(dw,wd,w,p,ix,iy,field,dir);
       //wd[fencode_cd1(p,ix,iy,f2)]+=wd[fencode_cd1(p,ix,iy,pressuret)];
     break;
     /*case mom3:
      computefluxmom(dw,wd,w,p,ix,iy,field);
      //wd[fencode_cd1(p,ix,iy,f3)]+=wd[fencode_cd1(p,ix,iy,pressuret)];
     break;*/
  }
  //return ( status);
}



__global__ void centdiff1_parallel(struct params *p, real *w, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real dt, int f, int dir)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int fid;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

   int ip,jp,ipg,jpg;
   fid=0;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));

   
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;


             //  for(int f=rho; f<=mom3; f++)
             //  {
			if(i<(ni) && j<(nj))
                        {
                            dwn1[fencode_cd1(p,i,j,f)]=0.0;
                  	    //for(fid=0;fid<2;fid++)
                               wd[fencode_cd1(p,i,j,flux)]=0.0;
                               //wmod[fencode_cd1(p,i,j,flux)+order*NVAR*(p->n[0])*(p->n[1])]=0.0;
                        }
 __syncthreads();
   }
                       



   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;

//1. 11/1/11 could swap cases below
                        switch(dir)
                        {
                         case 0:
                         if(i<(ni)  && j >1 &&  j<(nj-1))
                         //if(i>1 && i<(ni-1)  && j >2 &&  j<(nj-3))
                            computeflux(dwn1,wd,wmod+order*NVAR*(p->n[0])*(p->n[1]),p,i,j,f,dir); 
                         break;
                         case 1:
                         if(i>1 &&  i<(ni-1) && j<(nj))
                         //if(j>1 && i>2 &&  i<(ni-3) && j<(nj))
                            computeflux(dwn1,wd,wmod+order*NVAR*(p->n[0])*(p->n[1]),p,i,j,f,dir); 
                         break;
                        }
              //  }
                        //might need to set boundaries correctly
 __syncthreads(); 
}
                       

/*  for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
          if( i<(ni) && j<(nj))
             //for(fid=0;fid<2;fid++)
              #ifdef ADIABHYDRO
                  bc_cont_cd1(wd,p,i,j,flux);
              #else
                  bc_periodic1_cd1t1(wd,p,i,j,flux);


              #endif
                __syncthreads();
}



#ifndef ADIABHYDRO
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
          if( i<(ni) && j<(nj) )
             //for(fid=0;fid<2;fid++)
                  //bc_cont_cd1(dwn1,p,i,j,f1+fid);
             
                 bc_periodic2_cd1t1(wd,p,i,j,flux);
                __syncthreads();
}

#endif*/


}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_cd1(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}




int cucentdiff1(struct params **p, real **w, struct params **d_p, real **d_w,  real **d_wmod, real **d_dwn1, real **d_wd, int order, int ordero, real dt, int field, int dir)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;
 //  hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
 // if(order==0)
    hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
//hipDeviceSynchronize();

     centdiff1_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wmod, *d_dwn1,  *d_wd, order, ordero,dt,field,dir);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}


