#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "iotypes.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_adv.cuh"
#include "dervfields_adv.cuh"


__global__ void advance_parallel(struct params *p, real *w, real *wmod,  
    real *dwn1, real *wd, int order)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[0];
  real dx=p->dx[1];
  
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));


   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
if( i<((p->n[0])) && j<((p->n[1])))
	{		               
 
               float big=9999.0;
               for(int f=rho; f<NVAR; f++)
               {
                  
                   
                  if((p->rkon)==1)
                  {
                     //wmod[fencode_adv(p,i,j,f)]=wmod[fencode_adv(p,i,j,f)+((p->n[0])*(p->n[1])*NVAR)];
                    switch(order)
                     {
                        case 0:
                       wmod[fencode_adv(p,i,j,f)+(2*(p->n[0])*(p->n[1])*NVAR)]=wmod[fencode_adv(p,i,j,f)];
                       // wmod[fencode_adv(p,i,j,f)]=wmod[fencode_adv(p,i,j,f)+((p->n[0])*(p->n[1])*NVAR)];

                         break;
                        case 1:
                       wmod[fencode_adv(p,i,j,f)+(3*(p->n[0])*(p->n[1])*NVAR)]=wmod[fencode_adv(p,i,j,f)];
                      // wmod[fencode_adv(p,i,j,f)]=wmod[fencode_adv(p,i,j,f)+(2*(p->n[0])*(p->n[1])*NVAR)];

                         break;
                        case 2:
                       wmod[fencode_adv(p,i,j,f)+((p->n[0])*(p->n[1])*NVAR)]=(wmod[fencode_adv(p,i,j,f)+((p->n[0])*(p->n[1])*NVAR)]+2.0*wmod[fencode_adv(p,i,j,f)+(2*(p->n[0])*(p->n[1])*NVAR)]+wmod[fencode_adv(p,i,j,f)+(3*(p->n[0])*(p->n[1])*NVAR)]-4.0*wmod[fencode_adv(p,i,j,f)])/3;


                         break;
                        case 3:
                      // wmod[fencode_adv(p,i,j,f)]=wmod[fencode_adv(p,i,j,f)]+wmod[fencode_adv(p,i,j,f)+((p->n[0])*(p->n[1])*NVAR)];
                        wmod[fencode_adv(p,i,j,f)]=wmod[fencode_adv(p,i,j,f)]+wmod[fencode_adv(p,i,j,f)+((p->n[0])*(p->n[1])*NVAR)];

                         break;

                     }
                   }
                  else
                  {
                  //if((dwn1[fencode_adv(p,i,j,f)]<(big/100)) && ( dwn1[fencode_adv(p,i,j,f)]>(-big/100)) )
                  //  if( j!=2)
                       //wmod[fencode_adv(p,i,j,f)]=wmod[fencode_adv(p,i,j,f)+(order*(p->n[0])*(p->n[1])*NVAR)];
                      wmod[fencode_adv(p,i,j,f)]=wmod[fencode_adv(p,i,j,f)+((p->n[0])*(p->n[1])*NVAR)];
                   //lax-friedrichs
                  //wmod[fencode_adv(p,i,j,f)]=((w[fencode_adv(p,i+1,j,f)]+w[fencode_adv(p,i-1,j,f)]+w[fencode_adv(p,i,j+1,f)]+w[fencode_adv(p,i,j-1,f)])/4.0)+(dt)*(dwn1[fencode_adv(p,i,j,f)]);
                   }
                  
                   if(isnan(wmod[fencode_adv(p,i,j,f)])) wmod[fencode_adv(p,i,j,f)]=w[fencode_adv(p,i,j,f)];
                   if(wmod[fencode_adv(p,i,j,f)]>big)
                           wmod[fencode_adv(p,i,j,f)]=w[fencode_adv(p,i,j,f)];
                   if(wmod[fencode_adv(p,i,j,f)]<-big)
                           wmod[fencode_adv(p,i,j,f)]=w[fencode_adv(p,i,j,f)];

                     if(f==rho)
                            if(wmod[fencode_adv(p,i,j,f)]<0)
                               wmod[fencode_adv(p,i,j,f)]=1.00;
               }
               //computej_adv(wmod,wd,p,i,j);
               //computepk_adv(wmod,wd,p,i,j);
               //computept_adv(wmod,wd,p,i,j);


	}
}
 __syncthreads();





  
}
/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_adv(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}






int cuadvance(struct params **p, real **w, real **wmod,struct params **d_p, real **d_w, real **d_wmod,  real **d_dwn1, real **d_wd, int order)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wmod, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
     advance_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wmod,  *d_dwn1,  *d_wd, order);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wmod);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wmod);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wmod, *d_wmod, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}



