#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "iotypes.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_i.cuh"






__device__ __host__
void init_alftest (real *w, struct params *p,int i, int j) {
  int seg1,seg2,seg3,seg4;
  int width=10;
  real m2max=0.001;
  real start=((p->n[0])-width)/2;
  //seg1=((p->n[0])/3)-1;
  seg1=(p->n[0])/6;
  seg2=((p->n[0])/3);
  seg3=(2*(p->n[0])/3)-1;
  //seg4=(2*(p->n[0])/3);
  seg4=(p->n[0])-seg1;

	#ifdef USE_SAC

		    w[fencode_i(p,i,j,rhob)]=1.0;
		    w[fencode_i(p,i,j,b1b)]=1.0;
		    w[fencode_i(p,i,j,energy)]=0.01;

       #else
		    w[fencode_i(p,i,j,rho)]=1.0;
		    w[fencode_i(p,i,j,b1)]=1.0;
		    w[fencode_i(p,i,j,energy)]=0.01;

       #endif

		    //w[fencode_i(p,i,j,b1)]=15*j;
		    //w[fencode_i(p,i,j,b3)]=150*j;
		    
		   //if (i > seg2)
		    //if (i < seg3)
                   // if (i < seg1)
		   //   w[fencode_i(p,i,j,mom2)]=0.0;

		   if (i > seg1)
		    if (i < seg2)
		      w[fencode_i(p,i,j,mom2)]=m2max*(i-seg1)/(seg2-seg1);

		   if (i > seg2)
		    if (i < seg3)
		      //w[fencode_i(p,i,j,mom2)]=m2max*(i-seg2)/(seg3-seg2);
                      w[fencode_i(p,i,j,mom2)]=m2max;
		   if (i > seg3)
		    if (i < seg4)
		      w[fencode_i(p,i,j,mom2)]=m2max*(seg4-i)/(seg4-seg3);






}


__device__ __host__
void init_ozttest (real *w, struct params *p,int i, int j) {
                    
                    //real b0=1.0/sqrt(4.0*PI);
                    real b0=1.0;
                    //real ptot=5.0/(12.0*PI);
                    real ptot=5.0/3.0;
                    real rrho=25.0/(36.0*PI);
                    real rgamm1;
             real e1,e2;


	#ifdef USE_SAC
                    b0=1.0/sqrt(4.0*PI);
                    ptot=5.0/(12.0*PI);
		    w[fencode_i(p,i,j,rhob)]=25.0/(36.0*PI);
                    //w[fencode_i(p,i,j,rhob)]=25.0/9.0;

                    rgamm1=1.0/((p->gamma)-1);
		    
		    w[fencode_i(p,i,j,b1)]=b0*sin((4.0*PI*p->dx[1])*j);
		    w[fencode_i(p,i,j,b2)]=-b0*sin(2.0*PI*(p->dx[0])*i);
		    //w[fencode_i(p,i,j,b2)]=-b0*sin((1.0*p->dx[1])*j);
		    //w[fencode_i(p,i,j,b1)]=b0*sin(2.0*(p->dx[0])*i);

                    //vx=-sin(2pi y)
                    //vy=sin(2pi x)
		    //w[fencode_i(p,i,j,mom1)]=-w[fencode_i(p,i,j,rhob)]*sin(2.0*PI*j*(p->dx[1]));
                    //w[fencode_i(p,i,j,mom2)]=w[fencode_i(p,i,j,rhob)]*sin(2.0*PI*j*(p->dx[0]));

		    w[fencode_i(p,i,j,mom2)]=-w[fencode_i(p,i,j,rhob)]*sin(2.0*PI*i*(p->dx[0]));
                    w[fencode_i(p,i,j,mom1)]=w[fencode_i(p,i,j,rhob)]*sin(2.0*PI*j*(p->dx[1]));
		    //w[fencode_i(p,i,j,mom1)]=-w[fencode_i(p,i,j,rho)]*sin(1.0*i*(p->dx[1]));
                    //w[fencode_i(p,i,j,mom2)]=w[fencode_i(p,i,j,rho)]*sin(1.0*j*(p->dx[0]));

//gives agreement with vac ozt
//w[fencode_i(p,i,j,energy)]=-sin(1.0*i*(p->dx[0]));
                    //w[fencode_i(p,i,j,energy)]=ptot+(0.5*((p->gamma)-1)*(w[fencode_i(p,i,j,mom1)]*w[fencode_i(p,i,j,mom1)]+w[fencode_i(p,i,j,mom2)]*w[fencode_i(p,i,j,mom2)])/rrho);
e1=ptot*rgamm1+(0.5*(w[fencode_i(p,i,j,mom1)]*w[fencode_i(p,i,j,mom1)]+w[fencode_i(p,i,j,mom2)]*w[fencode_i(p,i,j,mom2)])/rrho);
                    //w[fencode_i(p,i,j,energy)]+=0.5*((p->gamma)-2)*(w[fencode_i(p,i,j,b1)]*w[fencode_i(p,i,j,b1)]+w[fencode_i(p,i,j,b2)]*w[fencode_i(p,i,j,b2)]);

                   e2=0.5*(w[fencode_i(p,i,j,b1)]*w[fencode_i(p,i,j,b1)]+w[fencode_i(p,i,j,b2)]*w[fencode_i(p,i,j,b2)]);
                    //w[fencode_i(p,i,j,energy)]*=rgamm1;
                    w[fencode_i(p,i,j,energyb)]=(e1+e2);

                   w[fencode_i(p,i,j,energy)]=w[fencode_i(p,i,j,energyb)];
                   w[fencode_i(p,i,j,energyb)]=0.0;

                   w[fencode_i(p,i,j,rho)]=w[fencode_i(p,i,j,rhob)];
                   w[fencode_i(p,i,j,rhob)]=0.0;



       #else
		    //w[fencode_i(p,i,j,rho)]=25.0/(36.0*PI);
                    w[fencode_i(p,i,j,rho)]=25.0/9.0;
		    //w[fencode_i(p,i,j,b1)]=-b0*sin((p->dx[0])*i);
		    //w[fencode_i(p,i,j,b2)]=b0*sin(2.0*(p->dx[1])*j);
		    //w[fencode_i(p,i,j,b1)]=b0*sin((2.0*p->dx[1])*(p->n[0] -j));
		    //w[fencode_i(p,i,j,b2)]=-b0*sin(1.0*(p->dx[0])*(p->n[1] -i));
		    w[fencode_i(p,i,j,b1)]=b0*sin((2.0*p->dx[1])*j);
		    w[fencode_i(p,i,j,b2)]=-b0*sin(1.0*(p->dx[0])*i);

		    //w[fencode_i(p,i,j,b3)]=0.0;

                    //vx=-sin(2pi y)
                    //vy=sin(2pi x)
		    //w[fencode_i(p,i,j,mom1)]=-w[fencode_i(p,i,j,rho)]*sin(2.0*PI*j*(p->dx[1]));
                    //w[fencode_i(p,i,j,mom2)]=w[fencode_i(p,i,j,rho)]*sin(2.0*PI*j*(p->dx[0]));
		    //w[fencode_i(p,i,j,mom1)]=-w[fencode_i(p,i,j,rho)]*sin(i*(p->dx[0]));
                    //w[fencode_i(p,i,j,mom2)]=w[fencode_i(p,i,j,rho)]*sin(j*(p->dx[1]));
		    w[fencode_i(p,i,j,mom1)]=w[fencode_i(p,i,j,rho)]*sin(1.0*j*(p->dx[1]));
                    w[fencode_i(p,i,j,mom2)]=-w[fencode_i(p,i,j,rho)]*sin(1.0*i*(p->dx[0]));

		    //w[fencode_i(p,i,j,mom3)]=0;

                    //p=5/12pi  use this to determine the energy
                    //p=(gamma -1)*(e-0.5 rho v**2 - b**2/2)
                    rrho=1.0/w[fencode_i(p,i,j,rho)];
                    rgamm1=1.0/((p->gamma)-1);
		    //w[fencode_i(p,i,j,energy)]=(ptot/((p->gamma)-1))+0.5*rrho*(w[fencode_i(p,i,j,mom1)]*w[fencode_i(p,i,j,mom1)]+w[fencode_i(p,i,j,mom2)]*w[fencode_i(p,i,j,mom2)])+0.5*(w[fencode_i(p,i,j,b1)]*w[fencode_i(p,i,j,b1)]+w[fencode_i(p,i,j,b2)]*w[fencode_i(p,i,j,b2)]);
//w[fencode_i(p,i,j,energy)]=(ptot/((p->gamma)-1))+0.5*rrho;
		    //w[fencode_i(p,i,j,energy)]=(ptot-(p->gamma)*0.5*(w[fencode_i(p,i,j,b1)]*w[fencode_i(p,i,j,b1)]+w[fencode_i(p,i,j,b2)]*w[fencode_i(p,i,j,b2)]))*rgamm1+0.5*rrho*(w[fencode_i(p,i,j,mom1)]*w[fencode_i(p,i,j,mom1)]+w[fencode_i(p,i,j,mom2)]*w[fencode_i(p,i,j,mom2)]);
//w[fencode_i(p,i,j,energy)]=(ptot/((p->gamma)-1))+0.5*rrho*(w[fencode_i(p,i,j,mom1)]*w[fencode_i(p,i,j,mom1)]+w[fencode_i(p,i,j,mom2)]*w[fencode_i(p,i,j,mom2)])+0.5*(w[fencode_i(p,i,j,b1)]*w[fencode_i(p,i,j,b1)]+w[fencode_i(p,i,j,b2)]*w[fencode_i(p,i,j,b2)]);

//gives agreement with vac ozt
w[fencode_i(p,i,j,energy)]=-sin(1.0*i*(p->dx[0]));

       #endif





}


//*d_p,*d_w, *d_wnew, *d_wmod, *d_dwn1,  *d_wd

__global__ void init_parallel(struct params *p, real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd, real *wtemp, real *wtemp1, real *wtemp2)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  // int i = blockIdx.x * blockDim.x + threadIdx.x;
  // int j = blockIdx.y * blockDim.y + threadIdx.y;

 int iindex = blockIdx.x * blockDim.x + threadIdx.x;
 // int index,k;
int ni=p->n[0];
  int nj=p->n[1];

// Block index
    int bx = blockIdx.x;
   // int by = blockIdx.y;
    // Thread index
    int tx = threadIdx.x;
   // int ty = threadIdx.y;
    
  real *u,  *v,  *h;

   int ord;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  int i,j;
   
   
   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));

   
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;


  if(i<p->n[0] && j<p->n[1])
	{
		//b[i+j*(p->n[0])]=0;

                 //Define b	

 


	//apply this special condition
	//initiate alfven wave propagtion 
	//if no initial config read
	if(p->readini==0)
	{
	    for(int f=0; f<NVAR; f++)
            { 
		          w[fencode_i(p,i,j,f)]=0;
                          for(ord=0;ord<(2+3*(p->rkon==1));ord++)
                              wmod[fencode_i(p,i,j,f)+ord*NVAR*(p->n[0])*(p->n[1])]=0;
	    }
            w[fencode_i(p,i,j,rho)]=1.0;
            #ifdef ADIABHYDRO
		    if(i> (((p->n[0])/2)-2) && i<(((p->n[0])/2)+2) && j>(((p->n[1])/2)-2) && j<(((p->n[1])/2)+2) ) 
				w[fencode_i(p,i,j,rho)]=1.3;
            #else
                   // init_alftest (real *w, struct params *p,int i, int j)
                   // init_alftest(w,p,i,j);
                   // init_ozttest (real *w, struct params *p,int i, int j)
                    init_ozttest(w,p,i,j);
           #endif

	}


//	 __syncthreads();

			}

        }	
	 __syncthreads();

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;


  if(i<p->n[0] && j<p->n[1])
	{
        for(int f=rho; f<NVAR; f++)
        {               
                  wmod[fencode_i(p,i,j,f)]=w[fencode_i(p,i,j,f)];              
                  dwn1[fencode_i(p,i,j,f)]=0;
                            
        }

        for(int f=tmp1; f<NTEMP; f++)
                 wtemp[fencode_i(p,i,j,f)]=0;


}
}
 __syncthreads();

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;


        if(i<p->n[0] && j<p->n[1])
               for(int f=vel1; f<NDERV; f++)
                    wd[fencode_i(p,i,j,f)]=0.0;
   }

 __syncthreads(); 
}

/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_i(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}



int cuinit(struct params **p, real **w, real **wnew, struct state **state, struct params **d_p, real **d_w, real **d_wnew, real **d_wmod, real **d_dwn1, real **d_wd, struct state **d_state, real **d_wtemp, real **d_wtemp1, real **d_wtemp2)
{



/////////////////////////////////////
  // (1) initialisations:
  //     - perform basic sanity checks
  //     - set device
  /////////////////////////////////////
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
   
 // if (deviceCount == 0)
 // {
 //   fprintf(stderr, "Sorry, no CUDA device fount");
 //   return 1;
//  }
  if (selectedDevice >= deviceCount)
  {
    fprintf(stderr, "Choose device ID between 0 and %d\n", deviceCount-1);
    return 1;
  }
  //hipSetDevice(selectedDevice);
  printf("device count %d selected %d\n", deviceCount,selectedDevice);
  checkErrors_i("initialisations");
  
	// Build empty u, v, b matrices

  printf("in cuinit\n");
 // real *adb;
  real *adw, *adwnew;
  struct params *adp;
  struct state *ads;

if(((*p)->rkon)==1)
  hipMalloc((void**)d_wmod, 5*NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real));
else
  hipMalloc((void**)d_wmod, 2*NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real));

  hipMalloc((void**)d_dwn1, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real));
  hipMalloc((void**)d_wd, NDERV*((*p)->n[0])* ((*p)->n[1])*sizeof(real));
  hipMalloc((void**)d_wtemp, NTEMP*((*p)->n[0])* ((*p)->n[1])*sizeof(real));


  #ifndef ADIABHYDRO
  hipMalloc((void**)d_wtemp1, NTEMP1*(((*p)->n[0])+1)* (((*p)->n[1])+1)*sizeof(real));
  hipMalloc((void**)d_wtemp2, NTEMP2*(((*p)->n[0])+2)* (((*p)->n[1])+2)*sizeof(real));
  #endif


  hipMalloc((void**)&adw, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real));
  hipMalloc((void**)&adwnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real));
  
  hipMalloc((void**)&adp, sizeof(struct params));
  hipMalloc((void**)&ads, sizeof(struct state));
  checkErrors_i("memory allocation");

printf("ni is %d\n",(*p)->n[1]);

   // *d_b=adb;
    *d_p=adp;
    *d_w=adw;
    *d_wnew=adwnew;
    *d_state=ads;

     
printf("allocating\n");
    hipMemcpy(*d_w, *w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyHostToDevice);
   // hipMemcpy(*d_wnew, *wnew, 8*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyHostToDevice);
    
    hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);
    hipMemcpy(*d_state, *state, sizeof(struct state), hipMemcpyHostToDevice);
    
    dim3 dimBlock(16, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;
   

    printf("calling initialiser\n");
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
    // init_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
    // init_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wnew, *d_b);
     init_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wnew, *d_wmod, *d_dwn1,  *d_wd, *d_wtemp, *d_wtemp1, *d_wtemp2);
     hipDeviceSynchronize();
	    printf("called initialiser\n");
	hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);

	hipMemcpy(*state, *d_state, sizeof(struct state), hipMemcpyDeviceToHost);
        hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);
	//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
	//hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

        // printf("mod times step %f %f\n",(*p)->dt, ((*wnew)[10+16*((*p)->n[0])+((*p)->n[0])*((*p)->n[1])*b1]));



  return 0;



}


