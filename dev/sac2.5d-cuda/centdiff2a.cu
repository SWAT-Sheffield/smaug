#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_cd2a.cuh"
#include "dervfields_cd2a.cuh"



__device__ __host__
real fluxe2(real *dw, real *wd, real *w, struct params *p,int ix, int iy, int dir) {

  real ddc=0;
  real fi, fim1;
  real  fip2=0, fim2=0;
 // real ddc1;
  real ddcx=0,ddcy=0;

   real flux=0;


        #ifdef USE_SAC
computept_cd2a(w,wd,p,ix,iy);

// wd[fencode_cd2a(p,ix,iy,ptb)]=  ((p->gamma)-1)*w[fencode_cd2a(p,ix,iy,energyb)]- 0.5*((p->gamma)-2)*(w[fencode_cd2a(p,ix,iy,b1b)]*w[fencode_cd2a(p,ix,iy,b1b)]+w[fencode_cd2a(p,ix,iy,b2b)]*w[fencode_cd2a(p,ix,iy,b2b)]) ;


      		flux= wd[fencode_cd2a(p,ix,iy,ptb)]*grad_cd2a(wd,p,ix,iy,vel1+dir,dir);
                //flux     +=(w[fencode_cd2a(p,ix,iy,b1b)]*(w[fencode_cd2a(p,ix,iy,b1b)]+w[fencode_cd2a(p,ix,iy,b2b)]) +w[fencode_cd2a(p,ix,iy,b2b)]*(w[fencode_cd2a(p,ix,iy,b1b)]+w[fencode_cd2a(p,ix,iy,b2b)])); 
               // flux *= ((grad_cd2a(wd,p,ix,iy,vel1+dir,dir))); 
               flux += -w[fencode_cd2a(p,ix,iy,b1b)]*w[fencode_cd2a(p,ix,iy,b1b+dir)]*grad_cd2a(wd,p,ix,iy,vel1,0)-w[fencode_cd2a(p,ix,iy,b2b)]*w[fencode_cd2a(p,ix,iy,b1b+dir)]*grad_cd2a(wd,p,ix,iy,vel1+1,1);
         #endif

  return flux;


  //return ( ddc1-ddc2);
}



__device__ __host__
int divflux_cd2a(real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field,int dir) {

  int direction;
  int status=0;
  real divflux=0;
  dw[fencode_cd2a(p,ix,iy,field)]= grad_cd2a(wd,p,ix,iy,flux,dir);//+grad_cd2a(wd,p,ix,iy,f2,1); 


 #ifdef USE_SAC
  if(field==energy)     
     dw[fencode_cd2a(p,ix,iy,field)]+=fluxe2(dw, wd, w, p,ix, iy,dir);


 #endif
  return ( status);
}







__global__ void centdiff2a_parallel(struct params *p, real *w, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real dt,int f,int dir)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j,fid;
 // int index;
  int ni=p->n[0];
  int nj=p->n[1];
 // real dt=p->dt;
  //real dy=p->dx[1];
 // real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  

   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));






   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
             // for(int f=energy; f<NVAR; f++)
              // {
			if(i>1 && j >1 && i<(ni-2) && j<(nj-2))
			//if( i<(ni) && j<(nj))
                                divflux_cd2a(dwn1,wd,wmod+order*NVAR*(p->n[0])*(p->n[1]),p,i,j,f,dir); 
               // }
__syncthreads();
}
                        





             // for(int f=energy; f<=NVAR; f++)
               //{

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;

			 /*if(i>1 && j >1 && i<(ni-2) && j<(nj-2))
                              //                                                                                  - sign here same as vac maybe a +
                             // wmod[fencode_cd2a(p,i,j,f)+ordero*NVAR*(p->n[0])*(p->n[1])]=wmod[fencode_cd2a(p,i,j,f)]-dt*dwn1[fencode_cd2a(p,i,j,f)];
                             wmod[fencode_cd2a(p,i,j,f)+ordero*NVAR*(p->n[0])*(p->n[1])]=wmod[fencode_cd2a(p,i,j,f)+ordero*NVAR*(p->n[0])*(p->n[1])]-dt*dwn1[fencode_cd2a(p,i,j,f)]; */ 
               // }



                        //switch(dir)
                        //{
                         //case 0:
                         //if(i<(ni)  && j >1 &&  j<(nj-2))
                         //if(i >1 &&  i<(ni-2)  && j >1 &&  j<(nj-2))
                         if(i>1 && j >1 && i<(ni-2) && j<(nj-2))
                         //if(i>2 && j >2 && i<(ni-3) && j<(nj-3))
                         //if(i<(ni)  && j >1 &&  j<(nj-2))
                              wmod[fencode_cd2a(p,i,j,f)+(ordero*NVAR*(p->n[0])*(p->n[1]))]=wmod[fencode_cd2a(p,i,j,f)+(ordero*NVAR*(p->n[0])*(p->n[1]))]-dt*dwn1[fencode_cd2a(p,i,j,f)]; 
                         //break;
                         //case 1:
                         //if(i>1 &&  i<(ni-2) && j<(nj))
                         //if(i >1 &&  i<(ni-2)  && j >1 &&  j<(nj-2))
                         //if(i>3 && j >3 && i<(ni-4) && j<(nj-4))
                         //if(i>1 &&  i<(ni-2) && j<(nj))
                         //     wmod[fencode_cd2a(p,i,j,f)+(ordero*NVAR*(p->n[0])*(p->n[1]))]=wmod[fencode_cd2a(p,i,j,f)+(ordero*NVAR*(p->n[0])*(p->n[1]))]-dt*dwn1[fencode_cd2a(p,i,j,f)];
                         //break;
                        //}
__syncthreads(); 

}
                         
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_cd2a(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}




int cucentdiff2a(struct params **p, real **w, struct params **d_p, real **d_w,  real **d_wmod, real **d_dwn1, real **d_wd, int order,int ordero, real dt, int field,int dir)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;
 //  hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
 // if(order==0)
    hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
     centdiff2a_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wmod, *d_dwn1,  *d_wd, order,ordero,dt,field,dir);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}


