#include "hip/hip_runtime.h"


__device__ __host__
void computej_MODID(real *wmod,real *wd,struct params *p,int i,int j)
{
 // int status=0;

 // real dbzdy, dbydz;
 // real dbzdx, dbxdz;
 // real dbydx, dbxdy;

 // dbzdy=grad_MODID(wmod,p,i,j,b3,1);
 // dbydz=0.0;
 // dbzdx=grad_MODID(wmod,p,i,j,b3,0);
//  dbxdz=0.0;
 // dbydx=grad_MODID(wmod,p,i,j,b2,0);
 // dbxdy=grad_MODID(wmod,p,i,j,b1,1);

 /* wd[fencode_MODID(p,i,j,current1)]=(grad_MODID(wmod,p,i,j,b3,1))/(p->mu);
  wd[fencode_MODID(p,i,j,current2)]=(grad_MODID(wmod,p,i,j,b3,0))/(p->mu);
  wd[fencode_MODID(p,i,j,current3)]=(grad_MODID(wmod,p,i,j,b2,0)-grad_MODID(wmod,p,i,j,b1,1))/(p->mu);*/
  
          #ifdef USE_SAC
	 /* wd[fencode_MODID(p,i,j,current1)]+=(grad_MODID(wmod,p,i,j,b3b,1))/(p->mu);
	  wd[fencode_MODID(p,i,j,current2)]+=(grad_MODID(wmod,p,i,j,b3b,0))/(p->mu);
	  wd[fencode_MODID(p,i,j,current3)]+=(grad_MODID(wmod,p,i,j,b2b,0)-grad_MODID(wmod,p,i,j,b1b,1))/(p->mu);*/


         #endif

 
  //return ( status);
}

__device__ __host__
void computebdotv_MODID(real *wmod,real *wd,struct params *p,int i,int j)
{
        #ifdef USE_SAC

//wd[fencode_MODID(p,i,j,bdotv)]=((wmod[fencode_MODID(p,i,j,b1)]+wmod[fencode_MODID(p,i,j,b1b)])*wmod[fencode_MODID(p,i,j,mom1)]+(wmod[fencode_MODID(p,i,j,b2)]+wmod[fencode_MODID(p,i,j,b2b)])*wmod[fencode_MODID(p,i,j,mom2)]+(wmod[fencode_MODID(p,i,j,b3)]+wmod[fencode_MODID(p,i,j,b3b)])*wmod[fencode_MODID(p,i,j,mom3)])/(wmod[fencode_MODID(p,i,j,rho)]+wmod[fencode_MODID(p,i,j,rhob)]);
wd[fencode_MODID(p,i,j,bdotv)]=((wmod[fencode_MODID(p,i,j,b1)]+wmod[fencode_MODID(p,i,j,b1b)])*wmod[fencode_MODID(p,i,j,mom1)]+(wmod[fencode_MODID(p,i,j,b2)]+wmod[fencode_MODID(p,i,j,b2b)])*wmod[fencode_MODID(p,i,j,mom2)])/(wmod[fencode_MODID(p,i,j,rho)]+wmod[fencode_MODID(p,i,j,rhob)]);
         #else
//wd[fencode_MODID(p,i,j,bdotv)]=(wmod[fencode_MODID(p,i,j,b1)]*wmod[fencode_MODID(p,i,j,mom1)]+wmod[fencode_MODID(p,i,j,b2)]*wmod[fencode_MODID(p,i,j,mom2)]+wmod[fencode_MODID(p,i,j,b3)]*wmod[fencode_MODID(p,i,j,mom3)])/wmod[fencode_MODID(p,i,j,rho)];
wd[fencode_MODID(p,i,j,bdotv)]=(wmod[fencode_MODID(p,i,j,b1)]*wmod[fencode_MODID(p,i,j,mom1)]+wmod[fencode_MODID(p,i,j,b2)]*wmod[fencode_MODID(p,i,j,mom2)])/wmod[fencode_MODID(p,i,j,rho)];
         #endif
 // return ( status);
}

__device__ __host__
void computedivb_MODID(real *wmod,real *wd,struct params *p,int i,int j)
{


wd[fencode_MODID(p,i,j,divb)]=grad_MODID(wmod,p,i,j,b1,0)+grad_MODID(wmod,p,i,j,b2,1);
        #ifdef USE_SAC
		wd[fencode_MODID(p,i,j,divb)]+=grad_MODID(wmod,p,i,j,b1b,0)+grad_MODID(wmod,p,i,j,b2b,1);
         #endif
 // return ( status);
}

__device__ __host__
void computevel_MODID(real *wmod,real *wd,struct params *p,int i,int j)
{



        #ifdef USE_SAC
		wd[fencode_MODID(p,i,j,vel1)]=wmod[fencode_MODID(p,i,j,mom1)]/(wmod[fencode_MODID(p,i,j,rho)]+wmod[fencode_MODID(p,i,j,rhob)]);
                wd[fencode_MODID(p,i,j,vel2)]=wmod[fencode_MODID(p,i,j,mom2)]/(wmod[fencode_MODID(p,i,j,rho)]+wmod[fencode_MODID(p,i,j,rhob)]);
         #else
		wd[fencode_MODID(p,i,j,vel1)]=wmod[fencode_MODID(p,i,j,mom1)]/(wmod[fencode_MODID(p,i,j,rho)]);
                wd[fencode_MODID(p,i,j,vel2)]=wmod[fencode_MODID(p,i,j,mom2)]/(wmod[fencode_MODID(p,i,j,rho)]);

         #endif
 // return ( status);
}

__device__ __host__
void computept_MODID(real *wmod,real *wd,struct params *p,int i,int j)
{
 // int status=0;

#ifdef ADIABHYDRO

/*below used for adiabatic hydrodynamics*/
 wd[fencode_MODID(p,i,j,pressuret)]=(p->adiab)*pow(wmod[fencode_MODID(p,i,j,rho)],p->gamma);
// wd[fencode_MODID(p,i,j,pressuret)]=(p->adiab)*wmod[fencode_MODID(p,i,j,rho)]*wmod[fencode_MODID(p,i,j,rho)];
//wd[fencode_MODID(p,i,j,pressuret)]=1.0;
#elif defined(USE_SAC)
 
 //wd[fencode_MODID(p,i,j,pressuret)]=  ((p->gamma)-2)*((wmod[fencode_MODID(p,i,j,b1b)]*wmod[fencode_MODID(p,i,j,b1)]+wmod[fencode_MODID(p,i,j,b2b)]*wmod[fencode_MODID(p,i,j,b2)]))+0.5*(wmod[fencode_MODID(p,i,j,b1)]*wmod[fencode_MODID(p,i,j,b1)]+wmod[fencode_MODID(p,i,j,b2)]*wmod[fencode_MODID(p,i,j,b2)]);

//wd[fencode_MODID(p,i,j,pressuret)]= ((p->gamma)-1)*(wmod[fencode_MODID(p,i,j,energy)]-0.5*(wmod[fencode_MODID(p,i,j,mom1)]*wmod[fencode_MODID(p,i,j,mom1)]+wmod[fencode_MODID(p,i,j,mom2)]*wmod[fencode_MODID(p,i,j,mom2)])/(wmod[fencode_MODID(p,i,j,rho)]+wmod[fencode_MODID(p,i,j,rhob)]))-wd[fencode_MODID(p,i,j,pressuret)];



wd[fencode_MODID(p,i,j,pressuret)]=((p->gamma)-1.0)*( wmod[fencode_MODID(p,i,j,energy)]-0.5*(wmod[fencode_MODID(p,i,j,mom1)]*wmod[fencode_MODID(p,i,j,mom1)]+wmod[fencode_MODID(p,i,j,mom2)]*wmod[fencode_MODID(p,i,j,mom2)])/(wmod[fencode_MODID(p,i,j,rho)]+wmod[fencode_MODID(p,i,j,rhob)]));
wd[fencode_MODID(p,i,j,pressuret)]=wd[fencode_MODID(p,i,j,pressuret)]-((p->gamma)-2.0)*((wmod[fencode_MODID(p,i,j,b1)]*wmod[fencode_MODID(p,i,j,b1b)]+wmod[fencode_MODID(p,i,j,b2)]*wmod[fencode_MODID(p,i,j,b2b)])+0.5*(wmod[fencode_MODID(p,i,j,b1)]*wmod[fencode_MODID(p,i,j,b1)]+wmod[fencode_MODID(p,i,j,b2)]*wmod[fencode_MODID(p,i,j,b2)]));

 

 wd[fencode_MODID(p,i,j,ptb)]=  ((p->gamma)-1)*wmod[fencode_MODID(p,i,j,energyb)]- 0.5*((p->gamma)-2)*(wmod[fencode_MODID(p,i,j,b1b)]*wmod[fencode_MODID(p,i,j,b1b)]+wmod[fencode_MODID(p,i,j,b2b)]*wmod[fencode_MODID(p,i,j,b2b)]) ;


#else

 

 //wd[fencode_MODID(p,i,j,pressuret)]=  ((p->gamma)-1.0)*wmod[fencode_MODID(p,i,j,energy)]+(1.0-0.5*(p->gamma))*(wmod[fencode_MODID(p,i,j,b1)]*wmod[fencode_MODID(p,i,j,b1)]+wmod[fencode_MODID(p,i,j,b2)]*wmod[fencode_MODID(p,i,j,b2)])+0.5*(1.0-(p->gamma))*(wmod[fencode_MODID(p,i,j,mom1)]*wmod[fencode_MODID(p,i,j,mom1)]+wmod[fencode_MODID(p,i,j,mom2)]*wmod[fencode_MODID(p,i,j,mom2)])/wmod[fencode_MODID(p,i,j,rho)];

wd[fencode_MODID(p,i,j,pressuret)]=  ((p->gamma)-1.0)*wmod[fencode_MODID(p,i,j,energy)]+(1.0-0.5*(p->gamma))*(wmod[fencode_MODID(p,i,j,b1)]*wmod[fencode_MODID(p,i,j,b1)]+wmod[fencode_MODID(p,i,j,b2)]*wmod[fencode_MODID(p,i,j,b2)])+0.5*(1.0-(p->gamma))*(wmod[fencode_MODID(p,i,j,mom1)]*wmod[fencode_MODID(p,i,j,mom1)]+wmod[fencode_MODID(p,i,j,mom2)]*wmod[fencode_MODID(p,i,j,mom2)])/wmod[fencode_MODID(p,i,j,rho)];

#endif



  if(wd[fencode_MODID(p,i,j,pressuret)]<0)
              wd[fencode_MODID(p,i,j,pressuret)]=0.001;


 // return ( status);
}
__device__ __host__
void computepk_MODID(real *wmod,real *wd,struct params *p,int i,int j)
{
  //int status=0;

#ifdef ADIABHYDRO

/*below used for adiabatic hydrodynamics*/
wd[fencode_MODID(p,i,j,pressurek)]=(p->adiab)*pow(wmod[fencode_MODID(p,i,j,rho)],p->gamma);
wd[fencode_MODID(p,i,j,vel1)]=wmod[fencode_MODID(p,i,j,mom1)]/(wmod[fencode_MODID(p,i,j,rho)]);
wd[fencode_MODID(p,i,j,vel2)]=wmod[fencode_MODID(p,i,j,mom2)]/(wmod[fencode_MODID(p,i,j,rho)]);
//wd[fencode_MODID(p,i,j,vel3)]=wmod[fencode_MODID(p,i,j,mom3)]/(wmod[fencode_MODID(p,i,j,rho)]);
#elif defined(USE_SAC)

//wd[fencode_MODID(p,i,j,vel1)]=wmod[fencode_MODID(p,i,j,mom1)]/(wmod[fencode_MODID(p,i,j,rho)]+wmod[fencode_MODID(p,i,j,rhob)]);
//wd[fencode_MODID(p,i,j,vel2)]=wmod[fencode_MODID(p,i,j,mom2)]/(wmod[fencode_MODID(p,i,j,rho)]+wmod[fencode_MODID(p,i,j,rhob)]);
//wd[fencode_MODID(p,i,j,vel3)]=wmod[fencode_MODID(p,i,j,mom3)]/(wmod[fencode_MODID(p,i,j,rho)]+wmod[fencode_MODID(p,i,j,rhob)]);

// wd[fencode_MODID(p,i,j,pressurek)]=((p->gamma)-1)*(wmod[fencode_MODID(p,i,j,energy)]- 0.5*(wmod[fencode_MODID(p,i,j,mom1)]*wmod[fencode_MODID(p,i,j,mom1)]+wmod[fencode_MODID(p,i,j,mom2)]*wmod[fencode_MODID(p,i,j,mom2)]+wmod[fencode_MODID(p,i,j,mom3)]*wmod[fencode_MODID(p,i,j,mom3)])-0.5*(wmod[fencode_MODID(p,i,j,b1)]*wmod[fencode_MODID(p,i,j,b1)]+wmod[fencode_MODID(p,i,j,b2)]*wmod[fencode_MODID(p,i,j,b2)]+wmod[fencode_MODID(p,i,j,b3)]*wmod[fencode_MODID(p,i,j,b3)]) -(wmod[fencode_MODID(p,i,j,b1b)]*wmod[fencode_MODID(p,i,j,b1)]+wmod[fencode_MODID(p,i,j,b2b)]*wmod[fencode_MODID(p,i,j,b2)]+wmod[fencode_MODID(p,i,j,b3b)]*wmod[fencode_MODID(p,i,j,b3)]) );
 wd[fencode_MODID(p,i,j,pressurek)]=((p->gamma)-1)*(wmod[fencode_MODID(p,i,j,energy)]
- 0.5*((wmod[fencode_MODID(p,i,j,mom1)]*wmod[fencode_MODID(p,i,j,mom1)]+wmod[fencode_MODID(p,i,j,mom2)]*wmod[fencode_MODID(p,i,j,mom2)])/(wmod[fencode_MODID(p,i,j,rho)]+wmod[fencode_MODID(p,i,j,rhob)]))-0.5*(wmod[fencode_MODID(p,i,j,b1)]*wmod[fencode_MODID(p,i,j,b1)]+wmod[fencode_MODID(p,i,j,b2)]*wmod[fencode_MODID(p,i,j,b2)]) -(wmod[fencode_MODID(p,i,j,b1b)]*wmod[fencode_MODID(p,i,j,b1)]+wmod[fencode_MODID(p,i,j,b2b)]*wmod[fencode_MODID(p,i,j,b2)]) );

//wd[fencode_MODID(p,i,j,pkb)]=((p->gamma)-1)*(wmod[fencode_MODID(p,i,j,energyb)]- 0.5*(wmod[fencode_MODID(p,i,j,b1b)]*wmod[fencode_MODID(p,i,j,b1b)]+wmod[fencode_MODID(p,i,j,b2b)]*wmod[fencode_MODID(p,i,j,b2b)]+wmod[fencode_MODID(p,i,j,b3b)]*wmod[fencode_MODID(p,i,j,b3b)]) );
wd[fencode_MODID(p,i,j,pkb)]=((p->gamma)-1)*(wmod[fencode_MODID(p,i,j,energyb)]- 0.5*(wmod[fencode_MODID(p,i,j,b1b)]*wmod[fencode_MODID(p,i,j,b1b)]+wmod[fencode_MODID(p,i,j,b2b)]*wmod[fencode_MODID(p,i,j,b2b)]) );

#else
//wd[fencode_MODID(p,i,j,vel1)]=wmod[fencode_MODID(p,i,j,mom1)]/(wmod[fencode_MODID(p,i,j,rho)]);
//wd[fencode_MODID(p,i,j,vel2)]=wmod[fencode_MODID(p,i,j,mom2)]/(wmod[fencode_MODID(p,i,j,rho)]);
//wd[fencode_MODID(p,i,j,vel3)]=wmod[fencode_MODID(p,i,j,mom3)]/(wmod[fencode_MODID(p,i,j,rho)]);

 // wd[fencode_MODID(p,i,j,pressurek)]=((p->gamma)-1)*(wmod[fencode_MODID(p,i,j,energy)]- 0.5*(wmod[fencode_MODID(p,i,j,mom1)]*wmod[fencode_MODID(p,i,j,mom1)]+wmod[fencode_MODID(p,i,j,mom2)]*wmod[fencode_MODID(p,i,j,mom2)]+wmod[fencode_MODID(p,i,j,mom3)]*wmod[fencode_MODID(p,i,j,mom3)])/wmod[fencode_MODID(p,i,j,rho)]-0.5*(wmod[fencode_MODID(p,i,j,b1)]*wmod[fencode_MODID(p,i,j,b1)]+wmod[fencode_MODID(p,i,j,b2)]*wmod[fencode_MODID(p,i,j,b2)]+wmod[fencode_MODID(p,i,j,b3)]*wmod[fencode_MODID(p,i,j,b3)]) );
 wd[fencode_MODID(p,i,j,pressurek)]=((p->gamma)-1)*(wmod[fencode_MODID(p,i,j,energy)]- 0.5*(wmod[fencode_MODID(p,i,j,mom1)]*wmod[fencode_MODID(p,i,j,mom1)]+wmod[fencode_MODID(p,i,j,mom2)]*wmod[fencode_MODID(p,i,j,mom2)])/wmod[fencode_MODID(p,i,j,rho)]-0.5*(wmod[fencode_MODID(p,i,j,b1)]*wmod[fencode_MODID(p,i,j,b1)]+wmod[fencode_MODID(p,i,j,b2)]*wmod[fencode_MODID(p,i,j,b2)]) );


#endif






  if(wd[fencode_MODID(p,i,j,pressurek)]<0)
              wd[fencode_MODID(p,i,j,pressurek)]=0.001;
  //return ( status);
}

__device__ __host__
void computec_MODID(real *wmod,real *wd,struct params *p,int i,int j)
{

  
#ifdef ADIABHYDRO
/*below used for adiabatic hydrodynamics*/
  wd[fencode_MODID(p,i,j,soundspeed)]=sqrt((p->adiab)/wmod[fencode_MODID(p,i,j,rho)]);
#elif defined(USE_SAC)
//wd[fencode_MODID(p,i,j,soundspeed)]=sqrt((  (p->gamma))*(wd[fencode_MODID(p,i,j,pressuret)]+wd[fencode_MODID(p,i,j,ptb)])/(wmod[fencode_MODID(p,i,j,rho)]+wmod[fencode_MODID(p,i,j,rhob)]   ));


wd[fencode_MODID(p,i,j,soundspeed)]=sqrt(((p->gamma))*(wd[fencode_MODID(p,i,j,pressurek)]+(((p->gamma))-1)*(wmod[fencode_MODID(p,i,j,energyb)] -0.5*(wmod[fencode_MODID(p,i,j,b1b)]*wmod[fencode_MODID(p,i,j,b1b)]+wmod[fencode_MODID(p,i,j,b2b)]*wmod[fencode_MODID(p,i,j,b2b)]))/(wmod[fencode_MODID(p,i,j,rho)]+wmod[fencode_MODID(p,i,j,rhob)]   )));


wd[fencode_MODID(p,i,j,cfast)]=sqrt((   ( 
(wmod[fencode_MODID(p,i,j,b1)]*wmod[fencode_MODID(p,i,j,b1)]+wmod[fencode_MODID(p,i,j,b2)]*wmod[fencode_MODID(p,i,j,b2)]) + (wmod[fencode_MODID(p,i,j,b1b)]*wmod[fencode_MODID(p,i,j,b1b)]+wmod[fencode_MODID(p,i,j,b2b)]*wmod[fencode_MODID(p,i,j,b2b)]) +2.0*(wmod[fencode_MODID(p,i,j,b1b)]*wmod[fencode_MODID(p,i,j,b1)]+wmod[fencode_MODID(p,i,j,b2b)]*wmod[fencode_MODID(p,i,j,b2)])    )/(wmod[fencode_MODID(p,i,j,rho)]+wmod[fencode_MODID(p,i,j,rhob)]))
+(wd[fencode_MODID(p,i,j,soundspeed)]*wd[fencode_MODID(p,i,j,soundspeed)]));

#else
wd[fencode_MODID(p,i,j,soundspeed)]=sqrt(((p->gamma))*wd[fencode_MODID(p,i,j,pressuret)]/wmod[fencode_MODID(p,i,j,rho)]);
//wd[fencode_MODID(p,i,j,cfast)]=sqrt(((wmod[fencode_MODID(p,i,j,b1)]*wmod[fencode_MODID(p,i,j,b1)]+wmod[fencode_MODID(p,i,j,b2)]*wmod[fencode_MODID(p,i,j,b2)]+wmod[fencode_MODID(p,i,j,b3)]*wmod[fencode_MODID(p,i,j,b3)])/wmod[fencode_MODID(p,i,j,rho)])+(wd[fencode_MODID(p,i,j,soundspeed)]*wd[fencode_MODID(p,i,j,soundspeed)]));

wd[fencode_MODID(p,i,j,cfast)]=sqrt(((wmod[fencode_MODID(p,i,j,b1)]*wmod[fencode_MODID(p,i,j,b1)]+wmod[fencode_MODID(p,i,j,b2)]*wmod[fencode_MODID(p,i,j,b2)])/wmod[fencode_MODID(p,i,j,rho)])+(wd[fencode_MODID(p,i,j,soundspeed)]*wd[fencode_MODID(p,i,j,soundspeed)]));

#endif



  
}

__device__ __host__
void computecmax_MODID(real *wmod,real *wd,struct params *p,int i,int j)
{
 //p->cmax=0.02;
#ifdef ADIABHYDRO
       if(wd[fencode_MODID(p,i,j,soundspeed)]>(p->cmax))
                    // atomicExch(&(p->cmax),(wd[fencode_MODID(p,i,j,soundspeed)]));
                    p->cmax=(wd[fencode_MODID(p,i,j,soundspeed)]);
#else
       if(wd[fencode_MODID(p,i,j,soundspeed)]>(p->cmax))
                    // atomicExch(&(p->cmax),(wd[fencode_MODID(p,i,j,soundspeed)]));
                    p->cmax=(wd[fencode_MODID(p,i,j,soundspeed)]);
       if(wd[fencode_MODID(p,i,j,cfast)]>(p->cmax))
                    // atomicExch(&(p->cmax),(wd[fencode_MODID(p,i,j,soundspeed)]));
                    p->cmax=(wd[fencode_MODID(p,i,j,cfast)]);
#endif

}


