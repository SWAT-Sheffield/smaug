#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_dc1.cuh"
__device__ __host__
real transportflux (real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field, int direction) {

  real ddc=0;
  real fi, fim1;
  real  fip2=0, fim2=0;
 // real ddc1;
  real ddcx=0,ddcy=0;

   real flux=0;

   //transport flux
    switch(direction)
  {
     case 0:
     flux= wd[fencode_dc1(p,ix,iy,vel1)]*w[fencode_dc1(p,ix,iy,field)];
     break;
     case 1:
     flux= wd[fencode_dc1(p,ix,iy,vel2)]*w[fencode_dc1(p,ix,iy,field)];
     break;
     case 2:
     ;//flux= wd[fencode_dc1(p,ix,iy,vel3)]*w[fencode_dc1(p,ix,iy,field)];
     break;
   }
  return flux;


  //return ( ddc1-ddc2);
}






__device__ __host__
real fluxmom1 (real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field, int direction) {

  real ddc=0;
  real fi, fim1;
  real  fip2=0, fim2=0;
 // real ddc1;
  real ddcx=0,ddcy=0;

   real flux=0;

   //transport flux
    switch(direction)
  {
     case 0:
        #ifdef USE_SAC
      		flux= -w[fencode_dc1(p,ix,iy,field+4)]*w[fencode_dc1(p,ix,iy,b1)]-w[fencode_dc1(p,ix,iy,field+4)]*w[fencode_dc1(p,ix,iy,b1b)]-w[fencode_dc1(p,ix,iy,field+9)]*w[fencode_dc1(p,ix,iy,b1)];
         #else
      		flux= -w[fencode_dc1(p,ix,iy,field+4)]*w[fencode_dc1(p,ix,iy,b1)];
         #endif
        if((field==mom1 ) )
                       flux+=wd[fencode_dc1(p,ix,iy,pressuret)];
     break;
     case 1:
        #ifdef USE_SAC
	      flux= -w[fencode_dc1(p,ix,iy,field+4)]*w[fencode_dc1(p,ix,iy,b2)]-w[fencode_dc1(p,ix,iy,field+4)]*w[fencode_dc1(p,ix,iy,b2b)]-w[fencode_dc1(p,ix,iy,field+9)]*w[fencode_dc1(p,ix,iy,b2)];
         #else
	      flux= -w[fencode_dc1(p,ix,iy,field+4)]*w[fencode_dc1(p,ix,iy,b2)];
         #endif
         if((field==mom2 && direction==1)   )
                     flux+=wd[fencode_dc1(p,ix,iy,pressuret)];
     break;
     case 2:
        #ifdef USE_SAC
     	 	flux= -w[fencode_dc1(p,ix,iy,field+4)]*w[fencode_dc1(p,ix,iy,b3)]-w[fencode_dc1(p,ix,iy,field+4)]*w[fencode_dc1(p,ix,iy,b3b)]-w[fencode_dc1(p,ix,iy,field+9)]*w[fencode_dc1(p,ix,iy,b3)];
         #else
      		flux= -w[fencode_dc1(p,ix,iy,field+4)]*w[fencode_dc1(p,ix,iy,b3)];	
         #endif
         if((field==mom3 && direction==2))
                    flux+=wd[fencode_dc1(p,ix,iy,pressuret)];
     break;
   }




  return flux;


  //return ( ddc1-ddc2);
}










__device__ __host__
int computefluxrho (real *dw, real *wd, real *w, struct params *p,int ix, int iy) {

  int field, direction;
  int status=0;
  
   for(direction=0;direction<3;direction++)
         #ifdef USE_SAC
	      wd[fencode_dc1(p,ix,iy,f1+direction)]= transportflux(dw,wd,w,p,ix,iy,rho,direction)+transportflux(dw,wd,w,p,ix,iy,rhob,direction);
         #else
             wd[fencode_dc1(p,ix,iy,f1+direction)]= transportflux(dw,wd,w,p,ix,iy,rho,direction);
         #endif
  
  return ( status);
}

__device__ __host__
int computefluxmom (real *dw, real *wd, real *w, struct params *p,int ix, int iy, int field) {

  int direction;
  int status=0;
  for(direction=0;direction<3;direction++)
  {
    switch(direction)
  {
     case 0:
        #ifdef ADIABHYDRO
     		wd[fencode_dc1(p,ix,iy,f1)]= transportflux(dw,wd,w,p,ix,iy,field,direction);
        #else
    		 wd[fencode_dc1(p,ix,iy,f1)]= transportflux(dw,wd,w,p,ix,iy,field,direction)+fluxmom1(dw,wd,w,p,ix,iy,field,direction);
        #endif
     break;
     case 1:
        #ifdef ADIABHYDRO
	     wd[fencode_dc1(p,ix,iy,f2)]= transportflux(dw,wd,w,p,ix,iy,field,direction);
        #else
	     wd[fencode_dc1(p,ix,iy,f2)]= transportflux(dw,wd,w,p,ix,iy,field,direction)+fluxmom1(dw,wd,w,p,ix,iy,field,direction);
        #endif
     break;
     case 2:
        #ifdef ADIABHYDRO
	     wd[fencode_dc1(p,ix,iy,f3)]= transportflux(dw,wd,w,p,ix,iy,field,direction);
        #else
	     wd[fencode_dc1(p,ix,iy,f3)]= transportflux(dw,wd,w,p,ix,iy,field,direction)+fluxmom1(dw,wd,w,p,ix,iy,field,direction);
        #endif
     break;
   }
}
        
  return ( status);
}

__device__ __host__
int divflux1(real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field) {

  int direction;
  int status=0;
  real divflux=0;
  dw[fencode_dc1(p,ix,iy,field)]= -grad_dc1(wd,p,ix,iy,f1,0)-grad_dc1(wd,p,ix,iy,f2,1);      

  return ( status);
}





//rho, mom1, mom2, mom3, energy, b1, b2, b3
__device__ __host__
void computeflux (real *dw, real *wd, real *w, struct params *p,int ix, int iy, int field) {

  //int status=0;
  switch(field)
  {
     case rho:
      computefluxrho(dw,wd,w,p,ix,iy);
     break;
     case mom1:
      computefluxmom(dw,wd,w,p,ix,iy,field);
     break;
     case mom2:
       computefluxmom(dw,wd,w,p,ix,iy,field);
     break;
     case mom3:
      computefluxmom(dw,wd,w,p,ix,iy,field);
     break;
  }
  //return ( status);
}



__global__ void derivcurrent1_parallel(struct params *p, real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd, int order)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  

   j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);

               for(int f=rho; f<=mom3; f++)
               {

			if(i>1 && j >1 && i<(ni-2) && j<(nj-2))
                  		computeflux(dwn1+(NVAR*(p->n[0])*(p->n[1])*order),wd,wmod,p,i,j,f); 

                        //might need to set boundaries correctly 
                        __syncthreads();
			//if(i>1 && j >1 && i<(ni-2) && j<(nj-2))
                        //        divflux1(dwn1+(NVAR*(p->n[0])*(p->n[1])*order),wd,wmod,p,i,j,f);

			 if(i>1 && j >1 && i<(ni-2) && j<(nj-2))
                               divflux1(dwn1+(NVAR*(p->n[0])*(p->n[1])*order),wd,wmod,p,i,j,f);  
                }
     
	

  
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_dc1(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}




int cuderivcurrent1(struct params **p, real **w, real **wnew, struct params **d_p, real **d_w, real **d_wnew,  real **d_wmod, real **d_dwn1, real **d_wd, int order)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;
 //  hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
 // if(order==0)
    hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
     derivcurrent1_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd, order);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}


