#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_b.cuh"

__global__ void boundary_parallel(struct params *p, real *w, real *wnew, real *wd, real *wmod, int order)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int f;

  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[0];
  real dx=p->dx[1];
                real val=0;
  
   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));


/*#ifdef ADIABHYDRO
;
#else
  //This second call makes sure corners are set correctly
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {
             for( f=rho; f<=b2; f++)
{
     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
  if(i<p->n[0] && j<p->n[1])

                  bc_periodic1a_b(wmod+order*NVAR*(p->n[0])*(p->n[1]),p,i,j,f);

 __syncthreads();
   } 
}

#endif*/




   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;

               for( f=rho; f<=b2; f++)
               {
  if(i<p->n[0] && j<p->n[1])
	{

               //default continuous BC for all
               //gradient kept zero by copying variable values from edge of mesh to ghost cells
                //  bc_cont_b(wmod+order*NVAR*(p->n[0])*(p->n[1]),p,i,j,rho);
               
                 // bc_fixed_b(wmod+order*NVAR*(p->n[0])*(p->n[1]),p,i,j,rho,1.0);
               //   bc_fixed(wnew,p,i,j,rho,1.0);
               //   bc_periodic_b(wmod+order*NVAR*(p->n[0])*(p->n[1]),p,i,j,rho);
               

               

#ifdef ADIABHYDRO
                  bc_cont_b(wmod+order*NVAR*(p->n[0])*(p->n[1]),p,i,j,f);
#else
                 bc_periodic1_b(wmod+order*NVAR*(p->n[0])*(p->n[1]),p,i,j,f);
#endif                

                //  bc_fixed_b(wmod+order*NVAR*(p->n[0])*(p->n[1]),p,i,j,f,0.0);
                 // bc_fixed(wnew,p,i,j,f,val);

                 // bc_periodic1_b(wmod+order*NVAR*(p->n[0])*(p->n[1]),p,i,j,f);
                




               /*for(int f=vel1; f<NDERV; f++)
               {
                  bc_cont_b(wd,p,i,j,f);

                 //bc_fixed_b(wd,p,i,j,f,0.0);
                 //   bc_periodic(wd,p,i,j,f);

                  
               }*/

	}
 __syncthreads();
               }
}


#ifdef ADIABHYDRO
;
#else
  //This second call makes sure corners are set correctly
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {
             for( f=rho; f<=b2; f++)
{
     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
  if(i<p->n[0] && j<p->n[1])

                  bc_periodic2_b(wmod+order*NVAR*(p->n[0])*(p->n[1]),p,i,j,f);

 __syncthreads();
   } 
}

#endif



  
}

int cuboundary(struct params **p, real **w, real **wnew, struct params **d_p, real **d_w, real **d_wnew, real **d_wmod, real **d_dwn1, real **d_wd, int order)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   // dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   //int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;
int numBlocks = ((dimproduct_b(*p)+numThreadsPerBlock-1)) / numThreadsPerBlock;
//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
 	    //printf("called prop\n"); 
    // hipDeviceSynchronize();
    boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wd, *d_wmod, order);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
	    //printf("called update\n"); 
    hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}

