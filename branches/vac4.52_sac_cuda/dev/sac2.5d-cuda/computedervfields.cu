#include "hip/hip_runtime.h"
//#define MODID pre


#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"
#include "gradops_cdf.cuh"
#include "dervfields_cdf.cuh"
/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////


__global__ void computedervfields_parallel(struct params *p,  real *w,  real *wmod, 
    real *dwn1, real *wd, int order,int ordero)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
//  real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  

   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));


if(order == 0)
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;




if(i<((p->n[0])) && j<((p->n[1])))
	{		
 
               for(int f=rho; f<=b2; f++)
                  wmod[fencode_cdf(p,i,j,f)+((p->n[0]))*((p->n[1]))*NVAR]=wmod[fencode_cdf(p,i,j,f)]; 
        }
               __syncthreads();
}


   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
if(i<((p->n[0])) && j<((p->n[1])))
	{		

               for(int f=vel1; f<NDERV; f++)
                 ;// wd[fencode_cdf(p,i,j,f)]=0; 
               for(int f=rho; f<NVAR; f++)
                 ;// dwn1[fencode_cdf(p,i,j,f)]=0; 
        }
               __syncthreads();
}


//if(i>20 && j >20 && i<90 && j<90)
//	{
//               computepk_cdf(wmod+(order*((p->n[0]))*((p->n[1]))*NVAR),wd,p,i,j);
//              computept_cdf(wmod+(order*((p->n[0]))*((p->n[1]))*NVAR),wd,p,i,j);
//}
//              __syncthreads();


   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
#ifdef USE_VAC
 if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
                    computej_cdf(wmod+(order*((p->n[0]))*((p->n[1]))*NVAR),wd,p,i,j);
#endif

#ifdef USE_SAC
 if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
               ;//     computej_cdf(wmod+(order*((p->n[0]))*((p->n[1]))*NVAR),wd,p,i,j);
#endif
__syncthreads();
}



  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
  if(i<((p->n[0])) && j<((p->n[1])))
	{		               
             #ifdef ADIABHYDRO
               computepk_cdf(wmod+(order*((p->n[0]))*((p->n[1]))*NVAR),wd,p,i,j);
               computept_cdf(wmod+(order*((p->n[0]))*((p->n[1]))*NVAR),wd,p,i,j);
             #else
               //computej_cdf(wmod+(order*((p->n[0]))*((p->n[1]))*NVAR),wd,p,i,j);
               computepk_cdf(wmod+(order*((p->n[0]))*((p->n[1]))*NVAR),wd,p,i,j);
               computept_cdf(wmod+(order*((p->n[0]))*((p->n[1]))*NVAR),wd,p,i,j);

               computebdotv_cdf(wmod+(order*((p->n[0]))*((p->n[1]))*NVAR),wd,p,i,j);
               computedivb_cdf(wmod+(order*((p->n[0]))*((p->n[1]))*NVAR),wd,p,i,j);

             #endif

         }
              __syncthreads();
}


   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
  if(i<((p->n[0])) && j<((p->n[1])))
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
	{
 //determin cmax
               computec_cdf(wmod+(order*((p->n[0]))*((p->n[1]))*NVAR),wd,p,i,j);
        }
             __syncthreads();
}
 

if(iindex==0)
{
 //  for(ipg=0;ipg<(p->npgp[0]);ipg++)
 //  for(jpg=0;jpg<(p->npgp[1]);jpg++)
  // {

  //   i=ip*(p->npgp[0])+ipg;
 //    j=jp*(p->npgp[1])+jpg;
   //if( i<((p->n[0])) && j<((p->n[1])))
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
    p->cmax=0.0;
    for(i>1;i<((p->n[0])-2);i++)
      for(j>1;j<((p->n[1])-2);j++)
	{ 
               computecmax_cdf(wmod+(order*((p->n[0]))*((p->n[1]))*NVAR),wd,p,i,j);




	}

 //  }
}
 __syncthreads();

 /*if(i<(p->n[0]) && j<(p->n[1]))
	{ 
              // for(int f=vel1; f<NDERV; f++)
              for(int f=current1; f<=current2; f++)
                  bc_cont_cdf(wd,p,i,j,f);

	}
 __syncthreads();*/
  
}

/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_cdf(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}




int cucomputedervfields(struct params **p, real **w,  struct params **d_p, real **d_w,  real **d_wmod, real **d_dwn1, real **d_wd, int order,int ordero)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
     computedervfields_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wmod, *d_dwn1,  *d_wd, order,ordero);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
 

    hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);

     //following used for testing to check current soundspeeds etc
     //hipMemcpy(*w, *d_wd, 7*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}






