#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////



__device__ __host__
int encode (struct params *dp,int ix, int iy) {

  //int kSizeX=(dp)->ni;
  //int kSizeY=(dp)->nj;
  
  return ( iy * ((dp)->ni) + ix);
}

__device__ __host__
int fencode (struct params *dp,int ix, int iy, int field) {

  //int kSizeX=(dp)->ni;
  //int kSizeY=(dp)->nj;
  
  return ( (iy * ((dp)->ni) + ix)+(field*((dp)->ni)*((dp)->nj)));
}

__device__ __host__
float evalgrad(float fi, float fim1, float fip2, float fim2,struct params *p,int dir)
{
 //float valgrad;

 if(dir == 0)
 {
     //valgrad=(2.0/(3.0*(p->dx)))*(fi-fim1)-(1.0/(12.0*(p->dx)))*(fip2-fim2);
   return((1.0/(1.0*(p->dx)))*(fi-fim1));
 }
 else if(dir == 1)
 {
    // valgrad=(2.0/(3.0*(p->dy)))*(fi-fim1)-(1.0/(12.0*(p->dy)))*(fip2-fim2);
      return((1.0/(1.0*(p->dy)))*(fi-fim1));
 }

 return -1;
}


__device__ __host__
float grad(float *wmod,struct params *p,int i,int j,int field,int dir)
{
 //float valgrad;

 if(dir == 0)
 {
    // valgrad=(2.0/(3.0*(p->dx)))*(wmod[fencode(p,i,j,field)]-wmod[fencode(p,i-1,j,field)])-(1.0/(12.0*(p->dx)))*(wmod[fencode(p,i+2,j,field)]-wmod[fencode(p,i-2,j,field)]);
return((1.0/(1.0*(p->dx)))*(wmod[fencode(p,i+1,j,field)]-wmod[fencode(p,i-1,j,field)]));
 }
 else if(dir == 1)
 {
    // valgrad=(2.0/(3.0*(p->dy)))*(wmod[fencode(p,i,j,field)]-wmod[fencode(p,i,j-1,field)])-(1.0/(12.0*(p->dy)))*(wmod[fencode(p,i,j+2,field)]-wmod[fencode(p,i,j-2,field)]);
 return((1.0/(1.0*(p->dy)))*(wmod[fencode(p,i,j+1,field)]-wmod[fencode(p,i,j-1,field)]));

 }

 return -1;
}

__device__ __host__
void computej(float *wmod,float *wd,struct params *p,int i,int j)
{
 // int status=0;

 // float dbzdy, dbydz;
 // float dbzdx, dbxdz;
 // float dbydx, dbxdy;

 // dbzdy=grad(wmod,p,i,j,b3,1);
 // dbydz=0.0;
 // dbzdx=grad(wmod,p,i,j,b3,0);
//  dbxdz=0.0;
 // dbydx=grad(wmod,p,i,j,b2,0);
 // dbxdy=grad(wmod,p,i,j,b1,1);

  wd[fencode(p,i,j,0)]=(grad(wmod,p,i,j,b3,1))/(p->mu);
  wd[fencode(p,i,j,1)]=(grad(wmod,p,i,j,b3,0))/(p->mu);
  wd[fencode(p,i,j,2)]=(grad(wmod,p,i,j,b2,0)-grad(wmod,p,i,j,b1,1))/(p->mu);
 
  //return ( status);
}

__device__ __host__
void computebdotv(float *wmod,float *wd,struct params *p,int i,int j)
{
 // int status=0;
 //float bsq=wmod[fencode(p,i,j,b1)]*wmod[fencode(p,i,j,b1)]+wmod[fencode(p,i,j,b2)]*wmod[fencode(p,i,j,b2)]+wmod[fencode(p,i,j,b3)]*wmod[fencode(p,i,j,b3)];
//  wd[fencode(p,i,j,4)]=  wd[fencode(p,i,j,3)]+0.5*(wmod[fencode(p,i,j,b1)]*wmod[fencode(p,i,j,b1)]+wmod[fencode(p,i,j,b2)]*wmod[fencode(p,i,j,b2)]+wmod[fencode(p,i,j,b3)]*wmod[fencode(p,i,j,b3)]);

wd[fencode(p,i,j,bdotv)]=(wmod[fencode(p,i,j,b1)]*wmod[fencode(p,i,j,mom1)]+wmod[fencode(p,i,j,b2)]*wmod[fencode(p,i,j,mom2)]+wmod[fencode(p,i,j,b3)]*wmod[fencode(p,i,j,mom3)])/wmod[fencode(p,i,j,rho)];
 // return ( status);
}


__device__ __host__
void computepk(float *wmod,float *wd,struct params *p,int i,int j)
{
 // int status=0;
 //float bsq=wmod[fencode(p,i,j,b1)]*wmod[fencode(p,i,j,b1)]+wmod[fencode(p,i,j,b2)]*wmod[fencode(p,i,j,b2)]+wmod[fencode(p,i,j,b3)]*wmod[fencode(p,i,j,b3)];
  wd[fencode(p,i,j,4)]=  wd[fencode(p,i,j,3)]+0.5*(wmod[fencode(p,i,j,b1)]*wmod[fencode(p,i,j,b1)]+wmod[fencode(p,i,j,b2)]*wmod[fencode(p,i,j,b2)]+wmod[fencode(p,i,j,b3)]*wmod[fencode(p,i,j,b3)]);
 // return ( status);
}
__device__ __host__
void computept(float *wmod,float *wd,struct params *p,int i,int j)
{
  //int status=0;
  //float momsq=wmod[fencode(p,i,j,mom1)]*wmod[fencode(p,i,j,mom1)]+wmod[fencode(p,i,j,mom2)]*wmod[fencode(p,i,j,mom2)]+wmod[fencode(p,i,j,mom3)]*wmod[fencode(p,i,j,mom3)];
  //float bsq=wmod[fencode(p,i,j,b1)]*wmod[fencode(p,i,j,b1)]+wmod[fencode(p,i,j,b2)]*wmod[fencode(p,i,j,b2)]+wmod[fencode(p,i,j,b3)]*wmod[fencode(p,i,j,b3)];
  wd[fencode(p,i,j,3)]=((p->gamma)-1)*(wmod[fencode(p,i,j,energy)]- 0.5*(wmod[fencode(p,i,j,mom1)]*wmod[fencode(p,i,j,mom1)]+wmod[fencode(p,i,j,mom2)]*wmod[fencode(p,i,j,mom2)]+wmod[fencode(p,i,j,mom3)]*wmod[fencode(p,i,j,mom3)])/wmod[fencode(p,i,j,rho)]-0.5*(wmod[fencode(p,i,j,b1)]*wmod[fencode(p,i,j,b1)]+wmod[fencode(p,i,j,b2)]*wmod[fencode(p,i,j,b2)]+wmod[fencode(p,i,j,b3)]*wmod[fencode(p,i,j,b3)]) );
  //return ( status);
}

__device__ __host__
float sourcerho (float *dw, float *wd, float *w, struct params *p,int ix, int iy) {

 // float src=0;
 // int field=rho;
 
  return 0;
}

__device__ __host__
float sourcemom (float *dw, float *wd, float *w, struct params *p,int ix, int iy,int field, int direction) {

  //float src=0;
  switch(direction)
  {
	case 0:
         return(w[fencode(p,ix,iy,rho)]*(p->g1))-grad(wd,p,ix,iy,pressuret,0);
	break;
	case 1:
         return(w[fencode(p,ix,iy,rho)]*(p->g2))-grad(wd,p,ix,iy,pressuret,1);
	break;
	case 2:
         return(w[fencode(p,ix,iy,rho)]*(p->g3))-grad(wd,p,ix,iy,pressuret,2);
	break;
  }
  return 0;
}

__device__ __host__
float sourceb (float *dw, float *wd, float *w, struct params *p,int ix, int iy,int field, int direction) {

  //float src=0;
  switch(direction)
  {
	case 0:
         return(p->eta)*grad(wd,p,ix,iy,current3,1);
	break;
	case 1:
         return -(p->eta)*grad(wd,p,ix,iy,current3,0);
	break;
	case 2:
         return (p->eta)*(grad(wd,p,ix,iy,current2,0)-grad(wd,p,ix,iy,current1,1));
	break;
  }
  return 0;
}

__device__ __host__
float sourceenergy (float *dw, float *wd, float *w, struct params *p,int ix, int iy) {

 // float src=0;
  float srcg,srcb;
  int field=energy;
  float ddcx,ddcy;
  float fi,fim1;//fip2,fim2;
      srcg=(p->g1)*w[fencode(p,ix,iy,mom1)]+(p->g2)*w[fencode(p,ix,iy,mom2)]+(p->g3)*w[fencode(p,ix,iy,mom3)];

       fi=(w[fencode(p,ix+1,iy,b2)]*wd[fencode(p,ix+1,iy,current3)]-w[fencode(p,ix+1,iy,b3)]*wd[fencode(p,ix+1,iy,current2)]);
       fim1=(w[fencode(p,ix-1,iy,b2)]*wd[fencode(p,ix-1,iy,current3)]-w[fencode(p,ix-1,iy,b3)]*wd[fencode(p,ix-1,iy,current2)]);
      // fip2=(w[fencode(p,ix+2,iy,b2)]*wd[fencode(p,ix+2,iy,current3)]-w[fencode(p,ix+2,iy,b3)]*wd[fencode(p,ix+2,iy,current2)]);
     //  fim2=(w[fencode(p,ix-2,iy,b2)]*wd[fencode(p,ix-2,iy,current3)]-w[fencode(p,ix-2,iy,b3)]*wd[fencode(p,ix-2,iy,current2)]);
      // ddcx=evalgrad(fi,fim1,fip2,fim2,p,0);
      ddcx=evalgrad(fi,fim1,0,0,p,0);

       fi=(w[fencode(p,ix+1,iy,b3)]*wd[fencode(p,ix+1,iy,current1)]-w[fencode(p,ix+1,iy,b1)]*wd[fencode(p,ix+1,iy,current3)]);
       fim1=(w[fencode(p,ix,iy-1,b3)]*wd[fencode(p,ix,iy-1,current1)]-w[fencode(p,ix,iy-1,b1)]*wd[fencode(p,ix,iy-1,current3)]);
     //  fip2=(w[fencode(p,ix,iy+2,b3)]*wd[fencode(p,ix,iy+2,current1)]-w[fencode(p,ix,iy+2,b1)]*wd[fencode(p,ix,iy+2,current3)]);
     //  fim2=(w[fencode(p,ix,iy-2,b3)]*wd[fencode(p,ix,iy-2,current1)]-w[fencode(p,ix,iy-2,b1)]*wd[fencode(p,ix,iy-2,current3)]);
      // ddcx=evalgrad(fi,fim1,fip2,fim2,p,0);
      ddcy=evalgrad(fi,fim1,0,0,p,1);

      srcb=(isnan(ddcx)?0:ddcx)+(isnan(ddcy)?0:ddcy);

 // src=srcg+srcb;
  return ( srcg+srcb);
}


__device__ __host__
float ddotcurrentrho (float *dw, float *wd, float *w, struct params *p,int ix, int iy) {

  float ddc=0;
//  int field=rho;

      ddc= grad(w,p,ix,iy,mom1,0)+grad(w,p,ix,iy,mom2,1);
  return ( isnan(ddc)?0:ddc);
}

__device__ __host__
float ddotcurrentmom (float *dw, float *wd, float *w, struct params *p,int ix, int iy,int field, int direction) {

  float ddc=0;
  float fi, fim1;
  //float  fip2=0, fim2=0;
  float ddc1,ddc2;
  float ddcx,ddcy;
   //     ddc= grad(w,p,ix,iy,mom1,0)+grad(w,p,ix,iy,mom2,1);
//evalgrad(float fi, float fim1, float fip2, float fim2,struct params *p,int dir)
  //fi=w(fencode(p,ix,iy,rho))
  //calculate momentum current

//w[fencode(p,ix,iy,rho)])=1;
//w[fencode(p,ix-1,iy,rho)])=1;
//w[fencode(p,ix+2,iy,rho)])=1;
//w[fencode(p,ix-2,iy,rho)])=1;
//w[fencode(p,ix,iy,rho)])=1;
//w[fencode(p,ix,iy-1,rho)])=1;
//w[fencode(p,ix,iy+2,rho)])=1;
//w[fencode(p,ix,iy-2,rho)])=1;

  switch(direction)
  {
    case 0:
       fi=(w[fencode(p,ix+1,iy,mom1)]/w[fencode(p,ix+1,iy,rho)])*w[fencode(p,ix+1,iy,mom1)];
       fim1=(w[fencode(p,ix-1,iy,mom1)]/w[fencode(p,ix-1,iy,rho)])*w[fencode(p,ix-1,iy,mom1)];
    //   fip2=(w[fencode(p,ix+2,iy,mom1)]/w[fencode(p,ix+2,iy,rho)])*w[fencode(p,ix+2,iy,mom1)];
     //  fim2=(w[fencode(p,ix-2,iy,mom1)]/w[fencode(p,ix-2,iy,rho)])*w[fencode(p,ix-2,iy,mom1)];
      // ddcx=evalgrad(fi,fim1,fip2,fim2,p,0);
      ddcx=evalgrad(fi,fim1,0,0,p,0);
       //ddcx=fi-fim1;
       fi=(w[fencode(p,ix,iy+1,mom1)]/w[fencode(p,ix,iy+1,rho)])*w[fencode(p,ix,iy+1,mom2)];
       fim1=(w[fencode(p,ix,iy-1,mom1)]/w[fencode(p,ix,iy-1,rho)])*w[fencode(p,ix,iy-1,mom2)];
      // fip2=(w[fencode(p,ix,iy+2,mom1)]/w[fencode(p,ix,iy+2,rho)])*w[fencode(p,ix,iy+2,mom2)];
      // fim2=(w[fencode(p,ix,iy-2,mom1)]/w[fencode(p,ix,iy-2,rho)])*w[fencode(p,ix,iy-2,mom2)];
       //ddcy=fi;
       ddcy=evalgrad(fi,fim1,0,0,p,1);
       //ddcy=evalgrad(0,0,fip2,fim2,p,1);
    break;
    case 1:
       fi=(w[fencode(p,ix+1,iy,mom2)]/w[fencode(p,ix+1,iy,rho)])*w[fencode(p,ix+1,iy,mom1)];
       fim1=(w[fencode(p,ix-1,iy,mom2)]/w[fencode(p,ix-1,iy,rho)])*w[fencode(p,ix-1,iy,mom1)];
      // fip2=(w[fencode(p,ix+2,iy,mom2)]/w[fencode(p,ix+2,iy,rho)])*w[fencode(p,ix+2,iy,mom1)];
      // fim2=(w[fencode(p,ix-2,iy,mom2)]/w[fencode(p,ix-2,iy,rho)])*w[fencode(p,ix-2,iy,mom1)];
       ddcx=evalgrad(fi,fim1,0,0,p,0);
       fi=(w[fencode(p,ix,iy+1,mom2)]/w[fencode(p,ix,iy+1,rho)])*w[fencode(p,ix,iy+1,mom2)];
       fim1=(w[fencode(p,ix,iy-1,mom2)]/w[fencode(p,ix,iy-1,rho)])*w[fencode(p,ix,iy-1,mom2)];
      // fip2=(w[fencode(p,ix,iy+2,mom2)]/w[fencode(p,ix,iy+2,rho)])*w[fencode(p,ix,iy+2,mom2)];
      // fim2=(w[fencode(p,ix,iy-2,mom2)]/w[fencode(p,ix,iy-2,rho)])*w[fencode(p,ix,iy-2,mom2)];
       ddcy=evalgrad(fi,fim1,0,0,p,1);
    break;
    case 2:
       fi=(w[fencode(p,ix+1,iy,mom3)]/w[fencode(p,ix+1,iy,rho)])*w[fencode(p,ix+1,iy,mom1)];
       fim1=(w[fencode(p,ix-1,iy,mom3)]/w[fencode(p,ix-1,iy,rho)])*w[fencode(p,ix-1,iy,mom1)];
      // fip2=(w[fencode(p,ix+2,iy,mom3)]/w[fencode(p,ix+2,iy,rho)])*w[fencode(p,ix+2,iy,mom1)];
     //  fim2=(w[fencode(p,ix-2,iy,mom3)]/w[fencode(p,ix-2,iy,rho)])*w[fencode(p,ix-2,iy,mom1)];
       ddcx=evalgrad(fi,fim1,0,0,p,0);
       fi=(w[fencode(p,ix,iy+1,mom2)]/w[fencode(p,ix,iy+1,rho)])*w[fencode(p,ix,iy+1,mom2)];
       fim1=(w[fencode(p,ix,iy-1,mom3)]/w[fencode(p,ix,iy-1,rho)])*w[fencode(p,ix,iy-1,mom2)];
     //  fip2=(w[fencode(p,ix,iy+2,mom3)]/w[fencode(p,ix,iy+2,rho)])*w[fencode(p,ix,iy+2,mom2)];
     //  fim2=(w[fencode(p,ix,iy-2,mom3)]/w[fencode(p,ix,iy-2,rho)])*w[fencode(p,ix,iy-2,mom2)];
       ddcy=evalgrad(fi,fim1,0,0,p,1);
    break;
  }
  
  ddc1=(isnan(ddcx)?0:ddcx)+(isnan(ddcy)?0:ddcy);
//fip2=0, fim2=0;
  //calculate bfield current
  switch(direction)
  {
    case 0:
       fi=w[fencode(p,ix+1,iy,b1)]*w[fencode(p,ix+1,iy,b1)];
       fim1=w[fencode(p,ix-1,iy,b1)]*w[fencode(p,ix-1,iy,b1)];
     // fip2=w[fencode(p,ix+2,iy,b1)]*w[fencode(p,ix+2,iy,b1)];
     //  fim2=w[fencode(p,ix-2,iy,b1)]*w[fencode(p,ix-2,iy,b1)];
       ddcx=evalgrad(fi,fim1,0,0,p,0);
       fi=w[fencode(p,ix,iy+1,b1)]*w[fencode(p,ix,iy+1,b2)];
       fim1=w[fencode(p,ix,iy-1,b1)]*w[fencode(p,ix,iy-1,b2)];
     //  fip2=w[fencode(p,ix,iy+2,b1)]*w[fencode(p,ix,iy+2,b2)];
      // fim2=w[fencode(p,ix,iy-2,b1)]*w[fencode(p,ix,iy-2,b2)];
       ddcy=evalgrad(fi,fim1,0,0,p,1);
    break;
    case 1:
       fi=w[fencode(p,ix+1,iy,b2)]*w[fencode(p,ix+1,iy,b1)];
       fim1=w[fencode(p,ix-1,iy,b2)]*w[fencode(p,ix-1,iy,b1)];
     //  fip2=w[fencode(p,ix+2,iy,b2)]*w[fencode(p,ix+2,iy,b1)];
      // fim2=w[fencode(p,ix-2,iy,b2)]*w[fencode(p,ix-2,iy,b1)];
       ddcx=evalgrad(fi,fim1,0,0,p,0);
       fi=w[fencode(p,ix,iy+1,b2)]*w[fencode(p,ix,iy+1,b2)];
       fim1=w[fencode(p,ix,iy-1,b2)]*w[fencode(p,ix,iy-1,b2)];
      // fip2=w[fencode(p,ix,iy+2,b2)]*w[fencode(p,ix,iy+2,b2)];
      // fim2=w[fencode(p,ix,iy-2,b2)]*w[fencode(p,ix,iy-2,b2)];
       ddcy=evalgrad(fi,fim1,0,0,p,1);
    break;
    case 2:
       fi=w[fencode(p,ix+1,iy,b3)]*w[fencode(p,ix+1,iy,b1)];
       fim1=w[fencode(p,ix-1,iy,b3)]*w[fencode(p,ix-1,iy,b1)];
      // fip2=w[fencode(p,ix+2,iy,b3)]*w[fencode(p,ix+2,iy,b1)];
      // fim2=w[fencode(p,ix-2,iy,b3)]*w[fencode(p,ix-2,iy,b1)];
       ddcx=evalgrad(fi,fim1,0,0,p,0);
       fi=w[fencode(p,ix,iy+1,b3)]*w[fencode(p,ix,iy+1,b2)];
       fim1=w[fencode(p,ix,iy-1,b3)]*w[fencode(p,ix,iy-1,b2)];
      // fip2=w[fencode(p,ix,iy+2,b3)]*w[fencode(p,ix,iy+2,b2)];
     //  fim2=w[fencode(p,ix,iy-2,b3)]*w[fencode(p,ix,iy-2,b2)];
       ddcy=evalgrad(fi,fim1,0,0,p,1);
    break;
  }
  //ddc2=ddcx+ddcy;
  ddc2=(isnan(ddcx)?0:ddcx)+(isnan(ddcy)?0:ddcy);

  //ddc=ddc1-ddc2;

  return ( ddc1-ddc2);
}

__device__ __host__
float ddotcurrentb (float *dw, float *wd, float *w, struct params *p,int ix, int iy,int field, int direction) {

  //float ddc=0;

  float fi, fim1;// fip2=0, fim2=0;
  float ddc1,ddc2;
  float ddcx,ddcy;

  switch(direction)
  {
	case 0:
	       fi=w[fencode(p,ix+1,iy,mom1)]*w[fencode(p,ix+1,iy,b1)]/w[fencode(p,ix+1,iy,rho)];
	       fim1=w[fencode(p,ix-1,iy,mom1)]*w[fencode(p,ix-1,iy,b1)]/w[fencode(p,ix-1,iy,rho)];
	       //fip2=w[fencode(p,ix+2,iy,mom1)]*w[fencode(p,ix+2,iy,b1)]/w[fencode(p,ix+2,iy,rho)];
	       //fim2=w[fencode(p,ix-2,iy,mom1)]*w[fencode(p,ix-2,iy,b1)]/w[fencode(p,ix-2,iy,rho)];
	       ddcx=evalgrad(fi,fim1,0,0,p,0);
	       fi=w[fencode(p,ix,iy+1,mom1)]*w[fencode(p,ix,iy+1,b2)]/w[fencode(p,ix,iy+1,rho)];
	       fim1=w[fencode(p,ix,iy-1,mom1)]*w[fencode(p,ix,iy-1,b2)]/w[fencode(p,ix,iy-1,rho)];
	       //fip2=w[fencode(p,ix,iy+2,mom1)]*w[fencode(p,ix,iy+2,b2)]/w[fencode(p,ix,iy+2,rho)];
	       //fim2=w[fencode(p,ix,iy-2,mom1)]*w[fencode(p,ix,iy-2,b2)]/w[fencode(p,ix,iy-2,rho)];
	       ddcy=evalgrad(fi,fim1,0,0,p,1);
        break;
	case 1:
	       fi=w[fencode(p,ix+1,iy,mom2)]*w[fencode(p,ix+1,iy,b1)]/w[fencode(p,ix+1,iy,rho)];
	       fim1=w[fencode(p,ix-1,iy,mom2)]*w[fencode(p,ix-1,iy,b1)]/w[fencode(p,ix-1,iy,rho)];
	       //fip2=w[fencode(p,ix+2,iy,mom2)]*w[fencode(p,ix+2,iy,b1)]/w[fencode(p,ix+2,iy,rho)];
	       //fim2=w[fencode(p,ix-2,iy,mom2)]*w[fencode(p,ix-2,iy,b1)]/w[fencode(p,ix-2,iy,rho)];
	       ddcx=evalgrad(fi,fim1,0,0,p,0);
	       fi=w[fencode(p,ix,iy+1,mom2)]*w[fencode(p,ix,iy+1,b2)]/w[fencode(p,ix,iy+1,rho)];
	       fim1=w[fencode(p,ix,iy-1,mom2)]*w[fencode(p,ix,iy-1,b2)]/w[fencode(p,ix,iy-1,rho)];
	       //fip2=w[fencode(p,ix,iy+2,mom2)]*w[fencode(p,ix,iy+2,b2)]/w[fencode(p,ix,iy+2,rho)];
	       //fim2=w[fencode(p,ix,iy-2,mom2)]*w[fencode(p,ix,iy-2,b2)]/w[fencode(p,ix,iy-2,rho)];
	       ddcy=evalgrad(fi,fim1,0,0,p,1);
        break;
	case 2:
	       fi=w[fencode(p,ix+1,iy,mom3)]*w[fencode(p,ix+1,iy,b1)]/w[fencode(p,ix+1,iy,rho)];
	       fim1=w[fencode(p,ix-1,iy,mom3)]*w[fencode(p,ix-1,iy,b1)]/w[fencode(p,ix-1,iy,rho)];
	       //fip2=w[fencode(p,ix+2,iy,mom3)]*w[fencode(p,ix+2,iy,b1)]/w[fencode(p,ix+2,iy,rho)];
	       //fim2=w[fencode(p,ix-2,iy,mom3)]*w[fencode(p,ix-2,iy,b1)]/w[fencode(p,ix-2,iy,rho)];
	       ddcx=evalgrad(fi,fim1,0,0,p,0);
	       fi=w[fencode(p,ix,iy+1,mom3)]*w[fencode(p,ix,iy+1,b2)]/w[fencode(p,ix,iy+1,rho)];
	       fim1=w[fencode(p,ix,iy-1,mom3)]*w[fencode(p,ix,iy-1,b2)]/w[fencode(p,ix,iy-1,rho)];
	       //fip2=w[fencode(p,ix,iy+2,mom3)]*w[fencode(p,ix,iy+2,b2)]/w[fencode(p,ix,iy+2,rho)];
	       //fim2=w[fencode(p,ix,iy-2,mom3)]*w[fencode(p,ix,iy-2,b2)]/w[fencode(p,ix,iy-2,rho)];
	       ddcy=evalgrad(fi,fim1,0,0,p,1);

        break;
  }
  ddc1=(isnan(ddcx)?0:ddcx)+(isnan(ddcy)?0:ddcy);



  switch(direction)
  {
	case 0:
	       fi=w[fencode(p,ix+1,iy,b1)]*w[fencode(p,ix+1,iy,mom1)]/w[fencode(p,ix+1,iy,rho)];
	       fim1=w[fencode(p,ix-1,iy,b1)]*w[fencode(p,ix-1,iy,mom1)]/w[fencode(p,ix-1,iy,rho)];
	       //fip2=w[fencode(p,ix+2,iy,b1)]*w[fencode(p,ix+2,iy,mom1)]/w[fencode(p,ix+2,iy,rho)];
	      // fim2=w[fencode(p,ix-2,iy,b1)]*w[fencode(p,ix-2,iy,mom1)]/w[fencode(p,ix-2,iy,rho)];
	       ddcx=evalgrad(fi,fim1,0,0,p,0);
	       fi=w[fencode(p,ix,iy+1,b1)]*w[fencode(p,ix,iy+1,mom2)]/w[fencode(p,ix,iy+1,rho)];
	       fim1=w[fencode(p,ix,iy-1,b1)]*w[fencode(p,ix,iy-1,mom2)]/w[fencode(p,ix,iy-1,rho)];
	       //fip2=w[fencode(p,ix,iy+2,b1)]*w[fencode(p,ix,iy+2,mom2)]/w[fencode(p,ix,iy+2,rho)];
	       //fim2=w[fencode(p,ix,iy-2,b1)]*w[fencode(p,ix,iy-2,mom2)]/w[fencode(p,ix,iy-2,rho)];
	       ddcy=evalgrad(fi,fim1,0,0,p,1);
        break;
	case 1:
	       fi=w[fencode(p,ix+1,iy,b2)]*w[fencode(p,ix+1,iy,mom1)]/w[fencode(p,ix+1,iy,rho)];
	       fim1=w[fencode(p,ix-1,iy,b2)]*w[fencode(p,ix-1,iy,mom1)]/w[fencode(p,ix-1,iy,rho)];
	       //fip2=w[fencode(p,ix+2,iy,b2)]*w[fencode(p,ix+2,iy,mom1)]/w[fencode(p,ix+2,iy,rho)];
	      // fim2=w[fencode(p,ix-2,iy,b2)]*w[fencode(p,ix-2,iy,mom1)]/w[fencode(p,ix-2,iy,rho)];
	       ddcx=evalgrad(fi,fim1,0,0,p,0);
	       fi=w[fencode(p,ix,iy+1,b2)]*w[fencode(p,ix,iy+1,mom2)]/w[fencode(p,ix,iy+1,rho)];
	       fim1=w[fencode(p,ix,iy-1,b2)]*w[fencode(p,ix,iy-1,mom2)]/w[fencode(p,ix,iy-1,rho)];
	      // fip2=w[fencode(p,ix,iy+2,b2)]*w[fencode(p,ix,iy+2,mom2)]/w[fencode(p,ix,iy+2,rho)];
	      // fim2=w[fencode(p,ix,iy-2,b2)]*w[fencode(p,ix,iy-2,mom2)]/w[fencode(p,ix,iy-2,rho)];
	       ddcy=evalgrad(fi,fim1,0,0,p,1);
        break;
	case 2:
	       fi=w[fencode(p,ix+1,iy,b3)]*w[fencode(p,ix+1,iy,mom1)]/w[fencode(p,ix+1,iy,rho)];
	       fim1=w[fencode(p,ix-1,iy,b3)]*w[fencode(p,ix-1,iy,mom1)]/w[fencode(p,ix-1,iy,rho)];
	       //fip2=w[fencode(p,ix+2,iy,b3)]*w[fencode(p,ix+2,iy,mom1)]/w[fencode(p,ix+2,iy,rho)];
	       //fim2=w[fencode(p,ix-2,iy,b3)]*w[fencode(p,ix-2,iy,mom1)]/w[fencode(p,ix-2,iy,rho)];
	       ddcx=evalgrad(fi,fim1,0,0,p,0);
	       fi=w[fencode(p,ix,iy+1,b3)]*w[fencode(p,ix,iy+1,mom2)]/w[fencode(p,ix,iy+1,rho)];
	       fim1=w[fencode(p,ix,iy-1,b3)]*w[fencode(p,ix,iy-1,mom2)]/w[fencode(p,ix,iy-1,rho)];
	       //fip2=w[fencode(p,ix,iy+2,b3)]*w[fencode(p,ix,iy+2,mom2)]/w[fencode(p,ix,iy+2,rho)];
	       //fim2=w[fencode(p,ix,iy-2,b3)]*w[fencode(p,ix,iy-2,mom2)]/w[fencode(p,ix,iy-2,rho)];
	       ddcy=evalgrad(fi,fim1,0,0,p,1);
        break;
  }
  ddc2=(isnan(ddcx)?0:ddcx)+(isnan(ddcy)?0:ddcy);
  return(ddc1-ddc2);

}

__device__ __host__
float ddotcurrentenergy (float *dw, float *wd, float *w, struct params *p,int ix, int iy) {

 // float ddc=0;
  float dd1,dd2,dd3;
 
  float ddcx,ddcy;
  //float fi, fim1;//fip2=0, fim2=0;
  //float dpi, dpim1;//, dpip2=0, dpim2=0;


  //int field=energy;

  //fi=w[fencode(p,ix+1,iy,energy)]*w[fencode(p,ix+1,iy,mom1)]/w[fencode(p,ix,iy,rho)];
  //fim1=w[fencode(p,ix-1,iy,energy)]*w[fencode(p,ix-1,iy,mom1)]/w[fencode(p,ix-1,iy,rho)];
  //fip2=w[fencode(p,ix+2,iy,energy)]*w[fencode(p,ix+2,iy,mom1)]/w[fencode(p,ix+2,iy,rho)];
 // fim2=w[fencode(p,ix-2,iy,energy)]*w[fencode(p,ix-2,iy,mom1)]/w[fencode(p,ix-2,iy,rho)];
 // ddcx=evalgrad(fi,fim1,0,0,p,0);
  ddcx=evalgrad(w[fencode(p,ix+1,iy,energy)]*w[fencode(p,ix+1,iy,mom1)]/w[fencode(p,ix,iy,rho)],w[fencode(p,ix-1,iy,energy)]*w[fencode(p,ix-1,iy,mom1)]/w[fencode(p,ix-1,iy,rho)],0,0,p,0);

 // fi=w[fencode(p,ix,iy+1,energy)]*w[fencode(p,ix,iy+1,mom2)]/w[fencode(p,ix,iy+1,rho)];
 // fim1=w[fencode(p,ix,iy-1,energy)]*w[fencode(p,ix,iy-1,mom2)]/w[fencode(p,ix,iy-1,rho)];
 // fip2=w[fencode(p,ix,iy+2,energy)]*w[fencode(p,ix,iy+2,mom2)]/w[fencode(p,ix,iy+2,rho)];
  //fim2=w[fencode(p,ix,iy-2,energy)]*w[fencode(p,ix,iy-2,mom2)]/w[fencode(p,ix,iy-2,rho)];
  //ddcy=evalgrad(fi,fim1,0,0,p,1);
  ddcy=evalgrad(w[fencode(p,ix,iy+1,energy)]*w[fencode(p,ix,iy+1,mom2)]/w[fencode(p,ix,iy+1,rho)],w[fencode(p,ix,iy-1,energy)]*w[fencode(p,ix,iy-1,mom2)]/w[fencode(p,ix,iy-1,rho)],0,0,p,1);

  dd1=(isnan(ddcx)?0:ddcx)+(isnan(ddcy)?0:ddcy);


 // dpi=(w[fencode(p,ix+1,iy,b1)]*w[fencode(p,ix+1,iy,mom1)]+w[fencode(p,ix+1,iy,b2)]*w[fencode(p,ix+1,iy,mom2)]+w[fencode(p,ix+1,iy,b3)]*w[fencode(p,ix+1,iy,mom3)])/w[fencode(p,ix+1,iy,rho)];
 // dpim1=(w[fencode(p,ix-1,iy,b1)]*w[fencode(p,ix-1,iy,mom1)]+w[fencode(p,ix-1,iy,b2)]*w[fencode(p,ix-1,iy,mom2)]+w[fencode(p,ix-1,iy,b3)]*w[fencode(p,ix-1,iy,mom3)])/w[fencode(p,ix-1,iy,rho)];
  //dpip2=(w[fencode(p,ix+2,iy,b1)]*w[fencode(p,ix+2,iy,mom1)]+w[fencode(p,ix+2,iy,b2)]*w[fencode(p,ix+2,iy,mom2)]+w[fencode(p,ix+2,iy,b3)]*w[fencode(p,ix+2,iy,mom3)])/w[fencode(p,ix+2,iy,rho)];
 // dpim2=(w[fencode(p,ix-2,iy,b1)]*w[fencode(p,ix-2,iy,mom1)]+w[fencode(p,ix-2,iy,b2)]*w[fencode(p,ix-2,iy,mom2)]+w[fencode(p,ix-2,iy,b3)]*w[fencode(p,ix-2,iy,mom3)])/w[fencode(p,ix-2,iy,rho)];

 // fi=dpi*w[fencode(p,ix+1,iy,b1)];
 // fim1=dpim1*w[fencode(p,ix-1,iy,b1)];
  //fip2=dpip2*w[fencode(p,ix+2,iy,b1)];
 // fim2=dpim2*w[fencode(p,ix-2,iy,b1)];
 // ddcx=evalgrad(fi,fim1,0,0,p,0);
 //  ddcx=evalgrad(((w[fencode(p,ix+1,iy,b1)]*w[fencode(p,ix+1,iy,mom1)]+w[fencode(p,ix+1,iy,b2)]*w[fencode(p,ix+1,iy,mom2)]+w[fencode(p,ix+1,iy,b3)]*w[fencode(p,ix+1,iy,mom3)])/w[fencode(p,ix+1,iy,rho)])*w[fencode(p,ix+1,iy,b1)],((w[fencode(p,ix-1,iy,b1)]*w[fencode(p,ix-1,iy,mom1)]+w[fencode(p,ix-1,iy,b2)]*w[fencode(p,ix-1,iy,mom2)]+w[fencode(p,ix-1,iy,b3)]*w[fencode(p,ix-1,iy,mom3)])/w[fencode(p,ix-1,iy,rho)])*w[fencode(p,ix-1,iy,b1)],0,0,p,0);
  ddcx=evalgrad(wd[fencode(p,ix+1,iy,bdotv)]*w[fencode(p,ix+1,iy,b1)],wd[fencode(p,ix-1,iy,bdotv)]*w[fencode(p,ix-1,iy,b1)],0,0,p,1);

 // dpi=(w[fencode(p,ix,iy+1,b1)]*w[fencode(p,ix,iy+1,mom1)]+w[fencode(p,ix,iy+1,b2)]*w[fencode(p,ix,iy+1,mom2)]+w[fencode(p,ix,iy+1,b3)]*w[fencode(p,ix,iy+1,mom3)])/w[fencode(p,ix,iy+1,rho)];
 // dpim1=(w[fencode(p,ix,iy-1,b1)]*w[fencode(p,ix,iy-1,mom1)]+w[fencode(p,ix,iy-1,b2)]*w[fencode(p,ix,iy-1,mom2)]+w[fencode(p,ix,iy-1,b3)]*w[fencode(p,ix,iy-1,mom3)])/w[fencode(p,ix,iy-1,rho)];  
  //dpip2=(w[fencode(p,ix,iy+2,b1)]*w[fencode(p,ix,iy+2,mom1)]+w[fencode(p,ix,iy+2,b2)]*w[fencode(p,ix,iy+2,mom2)]+w[fencode(p,ix,iy+2,b3)]*w[fencode(p,ix,iy+2,mom3)])/w[fencode(p,ix,iy+2,rho)];
  //dpim2=(w[fencode(p,ix,iy-2,b1)]*w[fencode(p,ix,iy-2,mom1)]+w[fencode(p,ix,iy-2,b2)]*w[fencode(p,ix,iy-2,mom2)]+w[fencode(p,ix,iy-2,b3)]*w[fencode(p,ix,iy-2,mom3)])/w[fencode(p,ix,iy-2,rho)];

 // fi=dpi*w[fencode(p,ix,iy+1,b2)];
 // fim1=dpim1*w[fencode(p,ix,iy-1,b2)];
  //fip2=dpip2*w[fencode(p,ix,iy+2,b2)];
  //fim2=dpim2*w[fencode(p,ix,iy-2,b2)];

//fi=w[fencode(p,ix,iy+1,b2)];
//  fim1=w[fencode(p,ix,iy-1,b2)];
  ddcy=evalgrad(wd[fencode(p,ix,iy+1,bdotv)]*w[fencode(p,ix,iy+1,b2)],wd[fencode(p,ix,iy-1,bdotv)]*w[fencode(p,ix,iy-1,b2)],0,0,p,1);
//ddcx=0;
//ddcy=evalgrad(((w[fencode(p,ix,iy+1,b1)]*w[fencode(p,ix,iy+1,mom1)]+w[fencode(p,ix,iy+1,b2)]*w[fencode(p,ix,iy+1,mom2)]+w[fencode(p,ix,iy+1,b3)]*w[fencode(p,ix,iy+1,mom3)])/w[fencode(p,ix,iy+1,rho)])*w[fencode(p,ix,iy+1,b2)],((w[fencode(p,ix,iy-1,b1)]*w[fencode(p,ix,iy-1,mom1)]+w[fencode(p,ix,iy-1,b2)]*w[fencode(p,ix,iy-1,mom2)]+w[fencode(p,ix,iy-1,b3)]*w[fencode(p,ix,iy-1,mom3)])/w[fencode(p,ix,iy-1,rho)])*w[fencode(p,ix,iy-1,b2)],0,0,p,1);

  dd2=(isnan(ddcx)?0:ddcx)+(isnan(ddcy)?0:ddcy);



  ddcx=wd[fencode(p,ix,iy,pressuret)]*grad(w,p,ix,iy,mom1,0)/w[fencode(p,ix,iy,rho)];
  ddcy=wd[fencode(p,ix,iy,pressuret)]*grad(w,p,ix,iy,mom2,1)/w[fencode(p,ix,iy,rho)];


  dd3=(isnan(ddcx)?0:ddcx)+(isnan(ddcy)?0:ddcy);

  
  return(dd1+dd2+dd3);
 //return dd1;
 // return ( ddc);
}

__device__ __host__
int derivrho (float *dw, float *wd, float *w, struct params *p,int ix, int iy) {

  int status=0;
  int field=rho;
        dw[fencode(p,ix,iy,field)]=sourcerho(dw,wd,w,p,ix,iy)-ddotcurrentrho(dw,wd,w,p,ix,iy);
     	//dw[fencode(p,ix,iy,field)]=w[fencode(p,ix,iy,field)]+10;
  return ( status);
}

__device__ __host__
int derivmom (float *dw, float *wd, float *w, struct params *p,int ix, int iy,int field, int direction) {

  int status=0;
     	//dw[fencode(p,ix,iy,field)]=w[fencode(p,ix,iy,field)]+20+5*(2*direction+1);
        dw[fencode(p,ix,iy,field)]=sourcemom(dw,wd,w,p,ix,iy,field,direction)-ddotcurrentmom(dw,wd,w,p,ix,iy,field,direction);
        //dw[fencode(p,ix,iy,field)]=-ddotcurrentmom(dw,wd,w,p,ix,iy,field,direction);

  return ( status);
}

__device__ __host__
int derivb (float *dw, float *wd, float *w, struct params *p,int ix, int iy, int field, int direction) {

  int status=0;
        dw[fencode(p,ix,iy,field)]=sourceb(dw,wd,w,p,ix,iy,field,direction)-ddotcurrentb(dw,wd,w,p,ix,iy,field,direction);

  return ( status);
}

__device__ __host__
int derivenergy (float *dw, float *wd, float *w, struct params *p,int ix, int iy) {

  int status=0;
  int field=energy;
        dw[fencode(p,ix,iy,field)]=sourceenergy(dw,wd,w,p,ix,iy)-ddotcurrentenergy(dw,wd,w,p,ix,iy);

  return ( status);
}

//rho, mom1, mom2, mom3, energy, b1, b2, b3
__device__ __host__
void deriv (float *dw, float *wd, float *w, struct params *p,int ix, int iy, int field) {

  //int status=0;
  switch(field)
  {
     case rho:
      derivrho(dw,wd,w,p,ix,iy);
     break;
     case mom1:
      derivmom(dw,wd,w,p,ix,iy,field,0);
     break;
     case mom2:
      derivmom(dw,wd,w,p,ix,iy,field,1);
     break;
     case mom3:
      derivmom(dw,wd,w,p,ix,iy,field,2);
     break;
     case energy:
       derivenergy(dw,wd,w,p,ix,iy);
     break;
     case b1:
      derivb(dw,wd,w,p,ix,iy,field,0);
     break;
     case b2:
      derivb(dw,wd,w,p,ix,iy,field,1);
     break;
     case b3:
      derivb(dw,wd,w,p,ix,iy,field,2);
     break;
  }
  //return ( status);
}



__global__ void derivcurrent_parallel(struct params *p, float *w, float *wnew, float *wmod, 
    float *dwn1, float *wd)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->ni;
  int nj=p->nj;
  float dt=p->dt;
  float dy=p->dy;
  float dx=p->dx;
  float g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  

   j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);
  if(i>1 && j >1 && i<((p->ni)-2) && j<((p->nj)-2))
	{		               
               for(int f=rho; f<=b3; f++)               
                  wmod[fencode(p,i,j,f)]=w[fencode(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               computebdotv(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++)
               {              
                  deriv(dwn1,wd,wmod,p,i,j,f);
                  //dwn1[fencode(p,i,j,f)]=1.0;
                  __syncthreads();
               }
               
               /*for(int f=rho; f<=b3; f++) 
                  wmod[fencode(p,i,j,f)]=w[fencode(p,i,j,f)]+0.5*dt*dwn1[fencode(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn2,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  wmod[fencode(p,i,j,f)]=w[fencode(p,i,j,f)]+0.5*dt*dwn2[fencode(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn3,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  wmod[fencode(p,i,j,f)]=w[fencode(p,i,j,f)]+dt*dwn3[fencode(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn4,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  {
                  wnew[fencode(p,i,j,f)]=w[fencode(p,i,j,f)]+(dt/6.0)*(
                     dwn1[fencode(p,i,j,f)]+2.0*dwn2[fencode(p,i,j,f)]
                         +2.0*dwn3[fencode(p,i,j,f)]+dwn4[fencode(p,i,j,f)]);
               }*/
                __syncthreads();
              /* for(int f=rho; f<=b3; f++)
                   wnew[fencode(p,i,j,f)]=w[fencode(p,i,j,f)]+dt*dwn1[fencode(p,i,j,f)];
               computej(wnew,wd,p,i,j);
               computepk(wnew,wd,p,i,j);
               computept(wnew,wd,p,i,j);*/ 


	}
 __syncthreads();
  
}

__global__ void derivsource_parallel(struct params *p, float *w, float *wnew, float *wmod, 
    float *dwn1, float *wd)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->ni;
  int nj=p->nj;
  float dt=p->dt;
  float dy=p->dy;
  float dx=p->dx;
  float g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  

   j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);
  if(i>1 && j >1 && i<((p->ni)-2) && j<((p->nj)-2))
	{		               
               for(int f=rho; f<=b3; f++)               
                  wmod[fencode(p,i,j,f)]=w[fencode(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               computebdotv(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++)
               {              
                  deriv(dwn1,wd,wmod,p,i,j,f);
                  //dwn1[fencode(p,i,j,f)]=1.0;
                  __syncthreads();
               }
               
               /*for(int f=rho; f<=b3; f++) 
                  wmod[fencode(p,i,j,f)]=w[fencode(p,i,j,f)]+0.5*dt*dwn1[fencode(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn2,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  wmod[fencode(p,i,j,f)]=w[fencode(p,i,j,f)]+0.5*dt*dwn2[fencode(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn3,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  wmod[fencode(p,i,j,f)]=w[fencode(p,i,j,f)]+dt*dwn3[fencode(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn4,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  {
                  wnew[fencode(p,i,j,f)]=w[fencode(p,i,j,f)]+(dt/6.0)*(
                     dwn1[fencode(p,i,j,f)]+2.0*dwn2[fencode(p,i,j,f)]
                         +2.0*dwn3[fencode(p,i,j,f)]+dwn4[fencode(p,i,j,f)]);
               }*/
                __syncthreads();
              /* for(int f=rho; f<=b3; f++)
                   wnew[fencode(p,i,j,f)]=w[fencode(p,i,j,f)]+dt*dwn1[fencode(p,i,j,f)];
               computej(wnew,wd,p,i,j);
               computepk(wnew,wd,p,i,j);
               computept(wnew,wd,p,i,j);*/ 


	}
 __syncthreads();
  
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}




int cuderivcurrent(struct params **p, float **w, float **wnew, struct params **d_p, float **d_w, float **d_wnew, float **d_wmod, float **d_dwn1, float **d_wd)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
    dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
   int numBlocks = (((*p)->ni)*((*p)->nj)+numThreadsPerBlock-1) / numThreadsPerBlock;

//__global__ void prop_parallel(struct params *p, float *b, float *w, float *wnew, float *wmod, 
  //  float *dwn1, float *dwn2, float *dwn3, float *dwn4, float *wd)
     //init_parallel(struct params *p, float *b, float *u, float *v, float *h)
     derivcurrent_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->ni)* ((*p)->nj))*sizeof(float), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}

int cuderivsource(struct params **p, float **w, float **wnew,struct params **d_p, float **d_w, float **d_wnew,  float **d_wmod, float **d_dwn1, float **d_wd)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
    dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
   int numBlocks = (((*p)->ni)*((*p)->nj)+numThreadsPerBlock-1) / numThreadsPerBlock;

//__global__ void prop_parallel(struct params *p, float *b, float *w, float *wnew, float *wmod, 
  //  float *dwn1, float *dwn2, float *dwn3, float *dwn4, float *wd)
     //init_parallel(struct params *p, float *b, float *u, float *v, float *h)
     derivsource_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->ni)* ((*p)->nj))*sizeof(float), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}







