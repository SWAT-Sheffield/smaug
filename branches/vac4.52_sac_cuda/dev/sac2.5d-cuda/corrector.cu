#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_cor.cuh"
#include "dervfields_cor.cuh"



__global__ void corrector_parallel(struct params *p,  real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd, int order)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[0];
  real dx=p->dx[1];
  int ix[NDIM];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  

   j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);
   ix[0]=i;
   ix[1]=j;
   if(order==1 || order==2)
     dt=(p->dt)/2.0;

  //advance the solution for one of the corrector steps
  if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
	{ 
   
		for(int f=rho; f<NVAR; f++)           
 			//wmod[fencode_cor(p,i,j,f)]=((w[fencode_cor(p,i+1,j,f)]+w[fencode_cor(p,i-1,j,f)]+w[fencode_cor(p,i,j+1,f)]+w[fencode_cor(p,i,j-1,f)])/4.0)+dt*dwn1[(NVAR*ni*nj*(order-1))+fencode_cor(p,i,j,f)];
                   wmod[fencode_cor(p,i,j,f)]=(w[fencode_cor(p,i,j,f)])+dt*dwn1[(NVAR*ni*nj*(order-1))+fencode_cor(p,i,j,f)];
	}

 __syncthreads();

if(i<((p->n[0])) && j<((p->n[1])))
	{		
               //for(int f=rho; f<=b3; f++)
               //{               
               //   wmod[fencode_cor(p,i,j,f)]=w[fencode_cor(p,i,j,f)];
               //   wnew[fencode_cor(p,i,j,f)]=0.0;
               //}
               for(int f=f1; f<NDERV; f++)
                  wd[fencode_cor(p,i,j,f)]=0; 
        }
               __syncthreads();


  if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
	{		               
               #ifdef ADIABHYDRO
               computepk_cor(wmod,wd,p,i,j);
               computept_cor(wmod,wd,p,i,j);

               #else
               computej_cor(wmod,wd,p,i,j);
               computepk_cor(wmod,wd,p,i,j);
               computept_cor(wmod,wd,p,i,j);

               computebdotv_cor(wmod,wd,p,i,j);
               computedivb_cor(wmod,wd,p,i,j);
               #endif
         }
              __syncthreads();
  if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
	{
 //determin cmax
               computec_cor(wmod,wd,p,i,j);
        }
              __syncthreads();


  
}

/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_cor(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}




int cucorrector(struct params **p, real **w, real **wnew, struct params **d_p, real **d_w, real **d_wnew, real **d_wmod, real **d_dwn1, real **d_wd, int order)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (dimproduct_cor(*p)+numThreadsPerBlock-1) / numThreadsPerBlock;

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
     corrector_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd, order);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
 

  //  hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);

     //following used for testing to check current soundspeeds etc
     //hipMemcpy(*w, *d_wd, 7*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}






