#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_cd1a.cuh"
#include "dervfields_cd1a.cuh"



__device__ __host__
int divflux1(real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field,int dir) {

  int direction;
  int status=0;
  real divflux=0;
  //dw[fencode_cd1a(p,ix,iy,field)]= grad_cd1a(wd,p,ix,iy,f1,dir);//+grad_cd1a(wd,p,ix,iy,f2,1); 
dw[fencode_cd1a(p,ix,iy,field)]= grad_cd1a(wd,p,ix,iy,flux,dir);//+grad_cd1a(wd,p,ix,iy,f2,1); 
 /*switch(field)
  {
     case mom1:
       dw[fencode_cd1a(p,ix,iy,field)]+= grad_cd1a(wd,p,ix,iy,pressuret,0);
      break;

    case mom2:
      dw[fencode_cd1a(p,ix,iy,field)]+= grad_cd1a(wd,p,ix,iy,pressuret,1);
      break;


  } */   
 // dw[fencode_cd1a(p,ix,iy,field)]= gradd0_cd1a(wd,p,ix,iy,f1,0)+gradd1_cd1a(wd,p,ix,iy,f2,1);    
  return ( status);
}






__global__ void centdiff1a_parallel(struct params *p, real *w, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real dt, int f, int dir)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int fid;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

   int ip,jp,ipg,jpg;
   fid=0;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));


   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
			 if(i>1 && j >1 && i<(ni-2) && j<(nj-2))
                         //if( i<(ni) && j<(nj))
                               divflux1(dwn1,wd,wmod+order*NVAR*(p->n[0])*(p->n[1]),p,i,j,f,dir);  
 __syncthreads();
}
               // }
    

         // if( i<(ni) && j<(nj))
          //        bc_cont_cd1a(dwn1,p,i,j,f);
            //    __syncthreads();



             // for(int f=rho; f<=mom3; f++)
              // {

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;


                       // switch(dir)
                       // {
                       //  case 0:
                         //if(i<(ni)  && j >1 &&  j<(nj-2))
                         //if( i >1 &&  i<(ni-2)  && j >1 &&  j<(nj-2))
                         if(i>1 && j >1 && i<(ni-2) && j<(nj-2))
                         //if(i>2 && j >2 && i<(ni-3) && j<(nj-3))
                         //if(i<(ni)  && j >3 &&  j<(nj-4))
                         //if(i<(ni)  && j >1 &&  j<(nj-2))
                              wmod[fencode_cd1a(p,i,j,f)+(ordero*NVAR*(p->n[0])*(p->n[1]))]=wmod[fencode_cd1a(p,i,j,f)+(ordero*NVAR*(p->n[0])*(p->n[1]))]-dt*dwn1[fencode_cd1a(p,i,j,f)]; 
                      //   break;
                      //   case 1:
                         //if(i>1 &&  i<(ni-2) && j<(nj))
                         //if(i >1 &&  i<(ni-2)  && j >1 &&  j<(nj-2))
                         //if(i>3 && j >3 && i<(ni-4) && j<(nj-4))
                         //if(i>3 &&  i<(ni-4) && j<(nj))
                      //   if(i>1 &&  i<(ni-2) && j<(nj))
                      //        wmod[fencode_cd1a(p,i,j,f)+(ordero*NVAR*(p->n[0])*(p->n[1]))]=wmod[fencode_cd1a(p,i,j,f)+(ordero*NVAR*(p->n[0])*(p->n[1]))]-dt*dwn1[fencode_cd1a(p,i,j,f)];
                       //  break;
                      //  }


              //  }
 __syncthreads();	
}
 


}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_cd1a(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}




int cucentdiff1a(struct params **p, real **w, struct params **d_p, real **d_w,  real **d_wmod, real **d_dwn1, real **d_wd, int order, int ordero, real dt, int field, int dir)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;
 //  hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
 // if(order==0)
    hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
//hipDeviceSynchronize();

     centdiff1a_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wmod, *d_dwn1,  *d_wd, order, ordero,dt,field,dir);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}


