#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_hdm3.cuh"



__global__ void hyperdifmomsource3_parallel(struct params *p, real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim, int ii, int ii0, real dt)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1;
  real fip,fim1,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  real rdx;
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));

  rdx=(((p->dx[0])*(dim==0))+(p->dx[1])*(dim==1));

   
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;

  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
//if(i<((p->n[0])) && j<((p->n[1])))
  if(i<((p->n[0])) && j<((p->n[1])))
	{		               

dwn1[fencode_hdm3(p,i,j,energy)]=wtemp[fencode_hdm3(p,i,j,tmp6)]*wd[fencode_hdm3(p,i,j,hdnur)]*wtemp[fencode_hdm3(p,i,j,tmp8)]-wtemp[fencode_hdm3(p,i,j,tmp5)]*wd[fencode_hdm3(p,i,j,hdnul)]*wtemp[fencode_hdm3(p,i,j,tmp7)]/(rdx)/2;

dwn1[fencode_hdm3(p,i,j,mom1+ii0)]=(wtemp[fencode_hdm3(p,i,j,tmp3)]*wd[fencode_hdm3(p,i,j,hdnur)]*wtemp[fencode_hdm3(p,i,j,tmp8)]-wtemp[fencode_hdm3(p,i,j,tmp2)]*wd[fencode_hdm3(p,i,j,hdnul)]*wtemp[fencode_hdm3(p,i,j,tmp7)])/(rdx)/2;

/*dwn1[(NVAR*(p->n[0])*(p->n[1]))+fencode_hdm3(p,i,j,energy)]=(

(  wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm3(p,i,j,mom1+field)]+wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm3(p,i+(dim==0),j+(dim==1),mom1+field)])*wd[fencode_hdm3(p,i,j,hdnur)]*grad1r_hdm3(wtemp,p,i,j,tmp1,dim)

-(wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm3(p,i,j,mom1+field)]+wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm3(p,i-(dim==0),j+(dim==1),mom1+field)])*wd[fencode_hdm3(p,i,j,hdnul)]*grad1l_hdm3(wtemp,p,i,j,tmp1,dim))/(((p->dx[0])*(dim==0))+(p->dx[1])*(dim==1))/2;

dwn1[(NVAR*(p->n[0])*(p->n[1]))+fencode_hdm3(p,i,j,mom1+ii0)]=(wtemp[fencode_hdm3(p,i,j,tmp2)]*wd[fencode_hdm3(p,i,j,hdnur)]*grad1r_hdm3(wtemp,p,i,j,tmp1,dim)-wtemp[fencode_hdm3(p,i,j,tmp3)]*wd[fencode_hdm3(p,i,j,hdnul)]*grad1l_hdm3(wtemp,p,i,j,tmp1,dim))/(((p->dx[0])*(dim==0))+(p->dx[1])*(dim==1))/2;*/


/*dwn1[(NVAR*(p->n[0])*(p->n[1]))+fencode_hdm3(p,i,j,energy)]=(

(  wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm3(p,i,j,mom1+field)]+wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm3(p,i+(dim==0),j+(dim==1),mom1+field)])*wd[fencode_hdm3(p,i,j,hdnur)]*grad1r_hdm3(wtemp,p,i,j,tmp1,dim)

-(wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm3(p,i,j,mom1+field)]+wmod[(order*NVAR*(p->n[0])*(p->n[1]))+fencode_hdm3(p,i-(dim==0),j+(dim==1),mom1+field)])*wd[fencode_hdm3(p,i,j,hdnul)]*grad1l_hdm3(wtemp,p,i,j,tmp1,dim))/2;

dwn1[(NVAR*(p->n[0])*(p->n[1]))+fencode_hdm3(p,i,j,mom1+ii0)]=(wtemp[fencode_hdm3(p,i,j,tmp2)]*wd[fencode_hdm3(p,i,j,hdnur)]*grad1r_hdm3(wtemp,p,i,j,tmp1,dim)-wtemp[fencode_hdm3(p,i,j,tmp3)]*wd[fencode_hdm3(p,i,j,hdnul)]*grad1l_hdm3(wtemp,p,i,j,tmp1,dim))/2;*/


   }
}
 __syncthreads();



/*   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
          if( i<(ni) && j<(nj))
          {

                  bc_periodic1_hdm3(dwn1,p,i,j,mom1+ii0);
                  bc_periodic1_hdm3(dwn1,p,i,j,energy);
             }

}
                __syncthreads();

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
          if( i<(ni) && j<(nj))
          {

                  bc_periodic2_hdm3(dwn1,p,i,j,mom1+ii0);
                  bc_periodic2_hdm3(dwn1,p,i,j,energy);
             }
}
                __syncthreads();*/



   
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
			// if(i>1 && j >1 && i<(ni-2) && j<(nj-2))
                        if(i<((p->n[0])) && j<((p->n[1])))
                         {
                              //                                                                                  - sign here same as vac maybe a +
                              wmod[fencode_hdm3(p,i,j,mom1+ii0)+(ordero*NVAR*(p->n[0])*(p->n[1]))]=wmod[fencode_hdm3(p,i,j,mom1+ii0)+(ordero*NVAR*(p->n[0])*(p->n[1]))]+dt*dwn1[fencode_hdm3(p,i,j,mom1+ii0)]; 
                             wmod[fencode_hdm3(p,i,j,energy)+(ordero*NVAR*(p->n[0])*(p->n[1]))]=wmod[fencode_hdm3(p,i,j,energy)+(ordero*NVAR*(p->n[0])*(p->n[1]))]+dt*dwn1[fencode_hdm3(p,i,j,energy)]; 

                         }
              //  }	
}
  __syncthreads();


  
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_hdm3(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}





int cuhyperdifmomsource3(struct params **p, real **w, real **wnew, struct params **d_p, real **d_w, real **d_wnew,  real **d_wmod, real **d_dwn1, real **d_wd, int order, int ordero, real **d_wtemp, int field, int dim, int ii, int ii0, real dt)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
     hyperdifmomsource3_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,ii,ii0,dt);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}







