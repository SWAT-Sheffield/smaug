#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_db.cuh"
#include "dervfields_db.cuh"


__device__ __host__
real dbsourcerho (real *dw, real *wd, real *w, struct params *p,int ix, int iy) {

  real src=0;

  
 
  return src;
}

__device__ __host__
real dbsourcemom (real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field, int direction) {

  real src=0;
  switch(direction)
  {
	case 0:
         src= -wd[fencode_db(p,ix,iy,divb)]*w[fencode_db(p,ix,iy,b1)];
	break;
	case 1:
         src= -wd[fencode_db(p,ix,iy,divb)]*w[fencode_db(p,ix,iy,b2)];
	break;
	/*case 2:
         src= -wd[fencode_db(p,ix,iy,divb)]*w[fencode_db(p,ix,iy,b3)];
	break;*/
  }

  return(isnan(src)?0:src);


}

__device__ __host__
real dbsourceb (real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field, int direction) {

  real src=0;
  switch(direction)
  {
	case 0:
         src= -wd[fencode_db(p,ix,iy,divb)]*w[fencode_db(p,ix,iy,mom1)]/w[fencode_db(p,ix,iy,rho)];
	break;
	case 1:
         src= -wd[fencode_db(p,ix,iy,divb)]*w[fencode_db(p,ix,iy,mom2)]/w[fencode_db(p,ix,iy,rho)];
	break;
	/*case 2:
         src= -wd[fencode_db(p,ix,iy,divb)]*w[fencode_db(p,ix,iy,mom3)]/w[fencode_db(p,ix,iy,rho)];
	break;*/
  }
   return(isnan(src)?0:src);
}

__device__ __host__
real dbsourceenergy (real *dw, real *wd, real *w, struct params *p,int ix, int iy) {

 real src=0;
    src= -wd[fencode_db(p,ix,iy,divb)]*wd[fencode_db(p,ix,iy,bdotv)];
 
  return ( src);
}


__device__ __host__
int dbderivsourcerho (real *dw, real *wd, real *w, struct params *p,int ix, int iy) {

  int status=0;
  int field=rho;
        dw[fencode_db(p,ix,iy,field)]=dw[fencode_db(p,ix,iy,field)]+dbsourcerho(dw,wd,w,p,ix,iy);
     	//dw[fencode_db(p,ix,iy,field)]=w[fencode_db(p,ix,iy,field)]+10;
  return ( status);
}

__device__ __host__
int dbderivsourcemom (real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field, int direction) {

  int status=0;
     	//dw[fencode_db(p,ix,iy,field)]=w[fencode_db(p,ix,iy,field)]+20+5*(2*direction+1);
        dw[fencode_db(p,ix,iy,field)]=dw[fencode_db(p,ix,iy,field)]+dbsourcemom(dw,wd,w,p,ix,iy,field,direction);
        //dw[fencode_db(p,ix,iy,field)]=-ddotcurrentmom(dw,wd,w,p,ix,iy,field,direction);

  return ( status);
}

__device__ __host__
int dbderivsourceb (real *dw, real *wd, real *w, struct params *p,int ix, int iy, int field, int direction) {

  int status=0;
        dw[fencode_db(p,ix,iy,field)]=dw[fencode_db(p,ix,iy,field)]+dbsourceb(dw,wd,w,p,ix,iy,field,direction);

  return ( status);
}

__device__ __host__
int dbderivsourceenergy (real *dw, real *wd, real *w, struct params *p,int ix, int iy) {

  int status=0;
  int field=energy;
        dw[fencode_db(p,ix,iy,field)]=dw[fencode_db(p,ix,iy,field)]+dbsourceenergy(dw,wd,w,p,ix,iy);

  return ( status);
}

//rho, mom1, mom2, mom3, energy, b1, b2, b3
__device__ __host__
void dbderivsource (real *dw, real *wd, real *w, struct params *p,int ix, int iy, int field) {

  //int status=0;
  switch(field)
  {
     case rho:
      dbderivsourcerho(dw,wd,w,p,ix,iy);
     break;
     case mom1:
      dbderivsourcemom(dw,wd,w,p,ix,iy,field,0);
     break;
     case mom2:
      dbderivsourcemom(dw,wd,w,p,ix,iy,field,1);
     break;
     /*case mom3:
      dbderivsourcemom(dw,wd,w,p,ix,iy,field,2);
     break;*/
     case energy:
       dbderivsourceenergy(dw,wd,w,p,ix,iy);
     break;
     case b1:
      dbderivsourceb(dw,wd,w,p,ix,iy,field,0);
     break;
     case b2:
      dbderivsourceb(dw,wd,w,p,ix,iy,field,1);
     break;
    /* case b3:
      dbderivsourceb(dw,wd,w,p,ix,iy,field,2);
     break;*/
  }
  //return ( status);
}


__global__ void divb_parallel(struct params *p, real *w, real *wmod, 
    real *dwn1, real *wd, int order,int ordero, real dt)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;

  int ni=p->n[0];
  int nj=p->n[1];

   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
  if(i<(ni) && j<(nj))
     for(int f=rho; f<=b2; f++)
                dwn1[fencode_db(p,i,j,f)]=0;
   }
 __syncthreads();

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
  if(i>2 && j>2 && i<(ni-2) && j<(nj-2))
	{
           if(p->divbfix)
           {   

               wd[fencode_db(p,i,j,divb)]=grad_db(wmod+order*NVAR*(p->n[0])*(p->n[1]),p,i,j,b1,0)+grad_db(wmod+order*NVAR*(p->n[0])*(p->n[1]),p,i,j,b2,1);
               #ifdef USE_SAC
		wd[fencode_db(p,i,j,divb)]+=grad_db(wmod+order*NVAR*(p->n[0])*(p->n[1]),p,i,j,b1b,0)+grad_db(wmod+order*NVAR*(p->n[0])*(p->n[1]),p,i,j,b2b,1);
                #endif

               for(int f=rho; f<=b2; f++) 
               //for(int f=rho; f<=b3; f++)
               {              
                  //dbderivsource(dwn1+(NVAR*(p->n[0])*(p->n[1])*order),wd,wmod,p,i,j,f);
                  dbderivsource(dwn1,wd,wmod+order*NVAR*(p->n[0])*(p->n[1]),p,i,j,f);
 
               }
            }

	}
}
 __syncthreads();


   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
    if(i>1 && j >1 && i<(ni-2) && j<(nj-2))
                         {
                         if(p->divbfix)
                          { 
                             for(int f=rho; f<=b2; f++) 
                             //                                                  - sign here same as vac maybe a +
                              wmod[fencode_db(p,i,j,f)+(ordero*NVAR*(p->n[0])*(p->n[1]))]=wmod[fencode_db(p,i,j,f)+(ordero*NVAR*(p->n[0])*(p->n[1]))]-dt*dwn1[fencode_db(p,i,j,f)]; 
                          }

                         }
              //  }	
}
  __syncthreads();



  
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_db(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}

int cudivb(struct params **p, real **w,  struct state **state,struct params **d_p, real **d_w,  real **d_wmod, real **d_dwn1, real **d_wd, struct state **d_state, int order,int ordero, real dt)
{
int status=0;

//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
    // prop_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     //hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
    divb_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wmod, *d_dwn1,  *d_wd, order,ordero,dt);
	    //printf("called update\n"); 
    hipDeviceSynchronize();
    //hipMemcpy(*w, *d_w, 8*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
   // hipMemcpy(*state, *d_state, sizeof(struct state), hipMemcpyDeviceToHost);

//hipMemcpy(*wnew, *d_wnew, 8*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 return status;


}



