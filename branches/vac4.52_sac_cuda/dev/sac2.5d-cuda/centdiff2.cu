#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_cd2.cuh"
#include "dervfields_cd2.cuh"


__device__ __host__
real transportflux_cd2 (real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field, int direction) {

  real ddc=0;
  real fi, fim1;
  real  fip2=0, fim2=0;
 // real ddc1;
  real ddcx=0,ddcy=0;

   real flux=0;

   //transport flux
    switch(direction)
  {
     case 0:
     //flux= wd[fencode_cd2(p,ix,iy,vel1)]*w[fencode_cd2(p,ix,iy,field)];
        #ifdef USE_SAC
     flux= w[fencode_cd2(p,ix,iy,mom1)]*w[fencode_cd2(p,ix,iy,field)]/(w[fencode_cd2(p,ix,iy,rho)]+w[fencode_cd2(p,ix,iy,rhob)]);
     //flux= w[fencode_cd2(p,ix,iy,mom1)]*w[fencode_cd2(p,ix,iy,field)]/w[fencode_cd2(p,ix,iy,rho)];

        #else
     flux= w[fencode_cd2(p,ix,iy,mom1)]*w[fencode_cd2(p,ix,iy,field)]/w[fencode_cd2(p,ix,iy,rho)];

        #endif
     break;
     case 1:
        #ifdef USE_SAC
      flux= w[fencode_cd2(p,ix,iy,mom2)]*w[fencode_cd2(p,ix,iy,field)]/(w[fencode_cd2(p,ix,iy,rho)]+w[fencode_cd2(p,ix,iy,rhob)]);
    // flux= w[fencode_cd2(p,ix,iy,mom2)]*w[fencode_cd2(p,ix,iy,field)]/w[fencode_cd2(p,ix,iy,rho)];

        #else
     //flux= wd[fencode_cd2(p,ix,iy,vel2)]*w[fencode_cd2(p,ix,iy,field)];
     flux= w[fencode_cd2(p,ix,iy,mom2)]*w[fencode_cd2(p,ix,iy,field)]/w[fencode_cd2(p,ix,iy,rho)];

        #endif
     break;
    /* case 2:
     flux= wd[fencode_cd2(p,ix,iy,vel3)]*w[fencode_cd2(p,ix,iy,field)];
     break;*/
   }
  return flux;


  //return ( ddc1-ddc2);
}




__device__ __host__
real fluxb1(real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field, int direction) {

  real ddc=0;
  real fi, fim1;
  real  fip2=0, fim2=0;
 // real ddc1;
  real ddcx=0,ddcy=0;

   real flux=0;

    switch(field)
    {
      case b1:


      //if(direction !=0)
        #ifdef USE_SAC





             //  flux= -(w[fencode_cd2(p,ix,iy,field+direction)]+w[fencode_cd2(p,ix,iy,field+4+direction)])*w[fencode_cd2(p,ix,iy,mom1)]/(w[fencode_cd2(p,ix,iy,rho)]+w[fencode_cd2(p,ix,iy,rhob)]);
  flux= -(w[fencode_cd2(p,ix,iy,field+direction)]+w[fencode_cd2(p,ix,iy,field+4+direction)])*w[fencode_cd2(p,ix,iy,mom1)]/(w[fencode_cd2(p,ix,iy,rho)]+w[fencode_cd2(p,ix,iy,rhob)]);

flux+= (w[fencode_cd2(p,ix,iy,field+4)])*w[fencode_cd2(p,ix,iy,mom1+direction)]/(w[fencode_cd2(p,ix,iy,rho)]+w[fencode_cd2(p,ix,iy,rhob)]);
         #endif
        #ifdef USE_VAC
      		//flux= -w[fencode_cd2(p,ix,iy,field)]*w[fencode_cd2(p,ix,iy,mom1+direction)]/w[fencode_cd2(p,ix,iy,rho)];
                flux= -w[fencode_cd2(p,ix,iy,b1+direction)]*w[fencode_cd2(p,ix,iy,mom1)]/w[fencode_cd2(p,ix,iy,rho)];
         #endif
       break;

      case b2:

      //if(direction !=1)
        #ifdef USE_SAC
		flux= -(w[fencode_cd2(p,ix,iy,b1+direction)]+w[fencode_cd2(p,ix,iy,b1+4+direction)])*w[fencode_cd2(p,ix,iy,mom2)]/(w[fencode_cd2(p,ix,iy,rho)]+w[fencode_cd2(p,ix,iy,rhob)]);

               flux+= (w[fencode_cd2(p,ix,iy,field+4)])*w[fencode_cd2(p,ix,iy,mom1+direction)]/(w[fencode_cd2(p,ix,iy,rho)]+w[fencode_cd2(p,ix,iy,rhob)]);


         #endif
        #ifdef USE_VAC

                flux= -w[fencode_cd2(p,ix,iy,b1+direction)]*w[fencode_cd2(p,ix,iy,mom2)]/w[fencode_cd2(p,ix,iy,rho)];

         #endif
       break;

     /* case b3:
      if(direction !=2)
        #ifdef USE_SAC
      		flux= -(w[fencode_cd2(p,ix,iy,field)]+w[fencode_cd2(p,ix,iy,field+5)])*w[fencode_cd2(p,ix,iy,mom1+direction)]/(w[fencode_cd2(p,ix,iy,rho)]+w[fencode_cd2(p,ix,iy,rhob)]);
         #endif
        #ifdef USE_VAC
      		flux= -w[fencode_cd2(p,ix,iy,field)]*w[fencode_cd2(p,ix,iy,mom1+direction)]/w[fencode_cd2(p,ix,iy,rho)];
         #endif
       break;*/

     }


  return flux;
}



__device__ __host__
real fluxe1(real *dw, real *wd, real *w, struct params *p,int ix, int iy, int direction) {

  real ddc=0;
  real fi, fim1;
  real  fip2=0, fim2=0;
 // real ddc1;
  real ddcx=0,ddcy=0;

   real flux=0;

computept_cd2(w,wd,p,ix,iy);

        #ifdef USE_SAC
                
//wd[fencode_cd2(p,ix,iy,ptb)]=  ((p->gamma)-1)*w[fencode_cd2(p,ix,iy,energyb)]- 0.5*((p->gamma)-2)*(w[fencode_cd2(p,ix,iy,b1b)]*w[fencode_cd2(p,ix,iy,b1b)]+w[fencode_cd2(p,ix,iy,b2b)]*w[fencode_cd2(p,ix,iy,b2b)]) ;

 //wd[fencode_cd2(p,ix,iy,pressuret)]=((p->gamma)-1.0)*( w[fencode_cd2(p,ix,iy,energy)]-0.5*(w[fencode_cd2(p,ix,iy,mom1)]*w[fencode_cd2(p,ix,iy,mom1)]+w[fencode_cd2(p,ix,iy,mom2)]*w[fencode_cd2(p,ix,iy,mom2)])/(w[fencode_cd2(p,ix,iy,rho)]+w[fencode_cd2(p,ix,iy,rhob)]));
//wd[fencode_cd2(p,ix,iy,pressuret)]=wd[fencode_cd2(p,ix,iy,pressuret)]-((p->gamma)-2.0)*((w[fencode_cd2(p,ix,iy,b1)]*w[fencode_cd2(p,ix,iy,b1b)]+w[fencode_cd2(p,ix,iy,b2)]*w[fencode_cd2(p,ix,iy,b2b)])+0.5*(w[fencode_cd2(p,ix,iy,b1)]*w[fencode_cd2(p,ix,iy,b1)]+w[fencode_cd2(p,ix,iy,b2)]*w[fencode_cd2(p,ix,iy,b2)]));



      		flux= -w[fencode_cd2(p,ix,iy,b1+direction)]*wd[fencode_cd2(p,ix,iy,bdotv)]+(w[fencode_cd2(p,ix,iy,mom1+direction)]*(wd[fencode_cd2(p,ix,iy,pressuret)]+wd[fencode_cd2(p,ix,iy,ptb)])/(w[fencode_cd2(p,ix,iy,rho)]+w[fencode_cd2(p,ix,iy,rhob)]))+(w[fencode_cd2(p,ix,iy,mom1+direction)]*wd[fencode_cd2(p,ix,iy,energyb)]/(w[fencode_cd2(p,ix,iy,rho)]+w[fencode_cd2(p,ix,iy,rhob)]));

flux -= w[fencode_cd2(p,ix,iy,b1b+direction)]*(w[fencode_cd2(p,ix,iy,b1)]*w[fencode_cd2(p,ix,iy,mom1)]+w[fencode_cd2(p,ix,iy,b2)]*w[fencode_cd2(p,ix,iy,mom2)])/(w[fencode_cd2(p,ix,iy,rho)]+w[fencode_cd2(p,ix,iy,rhob)])
            - w[fencode_cd2(p,ix,iy,b1+direction)]*(w[fencode_cd2(p,ix,iy,b1b)]*w[fencode_cd2(p,ix,iy,mom1)]+w[fencode_cd2(p,ix,iy,b2b)]*w[fencode_cd2(p,ix,iy,mom2)])/(w[fencode_cd2(p,ix,iy,rho)]+w[fencode_cd2(p,ix,iy,rhob)]);

        


         #endif
        #ifdef USE_VAC

wd[fencode_cd2(p,ix,iy,bdotv)]=(w[fencode_cd2(p,ix,iy,b1)]*w[fencode_cd2(p,ix,iy,mom1)]+w[fencode_cd2(p,ix,iy,b2)]*w[fencode_cd2(p,ix,iy,mom2)])/w[fencode_cd2(p,ix,iy,rho)];

//wd[fencode_cd2(p,ix,iy,pressuret)]=(((p->gamma)-1.0)*w[fencode_cd2(p,ix,iy,energy)]+(1.0-0.5*(p->gamma))*(w[fencode_cd2(p,ix,iy,b1)]*w[fencode_cd2(p,ix,iy,b1)]+w[fencode_cd2(p,ix,iy,b2)]*w[fencode_cd2(p,ix,iy,b2)])+0.5*(1.0-(p->gamma))*(w[fencode_cd2(p,ix,iy,mom1)]*w[fencode_cd2(p,ix,iy,mom1)]+w[fencode_cd2(p,ix,iy,mom2)]*w[fencode_cd2(p,ix,iy,mom2)])/w[fencode_cd2(p,ix,iy,rho)]);

 flux= -w[fencode_cd2(p,ix,iy,b1+direction)]*wd[fencode_cd2(p,ix,iy,bdotv)]+(w[fencode_cd2(p,ix,iy,mom1+direction)]*wd[fencode_cd2(p,ix,iy,pressuret)]/w[fencode_cd2(p,ix,iy,rho)]);
//flux= -w[fencode_cd2(p,ix,iy,b1+direction)]*wd[fencode_cd2(p,ix,iy,bdotv)];    
         #endif

  return flux;


  //return ( ddc1-ddc2);
}








__device__ __host__
int computefluxe(real *dw, real *wd, real *w, struct params *p,int ix, int iy,int direction) {

  int field;//, direction;
  int status=0;
  //for(direction=0;direction<2;direction++)
         #ifdef USE_SAC
	     wd[fencode_cd2(p,ix,iy,flux)]= transportflux_cd2(dw,wd,w,p,ix,iy,energy,direction)+fluxe1(dw,wd,w,p,ix,iy,direction);
         #endif
         #ifdef USE_VAC
             wd[fencode_cd2(p,ix,iy,flux)]= transportflux_cd2(dw,wd,w,p,ix,iy,energy,direction)+fluxe1(dw,wd,w,p,ix,iy,direction);
         #endif
        
  return ( status);
}

__device__ __host__
int computefluxb (real *dw, real *wd, real *w, struct params *p,int ix, int iy, int field,int direction) {

 // int direction;
  int status=0;
//  for(direction=0;direction<2;direction++)
//  {

     switch(field)
     {
       case b1 :
         #ifdef USE_SAC
      if(direction==0)
     //wd[fencode_cd2(p,ix,iy,f1)]= transportflux_cd2(dw,wd,w,p,ix,iy,field,direction)+transportflux_cd2(dw,wd,w,p,ix,iy,field+5,direction);
wd[fencode_cd2(p,ix,iy,flux)]= 0.0;
      else
wd[fencode_cd2(p,ix,iy,flux)]= transportflux_cd2(dw,wd,w,p,ix,iy,field,direction)+fluxb1(dw,wd,w,p,ix,iy,field,direction);
         #endif
         #ifdef USE_VAC
      if(direction==0)
    // wd[fencode_cd2(p,ix,iy,f1)]= transportflux_cd2(dw,wd,w,p,ix,iy,field,direction);
 wd[fencode_cd2(p,ix,iy,flux)]= 0.0;
      else
wd[fencode_cd2(p,ix,iy,flux)]= transportflux_cd2(dw,wd,w,p,ix,iy,field,direction)+fluxb1(dw,wd,w,p,ix,iy,field,direction);
         #endif
       break;

       case b2 :
         #ifdef USE_SAC
      if(direction==1)
     //wd[fencode_cd2(p,ix,iy,f1)]= transportflux_cd2(dw,wd,w,p,ix,iy,field,direction)+transportflux_cd2(dw,wd,w,p,ix,iy,field+5,direction);
wd[fencode_cd2(p,ix,iy,flux)]= 0.0;
else
wd[fencode_cd2(p,ix,iy,flux)]= transportflux_cd2(dw,wd,w,p,ix,iy,field,direction)+fluxb1(dw,wd,w,p,ix,iy,field,direction);
         #endif
         #ifdef USE_VAC
      if(direction==1)
    // wd[fencode_cd2(p,ix,iy,f1)]= transportflux_cd2(dw,wd,w,p,ix,iy,field,direction);
 wd[fencode_cd2(p,ix,iy,flux)]= 0.0;
      else
wd[fencode_cd2(p,ix,iy,flux)]= transportflux_cd2(dw,wd,w,p,ix,iy,field,direction)+fluxb1(dw,wd,w,p,ix,iy,field,direction);
         #endif
       break;

   /*    case b3 :
         #ifdef USE_SAC
      if(direction==2)
         //wd[fencode_cd2(p,ix,iy,f1)]= transportflux_cd2(dw,wd,w,p,ix,iy,field,direction)+transportflux_cd2(dw,wd,w,p,ix,iy,field+5,direction);
wd[fencode_cd2(p,ix,iy,f1)]= 0;
      else
         wd[fencode_cd2(p,ix,iy,f1)]= transportflux_cd2(dw,wd,w,p,ix,iy,field,direction)+fluxb1(dw,wd,w,p,ix,iy,field,direction);
         #endif
         #ifdef USE_VAC
       if(direction==2)
     //wd[fencode_cd2(p,ix,iy,f1)]= transportflux_cd2(dw,wd,w,p,ix,iy,field,direction);
wd[fencode_cd2(p,ix,iy,f1)]= 0;
       else
       wd[fencode_cd2(p,ix,iy,f1)]= transportflux_cd2(dw,wd,w,p,ix,iy,field,direction)+fluxb1(dw,wd,w,p,ix,iy,field,direction);
         #endif
       break;*/

    }
   
 // }     
  return ( status);
}






//rho, mom1, mom2, mom3, energy, b1, b2, b3
__device__ __host__
void computeflux_cd2 (real *dw, real *wd, real *w, struct params *p,int ix, int iy, int field,int dir) {

  //int status=0;
  switch(field)
  {
     case energy:
      computefluxe(dw,wd,w,p,ix,iy,dir);
      computevel_cd2(w,wd,p,ix,iy);
      // add the following terms for SAC
      // del((b bb+ bb b).v)+ptb del v - bb bb del v
     break;
     case b1:
      computefluxb(dw,wd,w,p,ix,iy,field,dir);
     break;
     case b2:
       computefluxb(dw,wd,w,p,ix,iy,field,dir);
     break;
     /*case b3:
      computefluxb(dw,wd,w,p,ix,iy,field);
     break;*/
  }
  //return ( status);
}



__global__ void centdiff2_parallel(struct params *p, real *w, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real dt,int f,int dir)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j,fid;
 // int index;
  int ni=p->n[0];
  int nj=p->n[1];
 // real dt=p->dt;
  //real dy=p->dx[1];
 // real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  

   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));



   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
               //for(int f=energy; f<NVAR; f++)
               //{
			if(i<(ni) && j<(nj))
                        {
                            dwn1[fencode_cd2(p,i,j,f)]=0.0;

                 	   // for(fid=0;fid<2;fid++)
                               wd[fencode_cd2(p,i,j,flux)]=0.0;
                            // wmod[fencode_cd2(p,i,j,flux)+order*NVAR*(p->n[0])*(p->n[1])]=0.0;

                        }
 __syncthreads();
}
                            

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;                             
	
			//if( i<(ni) && j<(nj))
                  		//computeflux_cd2(dwn1,wd,wmod,p,i,j,f);
                                //computeflux_cd2(dwn1,wd,wmod+order*NVAR*(p->n[0])*(p->n[1]),p,i,j,f,dir); 


                        switch(dir)
                        {
                         case 0:
                         if(i<(ni)  && j >1 &&  j<(nj-1))
                          //if(i>1 && i<(ni-1)  && j >2 &&  j<(nj-3))
                            computeflux_cd2(dwn1,wd,wmod+order*NVAR*(p->n[0])*(p->n[1]),p,i,j,f,dir); 
                         break;
                         case 1:
                         if(i>1 &&  i<(ni-1) && j<(nj))
                         //if(j>1 && i>2 &&  i<(ni-3) && j<(nj))
                            computeflux_cd2(dwn1,wd,wmod+order*NVAR*(p->n[0])*(p->n[1]),p,i,j,f,dir); 
                         break;
                        }

               //}
                        //might need to set boundaries correctly 
   __syncthreads();
}
                     



                         
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_cd2(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}




int cucentdiff2(struct params **p, real **w, struct params **d_p, real **d_w,  real **d_wmod, real **d_dwn1, real **d_wd, int order,int ordero, real dt, int field,int dir)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;
 //  hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
 // if(order==0)
    hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
     centdiff2_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wmod, *d_dwn1,  *d_wd, order,ordero,dt,field,dir);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}


