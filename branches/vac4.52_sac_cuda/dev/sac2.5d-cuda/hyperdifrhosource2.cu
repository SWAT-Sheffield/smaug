#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_hdr2.cuh"



__global__ void hyperdifrhosource2_parallel(struct params *p, real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim, real dt)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii,ii1,ii0;
  real fip,fim1,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;
  real rdx;

   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));


   
  rdx=(((p->dx[0])*(dim==0))+(p->dx[1])*(dim==1));

  
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;

  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
  //if(i>1 && j >1 && i<((p->n[0])-1) && j<((p->n[1])-1))
  //if(i>32 && j >32 && i<((p->n[0])-32) && j<((p->n[1])-32))
  if(i<((p->n[0])) && j<((p->n[1])))
  {
     

//dwn1[fencode_hdr2(p,i,j,field)]=( wd[fencode_hdr2(p,i,j,hdnur)] * grad1r_hdr2(wmod+order*NVAR*(p->n[0])*(p->n[1]),p,i,j,rho,dim) - wd[fencode_hdr2(p,i,j,hdnul)] *grad1l_hdr2(wmod+order*NVAR*(p->n[0])*(p->n[1]),p,i,j,rho,dim)             )/rdx;
//dwn1[fencode_hdr2(p,i,j,field)]=( wtemp[fencode_hdr2(p,i,j,hdnur)] * grad1r_hdr2(wmod+order*NVAR*(p->n[0])*(p->n[1]),p,i,j,rho,dim) - wtemp[fencode_hdr2(p,i,j,hdnul)] *grad1l_hdr2(wmod+order*NVAR*(p->n[0])*(p->n[1]),p,i,j,rho,dim)             );

dwn1[fencode_hdr2(p,i,j,field)]=( wd[fencode_hdr2(p,i,j,hdnur)] * wtemp[fencode_hdr2(p,i,j,tmp1)] - wd[fencode_hdr2(p,i,j,hdnul)] *wtemp[fencode_hdr2(p,i,j,tmp2)]            )/rdx;

                              wmod[fencode_hdr2(p,i,j,field)+(ordero*NVAR*(p->n[0])*(p->n[1]))]=wmod[fencode_hdr2(p,i,j,field)+(ordero*NVAR*(p->n[0])*(p->n[1]))]+dt*dwn1[fencode_hdr2(p,i,j,field)]; 
  }
}
__syncthreads();




 
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_hdr2(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}





int cuhyperdifrhosource2(struct params **p, real **w, real **wnew, struct params **d_p, real **d_w, real **d_wnew,  real **d_wmod, real **d_dwn1, real **d_wd, int order, int ordero,real **d_wtemp, int field, int dim, real dt)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)

     hyperdifrhosource2_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,dt);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}







