#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_hdv.cuh"

__device__ __host__
void bc_periodic1_temp2(real *wt, struct params *p,int i, int j, int f) {

                if(i==0 )                
                    wt[fencode_hdv(p,i,j,f)]=wt[fencode_hdv(p,5,j,f)];
                else if((i==((p->n[0])+1)) )                
                    wt[fencode_hdv(p,i,j,f)]=wt[fencode_hdv(p,i-6,j,f)];
                else if(j==0  )                
                  wt[fencode_hdv(p,i,j,f)]=wt[fencode_hdv(p,i,5,f)];
                else if((j==((p->n[1])+1)) )                
                  wt[fencode_hdv(p,i,j,f)]=wt[fencode_hdv(p,i,j-6,f)];

 


}

__device__ __host__
void bc_periodic2_temp2(real *wt, struct params *p,int i, int j, int f) {


               if(i<1 && j<1)
                {
                  if(i==j)
                    //wt[fencode_hdv(p,i,j,f)]=wt[fencode_hdv(p,(p->n[0])-3+i,j,f)];
                    wt[fencode_hdv(p,i,j,f)]=wt[fencode_hdv(p,i,5,f)];
                  else                  
                    //wt[fencode_hdv(p,i,j,f)]=wt[fencode_hdv(p,i,(p->n[1])-3+j,f)];
                    wt[fencode_hdv(p,i,j,f)]=wt[fencode_hdv(p,5,j,f)];                                    
                }
                else if(i<1 && j>((p->n[1])+1))
                {
                  if(i==(j-(p->n[1])+1))                  
                    //wt[fencode_hdv(p,i,j,f)]=wt[fencode_hdv(p,(p->n[0])-3+i,4-(p->n[1])+j,f)];
                    wt[fencode_hdv(p,i,j,f)]=wt[fencode_hdv(p,5,j,f)];                                     
                  else                  
                    wt[fencode_hdv(p,i,j,f)]=wt[fencode_hdv(p,i,j-6,f)];                                     
                }
                else if(i>((p->n[0])+1) && j<1)
                {
                  if((i-(p->n[0])+1)==j)                  
                    wt[fencode_hdv(p,i,j,f)]=wt[fencode_hdv(p,i-6,j,f)];                                    
                  else                  
                   wt[fencode_hdv(p,i,j,f)]=wt[fencode_hdv(p,i,5,f)];                                    
                }
                else if(i>((p->n[0])+1) && j>((p->n[1])+1))
                {
                  if(i==j)                  
                    wt[fencode_hdv(p,i,j,f)]=wt[fencode_hdv(p,i,j-6,f)];                                    
                  else                  
                    wt[fencode_hdv(p,i,j,f)]=wt[fencode_hdv(p,i-6,j,f)];                                    
                }                       
                 
                




}



__global__ void hyperdifvisc_parallel(struct params *p, real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd, int order, real *wtemp, real *wtemp1, real *wtemp2, int field, int dim,int hand)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int is,js;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  real maxt=0,max3=0, max1=0;
  
   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));

int bfac1,bfac2,bfac3;
//int bfac1=(field==rho || field>mom2)+(field>rho && field<energy);
//int bfac2= (field==rho || field>mom2);
//int bfac3=(field>rho && field<energy);
int shift=order*NVAR*(p->n[0])*(p->n[1]);

switch(field)
{
    case rho:
      bfac1=1.0;
      bfac2=1.0;
      bfac3=0.0;
    break;
    case mom1:
    case mom2:
      bfac1=1.0;
      bfac2=0.0;
      bfac3=1.0;
    break;
    case energy:
      bfac1=1.0;
      bfac2=1.0;
      bfac3=0.0;
    break;
    case b1:
    case b2:
      bfac1=1.0;
      bfac2=1.0;
      bfac3=0.0;
    break;
}

//init temp1 and temp2 to zero 
//the compute element initialising n[0] or n[1] element must do +1 and +2
//this is because we fit the problem geometrically to nixnj elements 
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
    //set viscosities
   if(i<((p->n[0])) && j<((p->n[1])))
   {

        for(int f=d1r; f<=d3l; f++)
                 wtemp1[fencode_hdv(p,i,j,f)]=0;
      wtemp2[fencode_hdv(p,i,j,tmpnui)]=0;
      if(i==((p->n[0])-1))
      {
        for(int f=d1r; f<=d3l; f++)
                 wtemp1[fencode_hdv(p,i+1,j,f)]=0;
        wtemp2[fencode_hdv(p,i+1,j,tmpnui)]=0;
        wtemp2[fencode_hdv(p,i+2,j,tmpnui)]=0;
      }
      if(j==((p->n[1])-1))
      {
          for(int f=d1r; f<=d3l; f++)
                 wtemp1[fencode_hdv(p,i,j+1,f)]=0;
      }
      wtemp2[fencode_hdv(p,i,j+1,tmpnui)]=0;
      wtemp2[fencode_hdv(p,i,j+2,tmpnui)]=0;


   }

  }
   __syncthreads();




   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
    //set viscosities
   if(i<((p->n[0])) && j<((p->n[1])))
   {
        for(int f=tmp1; f<=tmp6; f++)
                 wtemp[fencode_hdv(p,i,j,f)]=0;


        //temp value for viscosity

       //tmp6  tmpnu
#ifdef USE_SAC
     if(field !=energy)
        wtemp[fencode_hdv(p,i,j,tmp6)]=wmod[fencode_hdv(p,i,j,field)+shift]/(bfac2+bfac3*((wmod[fencode_hdv(p,i,j,rho)+shift] +wmod[fencode_hdv(p,i,j,rhob)+shift])));

     else
        wtemp[fencode_hdv(p,i,j,tmp6)]=wmod[fencode_hdv(p,i,j,energy)+shift]-0.5*(wmod[fencode_hdv(p,i,j,b1)+shift]*wmod[fencode_hdv(p,i,j,b1)+shift]+wmod[fencode_hdv(p,i,j,b2)+shift]*wmod[fencode_hdv(p,i,j,b2)+shift])+(wmod[fencode_hdv(p,i,j,mom1)+shift]*wmod[fencode_hdv(p,i,j,mom1)+shift]+wmod[fencode_hdv(p,i,j,mom2)+shift]*wmod[fencode_hdv(p,i,j,mom2)+shift])/(wmod[fencode_hdv(p,i,j,rho)+shift]+wmod[fencode_hdv(p,i,j,rhob)+shift] );

#else
     if(field !=energy)
        wtemp[fencode_hdv(p,i,j,tmp6)]=wmod[fencode_hdv(p,i,j,field)+shift]/(bfac2+bfac3*(wmod[fencode_hdv(p,i,j,rho)+shift] ));

     else
        wtemp[fencode_hdv(p,i,j,tmp6)]=wmod[fencode_hdv(p,i,j,energy)+shift]-0.5*(wmod[fencode_hdv(p,i,j,b1)+shift]*wmod[fencode_hdv(p,i,j,b1)+shift]+wmod[fencode_hdv(p,i,j,b2)+shift]*wmod[fencode_hdv(p,i,j,b2)+shift])+(wmod[fencode_hdv(p,i,j,mom1)+shift]*wmod[fencode_hdv(p,i,j,mom1)+shift]+wmod[fencode_hdv(p,i,j,mom2)+shift]*wmod[fencode_hdv(p,i,j,mom2)+shift])/(wmod[fencode_hdv(p,i,j,rho)+shift] );

#endif
        wd[fencode_hdv(p,i,j,hdnur)]=0;
        wd[fencode_hdv(p,i,j,hdnul)]=0;
   }

}
   __syncthreads();

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
    //set viscosities
   if( i<((p->n[0])) && j<((p->n[1])))
   {
     //tmp6 is tmp_nuI
     wtemp2[fencode_hdv(p,i+1,j+1,tmpnui)]=wtemp[fencode_hdv(p,i,j,tmp6)];

   }

   }
   __syncthreads();
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
          if( i<(ni) && j<(nj))
          {

                  bc_periodic1_temp2(wtemp2,p,i,j,tmpnui);
                  if(i==((p->n[0])-1))
                  {
                  bc_periodic1_temp2(wtemp2,p,i+1,j,tmpnui);
                  bc_periodic1_temp2(wtemp2,p,i+2,j,tmpnui);


                  }

                  if(j==((p->n[1])-1))
                  {
                  bc_periodic1_temp2(wtemp2,p,i,j+1,tmpnui);
                  bc_periodic1_temp2(wtemp2,p,i,j+2,tmpnui);
                   }

      
          }

}
                __syncthreads();

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
          if( i<(ni) && j<(nj))
          {
                  //bc_cont_cd1(dwn1,p,i,j,f1+fid);
                  bc_periodic2_temp2(wtemp2,p,i,j,tmpnui);
                  if(i==((p->n[0])-1))
                  {
                  bc_periodic2_temp2(wtemp2,p,i+1,j,tmpnui);
                  bc_periodic2_temp2(wtemp2,p,i+2,j,tmpnui);


                  }

                  if(j==((p->n[1])-1))
                  {
                  bc_periodic1_temp2(wtemp2,p,i,j+1,tmpnui);
                  bc_periodic1_temp2(wtemp2,p,i,j+2,tmpnui);
                   }
           }
}
                __syncthreads();

   //tmp1  tmp_nuI
   //tmp2  d3r
    //tmp3 d1r
//tmp4    md3r
//tmp5    md1r
//tmp6    d3l
//tmp7    d1l
//tmp8    md3l
//tmp9    md1l




   //tmp1  tmp_nuI
 
//compute d3r and d1r
   //tmp2  d3r
    //tmp3 d1r

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
 
   //if(i>1 && j>1 && i<((p->n[0])-1) && j<((p->n[1])-1))
   if(i<((p->n[0])-1) && j<((p->n[1])-1))            
   { 
     if(hand==0)
     {
           wtemp1[fencode_hdv(p,i+2,j+2,d3r)]=fabs(3.0*(wtemp2[fencode_hdv(p,i+2+(dim==0),j+2+(dim==1),tmpnui)] - wtemp2[fencode_hdv(p,i+2,j+2,tmpnui)] ) - (wtemp2[fencode_hdv(p,i+2*(dim==0),j+2*(dim==1),tmpnui)] - wtemp2[fencode_hdv(p,i+2-(dim==0),j+2-(dim==1),tmpnui)]    ));
     }
     else
     {
           wtemp1[fencode_hdv(p,i+2,j+2,d3l)]=fabs(3.0*(wtemp2[fencode_hdv(p,i+2+(dim==0),j+2+(dim==1),tmpnui)] - wtemp2[fencode_hdv(p,i+2,j+2,tmpnui)]) - (wtemp2[fencode_hdv(p,i+(dim==0),j+(dim==1),tmpnui)] - wtemp2[fencode_hdv(p,i+2-(dim==0),j-(dim==1),tmpnui)]    ));
     }
   }
}
   __syncthreads();
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;

   if(i<((p->n[0])+(dim==0)-1) && j<((p->n[1])+(dim==1)-1))            
   { 
     if(hand==0)
     {

           wtemp1[fencode_hdv(p,i,j,d1r)]=fabs((wtemp2[fencode_hdv(p,i+1+(dim==0),j+1+(dim==1),tmpnui)] - wtemp2[fencode_hdv(p,i+1,j+1,tmpnui)] ));
     }
     else
     {
           wtemp1[fencode_hdv(p,i,j,d1l)]=fabs((wtemp2[fencode_hdv(p,i+1,j+1,tmpnui)] - wtemp2[fencode_hdv(p,i+1-(dim==0),j+1-(dim==1),tmpnui)] ));
     }
   }
}
   __syncthreads();



/*   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
          if( i<(ni) && j<(nj))
            {
                  bc_periodic1_hdv(wtemp,p,i,j,tmp2);
                  bc_periodic1_hdv(wtemp,p,i,j,tmp3);
             }

}
                __syncthreads();

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
          if( i<(ni) && j<(nj))
            {
                  bc_periodic2_hdv(wtemp,p,i,j,tmp2);
                  bc_periodic2_hdv(wtemp,p,i,j,tmp3);
             }
}
                __syncthreads();*/



  //compute md3r and md1r
//tmp4    md3r
//tmp5    md1r
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
  if( i<((p->n[0])) && j<((p->n[1])))            
   {
         maxt=0;
         for(is=-(dim==0); is<=(dim==0); is++)
                for(js=-(dim==1); js<=(dim==1); js++)
                {
                   if(wtemp1[fencode_hdv(p,i+is,j+js,d3r+hand)]>maxt)
                         maxt=wtemp1[fencode_hdv(p,i+is,j+js,d3r+hand)];

                }
          wtemp[fencode_hdv(p,i,j,tmp4)]=maxt;

         maxt=0;
         for(is=-2*(dim==0); is<=2*(dim==0); is++)
                for(js=-2*(dim==1); js<=2*(dim==1); js++)
                {
                   if(wtemp1[fencode_hdv(p,i+is,j+js,d1r+hand)]>maxt)
                        maxt=wtemp1[fencode_hdv(p,i+is,j+js,d1r+hand)];

                }
          wtemp[fencode_hdv(p,i,j,tmp5)]=maxt;
   }
}
   __syncthreads();





 /*  for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
          if( i<(ni) && j<(nj))
            {
                  bc_periodic1_hdv(wtemp,p,i,j,tmp4);
                  bc_periodic1_hdv(wtemp,p,i,j,tmp5);
             }

}
                __syncthreads();

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
          if( i<(ni) && j<(nj))
            {
                  bc_periodic2_hdv(wtemp,p,i,j,tmp4);
                  bc_periodic2_hdv(wtemp,p,i,j,tmp5);
             }
}
                __syncthreads();*/



   p->maxviscoef=0;


    //finally update nur and nul
//tmp4    md3r
//tmp5    md1r
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;


   if(i<((p->n[0])) && j<((p->n[1])))
   {
     if(wtemp[fencode_hdv(p,i,j,tmp5)]>0)
	wd[fencode_hdv(p,i,j,hdnur+hand)]=((dim==0)*(p->dx[0])+(dim==1)*(p->dx[1]))*(p->cmax)*(p->chyp[field])*wtemp[fencode_hdv(p,i,j,tmp4)]/wtemp[fencode_hdv(p,i,j,tmp5)];


     else
        wd[fencode_hdv(p,i,j,hdnur+hand)]=0;


   }
}
 __syncthreads();



 
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_hdv(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}





int cuhyperdifvisc(struct params **p, real **w, real **wnew, struct params **d_p, real **d_w, real **d_wnew,  real **d_wmod, real **d_dwn1, real **d_wd, int order, real **d_wtemp, real **d_wtemp1, real **d_wtemp2, int field, int dim,int hand)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
     hyperdifvisc_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd, order, *d_wtemp,*d_wtemp1,*d_wtemp2, field, dim,hand);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_hdv,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_hdv,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_hdv,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_hdv, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}







